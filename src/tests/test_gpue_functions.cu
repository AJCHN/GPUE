#include "hip/hip_runtime.h"
/*
* test_gpue_functions.cu - GPUE2: Split Operator based GPU solver for Nonlinear 
* Schrodinger Equation, Copyright (C) 2018, Lee J. O'Riordan, James Schloss
*/

//###########################################################################################################//

#include<assert.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "../colonel/gpu_functions.cu"


void test_scalVectMult(){
	int xDim, yDim;
	xDim=256;
	yDim=256;
        double *v1, *v1_gpu;
        v1 = (double*) malloc(sizeof(double)*xDim*yDim);
        hipMalloc((void**) &v1_gpu, sizeof(double)*xDim*yDim);

        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
                        v1[i*yDim + j] = 1.0;
                }
        }
        hipMemcpy(v1_gpu, v1, sizeof(double)*xDim*yDim, hipMemcpyHostToDevice);
        scalVecMult_dd<<<256,256>>>(v1_gpu, 2.0, v1_gpu);       
        hipMemcpy(v1, v1_gpu, sizeof(double)*xDim*yDim, hipMemcpyDeviceToHost);
        printf("%e\n",v1[0]);
        vecVecMult_dd<<<256,256>>>(v1_gpu, v1_gpu, v1_gpu);       
        hipMemcpy(v1, v1_gpu, sizeof(double)*xDim*yDim, hipMemcpyDeviceToHost);
        printf("%e\n",v1[0]);
	free(v1);hipFree(v1_gpu);

	//#######################################################################

        double2 *v2, *v2_gpu;
        v2 = (double2*) malloc(sizeof(double2)*xDim*yDim);
        hipMalloc((void**) &v2_gpu, sizeof(double2)*xDim*yDim);

        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
                        v2[i*yDim + j].x = 1.0;
                        v2[i*yDim + j].y = 1.0;
                }
        }
        hipMemcpy(v2_gpu, v2, sizeof(double2)*xDim*yDim, hipMemcpyHostToDevice);
        scalVecMult_d2d<<<256,256>>>(v2_gpu, 2.0, v2_gpu);       
        hipMemcpy(v2, v2_gpu, sizeof(double2)*xDim*yDim, hipMemcpyDeviceToHost);
        printf("Re=%e	Im=%e\n",v2[0].x,v2[0].y);
        vecVecMult_d2d2<<<256,256>>>(v2_gpu, v2_gpu, v2_gpu);       
        hipMemcpy(v2, v2_gpu, sizeof(double2)*xDim*yDim, hipMemcpyDeviceToHost);
        printf("Re=%e	Im=%e\n",v2[0].x,v2[0].y);
}

void test_sum(){
	int xDim, yDim;
	const int threads = 128;
	xDim=256;
	yDim=256;
        double *v1, *v1_gpu;
        v1 = (double*) malloc(sizeof(double)*xDim*yDim);
        hipMalloc((void**) &v1_gpu, sizeof(double)*xDim*yDim);

        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
                        v1[i*yDim + j] = 1.0;
                }
        }
        hipMemcpy(v1_gpu, v1, sizeof(double)*xDim*yDim, hipMemcpyHostToDevice);
        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
                       v1[0] += v1[i*yDim + j];
                }
        }
       	printf("%e\n",v1[0]);
	sumVector_d<threads><<<xDim*yDim/threads,threads,threads*sizeof(double)>>>(v1_gpu, v1_gpu, (unsigned int)threads*8);
        hipMemcpy(v1, v1_gpu, sizeof(double)*xDim*yDim, hipMemcpyDeviceToHost);
	for(int i=0; i<xDim; i++)
		for(int j=0; j<yDim; ++j)
        		printf("[%d,%d]=%e\n",i,j,v1[i*yDim + j]);
       	printf("%e\n",v1[0]);
}
void test_transpose(){
	int xDim, yDim;
	int temp;
	const int threads = 16;
	xDim=20;
	yDim=20;
        int *v1, *v2, *v1_gpu, *v2_gpu;
        v1 = (int*) malloc(sizeof(int)*xDim*yDim);
        v2 = (int*) malloc(sizeof(int)*xDim*yDim);
        hipMalloc((void**) &v1_gpu, sizeof(int)*xDim*yDim);
        hipMalloc((void**) &v2_gpu, sizeof(int)*xDim*yDim);

	//printf("Allocate blocks\n");
        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
                        v1[i*yDim + j] = i*yDim + j;
                        v2[i*yDim + j] = i*yDim + j;
			//printf("V1[%d,%d]=%d	",i,j, v1[i*yDim +j]);
			printf("%d,%d,%d\n",i,j, v1[i*yDim +j]);
                }
        }
	printf("\n");

	printf("CPU Transpose 1 OP\n");
        hipMemcpy(v1_gpu, v1, sizeof(int)*xDim*yDim, hipMemcpyHostToDevice);
        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
			v2[i*yDim + j] = v1[j*xDim + i];
			printf("V2[%d,%d]=%d	",i,j, v2[i*yDim + j]);
                }
		printf("\n");
        }
	printf("\n");

	printf("CPU Transpose 2 IP\n");
        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
			temp = v1[i*yDim + j];
			v1[i*yDim + j] = v1[j*yDim + i];
			v1[j*xDim + i] = temp;
			printf("V1[%d,%d]=%d	",i,j, v1[i*yDim + j]);
                }
		printf("\n");
        }
	printf("\n");
	printf("GPU Transpose 1 IP\n");
	matTrans<<<1,128>>>(v1_gpu, v2_gpu);
	matTrans<<<1,128>>>(v1_gpu, v1_gpu);
        hipMemcpy(v1, v1_gpu, sizeof(int)*xDim*yDim, hipMemcpyDeviceToHost);
        hipMemcpy(v2, v2_gpu, sizeof(int)*xDim*yDim, hipMemcpyDeviceToHost);
	for(int i=0; i<xDim; i++){
		for(int j=0; j<yDim; ++j){
        		printf("G1[%d,%d]=%d	",i,j,v1[i*yDim + j]);
		}
		printf("\n");
	}
	printf("\n");
	printf("GPU Transpose 2 OP\n");
	for(int i=0; i<xDim; i++){
		for(int j=0; j<yDim; ++j){
        		printf("G2[%d,%d]=%d	",i,j,v2[i*yDim + j]);
		}
		printf("\n");
	}
}

int main(){
	//test_scalVectMult();
	//test_sum();
	test_transpose();
	return 0;
}
