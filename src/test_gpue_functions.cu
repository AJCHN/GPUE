#include "hip/hip_runtime.h"
#include<assert.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include"gpu_functions.cu"


void test_scalVectMult(){
	int xDim, yDim;
	xDim=256;
	yDim=256;
        double *v1, *v1_gpu;
        v1 = (double*) malloc(sizeof(double)*xDim*yDim);
        hipMalloc((void**) &v1_gpu, sizeof(double)*xDim*yDim);

        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
                        v1[i*yDim + j] = 1.0;
                }
        }
        hipMemcpy(v1_gpu, v1, sizeof(double)*xDim*yDim, hipMemcpyHostToDevice);
        scalVecMult_dd<<<256,256>>>(v1_gpu, 2.0, v1_gpu);       
        hipMemcpy(v1, v1_gpu, sizeof(double)*xDim*yDim, hipMemcpyDeviceToHost);
        printf("%e\n",v1[0]);
        vecVecMult_dd<<<256,256>>>(v1_gpu, v1_gpu, v1_gpu);       
        hipMemcpy(v1, v1_gpu, sizeof(double)*xDim*yDim, hipMemcpyDeviceToHost);
        printf("%e\n",v1[0]);
	free(v1);hipFree(v1_gpu);

	//#######################################################################

        double2 *v2, *v2_gpu;
        v2 = (double2*) malloc(sizeof(double2)*xDim*yDim);
        hipMalloc((void**) &v2_gpu, sizeof(double2)*xDim*yDim);

        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
                        v2[i*yDim + j].x = 1.0;
                        v2[i*yDim + j].y = 1.0;
                }
        }
        hipMemcpy(v2_gpu, v2, sizeof(double2)*xDim*yDim, hipMemcpyHostToDevice);
        scalVecMult_d2d<<<256,256>>>(v2_gpu, 2.0, v2_gpu);       
        hipMemcpy(v2, v2_gpu, sizeof(double2)*xDim*yDim, hipMemcpyDeviceToHost);
        printf("Re=%e	Im=%e\n",v2[0].x,v2[0].y);
        vecVecMult_d2d2<<<256,256>>>(v2_gpu, v2_gpu, v2_gpu);       
        hipMemcpy(v2, v2_gpu, sizeof(double2)*xDim*yDim, hipMemcpyDeviceToHost);
        printf("Re=%e	Im=%e\n",v2[0].x,v2[0].y);
}

void test_sum(){
	int xDim, yDim;
	const int threads = 128;
	xDim=256;
	yDim=256;
        double *v1, *v1_gpu;
        v1 = (double*) malloc(sizeof(double)*xDim*yDim);
        hipMalloc((void**) &v1_gpu, sizeof(double)*xDim*yDim);

        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
                        v1[i*yDim + j] = 1.0;
                }
        }
        hipMemcpy(v1_gpu, v1, sizeof(double)*xDim*yDim, hipMemcpyHostToDevice);
        for(int i=0; i<xDim; ++i){
                for(int j=0; j<yDim; ++j){
                       v1[0] += v1[i*yDim + j];
                }
        }
       	printf("%e\n",v1[0]);
	sumVector_d<threads><<<xDim*yDim/threads,threads,threads*sizeof(double)>>>(v1_gpu, v1_gpu, (unsigned int)threads);
        hipMemcpy(v1, v1_gpu, sizeof(double)*xDim*yDim, hipMemcpyDeviceToHost);
	for(int i=0; i<xDim; i++)
		for(int j=0; j<yDim; ++j)
        		printf("[%d,%d]=%e\n",i,j,v1[i*yDim + j]);
       	printf("%e\n",v1[0]);
}

int main(){
	test_scalVectMult();
	test_sum();
	return 0;
}
