#include "hip/hip_runtime.h"

#include "../include/split_op.h"
#include "../include/kernels.h"
#include "../include/constants.h"
#include "../include/fileIO.h"
#include "../include/tracker.h"
#include "../include/minions.h"
#include "../include/parser.h"

#include "../include/lattice.h"
#include "../include/node.h"
#include "../include/edge.h"
#include "../include/manip.h"
#include "../include/vort.h"
#include <string>
#include <iostream>


//Declare the static uid values to avoid conflicts. Upper limit of 2^32-1 different instances. Should be reasonable in
// any simulation of realistic timescales.
unsigned int LatticeGraph::Edge::suid = 0;
unsigned int LatticeGraph::Node::suid = 0;
//std::size_t Vtx::Vortex::suid = 0;

char buffer[100]; //Buffer for printing out. Need to replace by a better write-out procedure. Consider binary or HDF.
int verbose; //Print more info. Not curently implemented.
int device; //GPU ID choice.
int kick_it; //Kicking mode: 0 = off, 1 = multiple, 2 = single
int graph=0; //Generate graph from vortex lattice.
double gammaY; //Aspect ratio of trapping geometry.
double omega; //Rotation rate of condensate
double timeTotal;
double angle_sweep; //Rotation angle of condensate relative to x-axis
double x0_shift, y0_shift; //Optical lattice shift parameters.
double Rxy; //Condensate scaling factor.
double a0x, a0y; //Harmonic oscillator length in x and y directions
double sepMinEpsilon=0.0; //Minimum separation for epsilon.

/*
 * Checks CUDA routines have exitted correctly.
 */
int isError(int result, char* c){
    if(result!=0){
        printf("Error has occurred for method %s with return type %d\n",
               c,result);
        exit(result);
    }
    return result;
}

/*
 * Used to perform parallel summation on WFC for normalisation.
 */
void parSum(double2* gpuWfc, double2* gpuParSum, Grid &par){
    // May need to add double l
    int dimnum = par.ival("dimnum");
    double dx = par.dval("dx");
    double dy = par.dval("dy");
    double dz = par.dval("dz");
    dim3 threads = par.threads;
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    int zDim = par.ival("zDim");
    dim3 grid_tmp(xDim*yDim, 1, 1);
    int gsize = xDim*yDim;
    double dg = dx * dy;

    // Setting option for 3d
    if (dimnum == 3){
        grid_tmp.x *= zDim;
        gsize *= zDim;
        dg *= dz;
    }
    dim3 block(grid_tmp.x/threads.x, 1, 1);
    dim3 thread_tmp = threads;
    int pass = 0;

/*
    std::cout << "grid / threads = " << '\t'
              << (double)grid_tmp.x/threads.x << '\n'
              << "grid.x is: " << grid_tmp.x << '\t'
              << "threads.x are: " << threads.x << '\n';
*/

    dim3 grid = par.grid;
    while((double)grid_tmp.x/threads.x > 1.0){
        if(grid_tmp.x == gsize){
            multipass<<<block,threads,threads.x*sizeof(double2)>>>(&gpuWfc[0],
                &gpuParSum[0],pass);
        }
        else{
            multipass<<<block,thread_tmp,thread_tmp.x*sizeof(double2)>>>(
                &gpuParSum[0],&gpuParSum[0],pass);
        }
        grid_tmp.x /= threads.x;
        block = (int) ceil((double)grid_tmp.x/threads.x);
        pass++;
        //std::cout << grid_tmp << '\n';
    }
    thread_tmp = grid_tmp.x;
    multipass<<<1,thread_tmp,thread_tmp.x*sizeof(double2)>>>(&gpuParSum[0],
                                                           &gpuParSum[0], pass);

/*
    // Writing out in the parSum Function (not recommended, for debugging)
    double2 *sum;
    sum = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex)*gsize / threads.x);
    hipMemcpy(sum,gpuParSum,sizeof(hipfftDoubleComplex)*gsize/threads.x,
               hipMemcpyDeviceToHost);
    for (int i = 0; i < gsize/threads.x; i++){
        std::cout << sum[i].x << '\n';
    }
*/
    scalarDiv_wfcNorm<<<grid,threads>>>(gpuWfc, dg, gpuParSum, gpuWfc);
}

/**
** Matches the optical lattice to the vortex lattice.
** Moire super-lattice project.
**/
void optLatSetup(std::shared_ptr<Vtx::Vortex> centre, const double* V,
                 std::vector<std::shared_ptr<Vtx::Vortex>> &vArray, double theta_opt,
                 double intensity, double* v_opt, const double *x, const double *y,
                 Grid &par, Op &opr){
    std::string data_dir = par.sval("data_dir");
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    double dx = par.dval("dx");
    double dy = par.dval("dy");
    double dt = par.dval("dt");
    hipfftDoubleComplex *EV_opt = opr.cufftDoubleComplexval("EV_opt");
    int i,j;
    double sepMin = Tracker::vortSepAvg(vArray,centre);
    sepMin = sepMin*(1 + sepMinEpsilon);
    par.store("Vort_sep",(double)sepMin);

    // Defining the necessary k vectors for the optical lattice


    // Additional /2 as a result of lambda/2 period
    double k_mag = ((2*PI/(sepMin*dx))/2)*(2/sqrt(3));
    double2* k = (double2*) malloc(sizeof(double2)*3);
    par.store("kmag",(double)k_mag);
    k[0].x = k_mag * cos(0*PI/3 + theta_opt);
    k[0].y = k_mag * sin(0*PI/3 + theta_opt);
    k[1].x = k_mag * cos(2*PI/3 + theta_opt);
    k[1].y = k_mag * sin(2*PI/3 + theta_opt);
    k[2].x = k_mag * cos(4*PI/3 + theta_opt);
    k[2].y = k_mag * sin(4*PI/3 + theta_opt);

    double2 *r_opt = (double2*) malloc(sizeof(double2)*xDim);

    //FileIO::writeOut(buffer,data_dir + "r_opt",r_opt,xDim,0);
    par.store("k[0].x",(double)k[0].x);
    par.store("k[0].y",(double)k[0].y);
    par.store("k[1].x",(double)k[1].x);
    par.store("k[1].y",(double)k[1].y);
    par.store("k[2].x",(double)k[2].x);
    par.store("k[2].y",(double)k[2].y);

    // sin(theta_opt)*(sepMin);

    double x_shift = dx*(9+(0.5*xDim-1) - centre->getCoordsD().x);

    // cos(theta_opt)*(sepMin);
    double y_shift = dy*(0+(0.5*yDim-1) - centre->getCoordsD().y);

    printf("Xs=%e\nYs=%e\n",x_shift,y_shift);

    //#pragma omp parallel for private(j)
    for ( j=0; j<yDim; ++j ){
        for ( i=0; i<xDim; ++i ){
            v_opt[j*xDim + i] = intensity*(
                                pow( ( cos( k[0].x*( x[i] + x_shift ) +
                                       k[0].y*( y[j] + y_shift ) ) ), 2) +
                                pow( ( cos( k[1].x*( x[i] + x_shift ) +
                                       k[1].y*( y[j] + y_shift ) ) ), 2) +
                                pow( ( cos( k[2].x*( x[i] + x_shift ) +
                                       k[2].y*( y[j] + y_shift ) ) ), 2)
                                );
            EV_opt[(j*xDim + i)].x=cos( -(V[(j*xDim + i)] +
                                   v_opt[j*xDim + i])*(dt/(2*HBAR)));
            EV_opt[(j*xDim + i)].y=sin( -(V[(j*xDim + i)] +
                                   v_opt[j*xDim + i])*(dt/(2*HBAR)));
        }
    }

    // Storing changed variables
    opr.store("EV_opt", EV_opt);
    opr.store("V", V);
    opr.store("V_opt",v_opt);
}

/**
** Calculates energy and angular momentum of current state.
** Implementation not fully finished.
**/
double energy_angmom(double2 *V_op, double2 *K_op,
                     double2 *gpuWfc, int gState, Grid &par){
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    int zDim = 1;
    double dx = par.dval("dx");
    double dy = par.dval("dy");
    double dz = 1;

    // in case of 3d
    if (par.ival("dimnum") == 3){
        zDim = par.ival("zDim");
        dz = par.dval("dz");
    }
    int gSize = xDim * yDim * zDim;
    double dt = par.dval("dt");
    double gDenConst = par.dval("gDenConst");

    double mass = par.dval("mass");
    double omegaZ = par.dval("omegaZ");

    dim3 threads = par.threads;
    dim3 grid = par.grid;

    // Creating the 2d plan and defining other cuda variables
    hipfftHandle plan;
    hipfftResult result;
    hipError_t err;

    if (par.ival("dimnum") == 2){
        plan = par.ival("plan_2d");
    }
    if (par.ival("dimnum") == 3){
        plan = par.ival("plan_3d");
    }


    double renorm_factor_2d=1.0/pow(gSize,0.5);
    double out;

    // now allocating space on CPU and GPU for energy
    double2 *energy, *energy_gpu, *tmp_wfc;

    energy = (double2*)malloc(sizeof(double2)*gSize);
    //tmp_wfc = (double2*)malloc(sizeof(double2)*gSize);

    hipMalloc((void**) &energy_gpu, sizeof(double2)*gSize);
    hipMalloc((void**) &tmp_wfc, sizeof(double2)*gSize);

    for (int i=0; i < gSize; ++i){
        energy[i].x = 0.0; 
        energy[i].y = 0.0; 
    }

    // Now to memcpy the values over
    hipMemcpy(energy_gpu, energy, sizeof(double2)*gSize,
               hipMemcpyHostToDevice);

    energyCalc<<<grid,threads>>>( tmp_wfc, V_op, 0.5*dt, energy_gpu, gState,1,
                                  0.5*sqrt(omegaZ/mass), gDenConst);
    result = hipfftExecZ2Z( plan, gpuWfc, tmp_wfc, HIPFFT_FORWARD );

    scalarMult<<<grid,threads>>>(tmp_wfc, renorm_factor_2d, tmp_wfc);//Normalise
    energyCalc<<<grid,threads>>>( tmp_wfc, K_op, dt, energy_gpu, gState,0, 
                                  0.5*sqrt(omegaZ/mass), gDenConst);
    result = hipfftExecZ2Z( plan, tmp_wfc, tmp_wfc, HIPFFT_BACKWARD );
    scalarMult<<<grid,threads>>>(tmp_wfc, renorm_factor_2d, tmp_wfc);//Normalise
    
    err=hipMemcpy(energy, energy_gpu, 
                   sizeof(hipfftDoubleComplex)*gSize, 
                   hipMemcpyDeviceToHost);
    if(err!=hipSuccess){
        std::cout << "ERROR: Could not copy energy to host!" << '\n';
        exit(1);
    }
    
    for(int i=0; i<gSize; i++){
        out += energy[i].x + energy[i].y;
        //printf("En=%E\n",result*dx*dy*dz);
    }

    hipFree(energy_gpu);
    hipFree(tmp_wfc);
    free(energy);
    return out*dx*dy*dz;

}
/*

// Creates narrow Gaussian "delta" peaks for vortex kicking
void delta_define(double *x, double *y, double x0, double y0, double *delta,
                  Grid &par, Op &opr){
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    hipfftDoubleComplex *EV_opt = opr.cufftDoubleComplexval("EV_opt");
    double *V = opr.dsval("V");
    double dx = par.dval("dx");
    double dt = par.dval("dt");

    for (int i=0; i<xDim; ++i){
        for (int j=0; j<yDim; ++j){
            delta[j*xDim + i] = 1e6*HBAR*exp( -( pow( x[i] - x0, 2) +
                                pow( y[j] - y0, 2) )/(5*dx*dx) );
            EV_opt[(j*xDim + i)].x = cos( -(V[(j*xDim + i)] +
                                     delta[j*xDim + i])*(dt/(2*HBAR)));
            EV_opt[(j*xDim + i)].y = sin( -(V[(j*xDim + i)] +
                                     delta[j*xDim + i])*(dt/(2*HBAR)));
        }
    }
}
*/
