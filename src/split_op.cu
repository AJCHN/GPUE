#include "hip/hip_runtime.h"
/*
* split_op.cu - GPUE: Split Operator based GPU solver for Nonlinear 
Schrodinger Equation, Copyright (C) 2011-2015, Lee J. O'Riordan 
<loriordan@gmail.com>, Tadhg Morgan, Neil Crowley. All rights reserved.

Redistribution and use in source and binary forms, with or without 
modification, are permitted provided that the following conditions are 
met:

1. Redistributions of source code must retain the above copyright 
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright 
notice, this list of conditions and the following disclaimer in the 
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its 
contributors may be used to endorse or promote products derived from 
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT 
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR 
PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF 
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING 
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS 
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "../include/split_op.h"
#include "../include/kernels.h"
#include "../include/constants.h"
#include "../include/fileIO.h"
#include "../include/tracker.h"
#include "../include/minions.h"
#include "../include/ds.h"

#include "../include/lattice.h"
#include "../include/node.h"
#include "../include/edge.h"
#include "../include/manip.h"
#include "../include/vort.h"
#include <iostream>

unsigned int LatticeGraph::Edge::suid = 0;
unsigned int LatticeGraph::Node::suid = 0;

char buffer[100];
int verbose; //Print more info. Not curently implemented.
int device; //GPU ID choice.
int kick_it; //Kicking mode: 0 = off, 1 = multiple, 2 = single
int graph=0; //Generate graph from vortex lattice.
double gammaY; //Aspect ratio of trapping geometry.
double omega; //Rotation rate of condensate
double timeTotal;
double angle_sweep; //Rotation angle of condensate relative to x-axis
Params *paramS;
Array params;
double x0_shift, y0_shift; //Optical lattice shift parameters.
double Rxy; //Condensate scaling factor.
double a0x, a0y; //Harmonic oscillator length in x and y directions
double sepMinEpsilon=0.0; //Minimum separation for epsilon.

/*
 * Checks CUDA routines have exitted correctly.
 */
int isError(int result, char* c){
	if(result!=0){printf("Error has occurred for method %s with return type %d\n",c,result);
		exit(result);
	}
	return result;
}
int initialise(double omegaX, double omegaY, int N){
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
	unsigned int xD=1,yD=1,zD=1;
	threads = 128;
	unsigned int b = xDim*yDim/threads;  //number of blocks in simulation
	unsigned long long maxElements = 65536*65536ULL; //largest number of elements

	if( b < (1<<16) ){
		xD = b;
	}
	else if( (b >= (1<<16) ) && (b <= (maxElements)) ){
		int t1 = log(b)/log(2);
		float t2 = (float) t1/2;
		t1 = (int) t2;
		if(t2 > (float) t1){
			xD <<= t1;
			yD <<= (t1 + 1);
		}
		else if(t2 == (float) t1){
			xD <<= t1;
			yD <<= t1;
		}
	}
	else{
		printf("Outside range of supported indexing");
		exit(-1);
	}
	printf("Compute grid dimensions chosen as X=%d	Y=%d\n",xD,yD);
	
	grid.x=xD; 
	grid.y=yD; 
	grid.z=zD; 
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
	
	unsigned int i,j; //Used in for-loops for indexing
	
	unsigned int gSize = xDim*yDim;
	double xOffset, yOffset;
	xOffset=0.0;//5.0e-6;
	yOffset=0.0;//5.0e-6;
	
	mass = 1.0; //Rb 87 mass, kg
	appendData(&params,"Mass",mass);
	a_s = 0.0;//4.67e-9;
	appendData(&params,"a_s",a_s);

	double sum = 0.0;

	a0x = sqrt(1.0/2.0);
	a0y = sqrt(1.0/2.0);
	appendData(&params,"a0x",a0x);
	appendData(&params,"a0y",a0y);
	
	xMax = 10*a0x;//10*bec_length;//6*Rxy*a0x;
	yMax = 10*a0y;//10*bec_length;//
	appendData(&params,"xMax",xMax);
	appendData(&params,"yMax",yMax);

	double pxMax, pyMax;
	pxMax = (PI/xMax)*(xDim>>1);
	pyMax = (PI/yMax)*(yDim>>1);
	appendData(&params,"pyMax",pyMax);
	appendData(&params,"pxMax",pxMax);
	
	dx = xMax/(xDim>>1);
	dy = yMax/(yDim>>1);
	appendData(&params,"dx",dx);
	appendData(&params,"dy",dy);
	
	double dpx, dpy;
	dpx = PI/(xMax);
	dpy = PI/(yMax);
	appendData(&params,"dpx",dpx);
	appendData(&params,"dpy",dpy);

	//printf("a0x=%e  a0y=%e \n dx=%e   dx=%e\n R_xy=%e\n",a0x,a0y,dx,dy,Rxy);
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
	
	//double *x,*y,*xp,*yp;
	x = (double *) malloc(sizeof(double) * xDim);
	y = (double *) malloc(sizeof(double) * yDim);
	xp = (double *) malloc(sizeof(double) * xDim);
	yp = (double *) malloc(sizeof(double) * yDim);

	/*
	 * R-space and K-space grids
	 */
	for(i=0; i<xDim/2; ++i){
		x[i] = -xMax + (i)*dx;		
		x[i + (xDim/2)] = (i)*dx;
		
		y[i] = -yMax + (i)*dy;		
		y[i + (yDim/2)] = (i)*dy;
		
		xp[i] = (i)*dpx;
		xp[i + (xDim/2)] = -pxMax + (i)*dpx;
		
		yp[i] = (i)*dpy;
		yp[i + (yDim/2)] = -pyMax + (i)*dpy;
	}

/*
	 % maximum values
	  pxmax=pi*Ngx/(2*xmax);
	   % spacing in position and momentum space
	    dx=2*xmax/Ngx;   dpx=2*pxmax/Ngx;
	     % grid vectors, position and momentum space
	      x=(1:Ngx)'*dx-xmax;   pxn=((1:Ngx)*dpx-pxmax)';   
	       % reordination needed for the fourier transform
	        px(Ngx/2+2:Ngx,1)=pxn(1:Ngx/2-1,1);   px(1:Ngx/2+1,1)=pxn(Ngx/2:Ngx,1);
*/	
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
	
	/* Initialise wavefunction, momentum, position, angular momentum, imaginary and real-time evolution operators . */
	Energy = (double*) malloc(sizeof(double) * gSize);
	r = (double *) malloc(sizeof(double) * gSize);
	Phi = (double *) malloc(sizeof(double) * gSize);
	wfc = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
	wfc_backup = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * (gSize/threads));
	K = (double *) malloc(sizeof(double) * gSize);
	V = (double *) malloc(sizeof(double) * gSize);
	V_opt = (double *) malloc(sizeof(double) * gSize);
	GK = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
	GV = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
	EK = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
	EV = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
	EV_opt = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
	xPy = (double *) malloc(sizeof(double) * gSize);
	yPx = (double *) malloc(sizeof(double) * gSize);
	ExPy = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
	EyPx = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
	EappliedField = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
	
	/* Initialise wfc, EKp, and EVr buffers on GPU */
	hipMalloc((void**) &Energy_gpu, sizeof(double) * gSize);
	hipMalloc((void**) &wfc_gpu, sizeof(hipfftDoubleComplex) * gSize);
	hipMalloc((void**) &Phi_gpu, sizeof(double) * gSize);
	hipMalloc((void**) &K_gpu, sizeof(hipfftDoubleComplex) * gSize);
	hipMalloc((void**) &V_gpu, sizeof(hipfftDoubleComplex) * gSize);
	hipMalloc((void**) &xPy_gpu, sizeof(hipfftDoubleComplex) * gSize);
	hipMalloc((void**) &yPx_gpu, sizeof(hipfftDoubleComplex) * gSize);
	hipMalloc((void**) &par_sum, sizeof(hipfftDoubleComplex) * (gSize/threads));
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

	#ifdef __linux
	int cores = omp_get_num_procs();
	appendData(&params,"Cores_Total",cores);
	appendData(&params,"Cores_Max",cores/2); //Assuming dev system specifics (Xeon with HT -> cores detected / 2)
	omp_set_num_threads(cores/2);
	#pragma omp parallel for private(j)
	#endif
	for( i=0; i < xDim; i++ ){
		for( j=0; j < yDim; j++ ){
			Phi[(i*yDim + j)] = fmod(l*atan2(y[j], x[i]),2*PI);
			
			wfc[(i*yDim + j)].x =  (1/sqrt(2))*pow(1/PI,0.5) * exp( -0.5*( x[i]*x[i] + y[j]*y[j] ) )*(1+2*x[i]/sqrt(2));
			wfc[(i*yDim + j)].y = 0.0;
				
			V[(i*yDim + j)] = 0.5*( x[i]*x[i] + y[j]*y[j] );
			K[(i*yDim + j)] = 0.5*( xp[i]*xp[i] + yp[j]*yp[j] );

			GV[(i*yDim + j)].x = exp( -V[(i*xDim + j)]*(gdt/(2)));
			GK[(i*yDim + j)].x = exp( -K[(i*xDim + j)]*(gdt/1));
			GV[(i*yDim + j)].y = 0.0;
			GK[(i*yDim + j)].y = 0.0;
			
			xPy[(i*yDim + j)] = x[i]*yp[j];
			yPx[(i*yDim + j)] = -y[j]*xp[i];
			
			EV[(i*yDim + j)].x=cos( -V[(i*xDim + j)]*(dt/(2)));
			EV[(i*yDim + j)].y=sin( -V[(i*xDim + j)]*(dt/(2)));
			EK[(i*yDim + j)].x=cos( -K[(i*xDim + j)]*(dt/1));
			EK[(i*yDim + j)].y=sin( -K[(i*xDim + j)]*(dt/1));
			
			ExPy[(i*yDim + j)].x=cos(-omega*omegaX*xPy[(i*xDim + j)]*dt);
			ExPy[(i*yDim + j)].y=sin(-omega*omegaX*xPy[(i*xDim + j)]*dt);
			EyPx[(i*yDim + j)].x=cos(-omega*omegaX*yPx[(i*xDim + j)]*dt);
			EyPx[(i*yDim + j)].y=sin(-omega*omegaX*yPx[(i*xDim + j)]*dt);
	
			sum += (wfc[(i*xDim + j)].x*wfc[(i*xDim + j)].x + wfc[(i*xDim + j)].y*wfc[(i*xDim + j)].y);
		}
	}
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
	//hdfWriteDouble(xDim, V, 0, "V_0"); //HDF not required for current projects. Removed.
	//hdfWriteComplex(xDim, wfc, 0, "wfc_0");
	FileIO::writeOutDouble(buffer,"V",V,xDim*yDim,0);
	//FileIO::writeOutDouble(buffer,"V_opt",V_opt,xDim*yDim,0);
	FileIO::writeOutDouble(buffer,"K",K,xDim*yDim,0);
	FileIO::writeOutDouble(buffer,"xPy",xPy,xDim*yDim,0);
	FileIO::writeOutDouble(buffer,"yPx",yPx,xDim*yDim,0);
	FileIO::writeOut(buffer,"WFC",wfc,xDim*yDim,0);
	FileIO::writeOut(buffer,"ExPy",ExPy,xDim*yDim,0);
	FileIO::writeOut(buffer,"EyPx",EyPx,xDim*yDim,0);
	FileIO::writeOutDouble(buffer,"Phi",Phi,xDim*yDim,0);
	FileIO::writeOutDouble(buffer,"r",r,xDim*yDim,0);
	FileIO::writeOutDouble(buffer,"x",x,xDim,0);
	FileIO::writeOutDouble(buffer,"y",y,yDim,0);
	FileIO::writeOutDouble(buffer,"px",xp,xDim,0);
	FileIO::writeOutDouble(buffer,"py",yp,yDim,0);
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

	//free(V); 
	free(K); free(r); //free(Phi);

	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
	if (gsteps>0.0){
		sum=sqrt(sum*dx*dy);
		//#pragma omp parallel for reduction(+:sum) private(j)
		for (i = 0; i < xDim; i++){
			for (j = 0; j < yDim; j++){
				wfc[(i*yDim + j)].x = (wfc[(i*yDim + j)].x)/(sum);
				wfc[(i*yDim + j)].y = (wfc[(i*yDim + j)].y)/(sum);
			}
		}
	}
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
	
	result = hipfftPlan2d(&plan_2d, xDim, yDim, HIPFFT_Z2Z);
	if(result != HIPFFT_SUCCESS){
		printf("Result:=%d\n",result);
		printf("Error: Could not execute hipfftPlan2d(%s ,%d, %d).\n", "plan_2d", (unsigned int)xDim, (unsigned int)yDim);
		return -1;
	}

	result = hipfftPlan1d(&plan_1d, xDim, HIPFFT_Z2Z, yDim);
	if(result != HIPFFT_SUCCESS){
		printf("Result:=%d\n",result);
		printf("Error: Could not execute hipfftPlan3d(%s ,%d ,%d ).\n", "plan_1d", (unsigned int)xDim, (unsigned int)yDim);
		return -1;
	}
	
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
	
	return 0;
}

int evolve( hipfftDoubleComplex *gpuWfc, 
			hipfftDoubleComplex *gpuMomentumOp,
			hipfftDoubleComplex *gpuPositionOp,
			void *gpu1dyPx,
			void *gpu1dxPy,
			hipfftDoubleComplex *gpuParSum,			 
			int gridSize, int numSteps, int threads, 
			unsigned int gstate, int lz, int nonlin, int printSteps, int N, unsigned int ramp){

	//Because no two operations are created equally. Multiplimultiplication is faster than divisions.
	double renorm_factor_2d=1.0/pow(gridSize,0.5);
	double renorm_factor_1d=1.0/pow(xDim,0.5);

	clock_t begin, end;
	double time_spent;
	double Dt;
	if(gstate==0){
		Dt = gdt;
		printf("Timestep for groundstate solver set as: %E\n",Dt);
	}
	else{
		Dt = dt;
		printf("Timestep for evolution set as: %E\n",Dt);
	}
	begin = clock();
	double omega_0=omega*omegaX;

	#if 0 
	/** Determines the initial average density at the condensate central position and calculates a value for the healing length from this. Used thereafter as the lower limit for distances between vortices. **/
	int gridSum = 1<<6;
	double *densitySubset = (double*) malloc(sizeof(double)*gridSum);
	#pragma omp parallel for private(k)
	for (int j=0; j<gridSum; ++j){
		for (int k=0; k<gridSum; ++k){
			densitySubset[j*gridSum + k] = Minions::psi2(wfc[ ( (yDim/2) - (gridSum/2) + j )*yDim  + ( (xDim/2)  - (gridSum/2) + k )]);
		}
	}
	xi = 1/sqrt(8*PI*a_s*Minions::sumAvg(densitySubset,gridSum)/(dx*dy));//defined central condensate density
	printf("Avg healing length at centre=%E\n",xi);
	#endif

	/** ** ####################################################################################################### ** **/
	/** **					        HERE BE DRAGONS OF THE MOST DANGEROUS KIND!			                           ** **/
	/** ** ####################################################################################################### ** **/

	//Double buffering and will attempt to thread free and calloc operations to hide time penalty. Or may not bother.
	int num_vortices[2] = {0,0};
	int num_latt_max = 0;
	int* vortexLocation; //binary matrix of size xDim*yDim, 1 for vortex at specified index, 0 otherwise
	int* olMaxLocation = (int*) calloc(xDim*yDim,sizeof(int));

	struct Vtx::Vortex central_vortex; //vortex closest to the central position
	double vort_angle; //Angle of vortex lattice. Add to optical lattice for alignment.
	struct Vtx::Vortex *vortCoords = NULL; //array of vortex coordinates from vortexLocation 1's

	struct Vtx::Vortex *vortCoordsP = NULL; //Previous array of vortex coordinates from vortexLocation 1's

	int2 *olCoords = NULL; //array of vortex coordinates from vortexLocation 1's
	int2 *vortDelta = NULL;

	LatticeGraph::Lattice lattice; //Vortex lattice graph.
	double* adjMat;
	
	double vortOLSigma=0.0;
	double sepAvg = 0.0;
	
	int num_kick = 0;
	double t_kick = (2*PI/omega_0)/(6*Dt);
	
	for(int i=0; i < numSteps; ++i){
		if ( ramp == 1 ){
			omega_0=omegaX*((omega-0.39)*((double)i/(double)(numSteps)) + 0.39); //Adjusts omega for the appropriate trap frequency.
		}
		if(i % printSteps == 0) { //Print-out at pre-determined rate. Vortex & wfc analysis performed here also.
			printf("Step: %d	Omega: %lf\n", i, omega_0 / omegaX);
			hipMemcpy(wfc, gpuWfc, sizeof(hipfftDoubleComplex) * xDim * yDim, hipMemcpyDeviceToHost);
			end = clock();
			time_spent = (double) (end - begin) / CLOCKS_PER_SEC;
			printf("Time spent: %lf\n", time_spent);
			char *fileName = "";
			printf("ramp=%d		gstate=%d	rg=%d		\n", ramp, gstate, ramp | (gstate << 1));
			switch (ramp | (gstate << 1)) {
				case 0: //Groundstate solver, constant Omega value.
					fileName = "wfc_0_const";
			        break;
				case 1: //Groundstate solver, ramped Omega value.
					fileName = "wfc_0_ramp";
			        break;
				case 2: //Real-time evolution, constant Omega value.
					fileName = "wfc_ev";
					if(gpe==0){
						break;
					}
			        vortexLocation = (int *) calloc(xDim * yDim, sizeof(int));
			        num_vortices[0] = Tracker::findVortex(vortexLocation, wfc, 1e-4, xDim, x, i);

			        if (i == 0) { //If initial step, locate vortices, least-squares to find exact centre, calculate lattice angle, generate optical lattice.
				        vortCoords = (struct Vtx::Vortex *) malloc(
						        sizeof(struct Vtx::Vortex) * (2 * num_vortices[0]));
				        vortCoordsP = (struct Vtx::Vortex *) malloc(
						        sizeof(struct Vtx::Vortex) * (2 * num_vortices[0]));
				        Tracker::vortPos(vortexLocation, vortCoords, xDim, wfc);
				        Tracker::lsFit(vortCoords, wfc, num_vortices[0], xDim);
				        central_vortex = Tracker::vortCentre(vortCoords, num_vortices[0], xDim);
				        vort_angle = Tracker::vortAngle(vortCoords, central_vortex, num_vortices[0]);
				        appendData(&params, "Vort_angle", vort_angle);
				        optLatSetup(central_vortex, V, vortCoords, num_vortices[0],
				                    vort_angle + PI * angle_sweep / 180.0, laser_power * HBAR * sqrt(omegaX * omegaY),
				                    V_opt, x, y);
				        sepAvg = Tracker::vortSepAvg(vortCoords, central_vortex, num_vortices[0]);
				        if (kick_it == 2) {
					        printf("Kicked it 1\n");
					        hipMemcpy(V_gpu, EV_opt, sizeof(hipfftDoubleComplex) * xDim * yDim, hipMemcpyHostToDevice);
				        }
				        FileIO::writeOutDouble(buffer, "V_opt_1", V_opt, xDim * yDim, 0);
				        FileIO::writeOut(buffer, "EV_opt_1", EV_opt, xDim * yDim, 0);
				        appendData(&params, "Central_vort_x", (double) central_vortex.coords.x);
				        appendData(&params, "Central_vort_y", (double) central_vortex.coords.y);
				        appendData(&params, "Central_vort_winding", (double) central_vortex.wind);
				        appendData(&params, "Num_vort", (double) num_vortices[0]);
				        FileIO::writeOutParam(buffer, params, "Params.dat");
			        }
			        else if (num_vortices[0] > num_vortices[1]) {
				        printf("Number of vortices increased from %d to %d\n", num_vortices[1], num_vortices[0]);
				        Tracker::vortPos(vortexLocation, vortCoords, xDim, wfc);
				        Tracker::lsFit(vortCoords, wfc, num_vortices[0], xDim);
			        }
			        else {
				        Tracker::vortPos(vortexLocation, vortCoords, xDim, wfc);
				        Tracker::lsFit(vortCoords, wfc, num_vortices[0], xDim);
				        Tracker::vortArrange(vortCoords, vortCoordsP, num_vortices[0]);
			        }

			        if (graph == 1) {

				        for (unsigned int ii = 0; ii < num_vortices[0]; ++ii) {
					        std::shared_ptr<LatticeGraph::Node> n(new LatticeGraph::Node(vortCoords[ii]));
					        lattice.addVortex(std::move(n));
				        }
				        unsigned int *uids = (unsigned int *) malloc(
						        sizeof(unsigned int) * lattice.getVortices().size());
				        for (int a = 0; a < lattice.getVortices().size(); ++a) {
					        uids[a] = lattice.getVortexIdx(a)->getUid();
				        }
				        if(i==0) {
					        //Lambda for vortex annihilation/creation.
					        auto killIt=[&](int idx) {
					            WFC::phaseWinding(Phi, 1, x, y, dx, dy, lattice.getVortexUid(idx)->getData().coordsD.x,
					                          lattice.getVortexUid(idx)->getData().coordsD.y, xDim);
					            hipMemcpy(Phi_gpu, Phi, sizeof(double) * xDim * yDim, hipMemcpyHostToDevice);
					            cMultPhi <<<grid, threads>>> (gpuWfc, Phi_gpu, gpuWfc);
				            };
					        //killIt(44); //Kills vortex with UID 44


				        }
				        lattice.createEdges(1.5 * 2e-5 / dx);
				        adjMat = (double *) calloc(lattice.getVortices().size() * lattice.getVortices().size(),
				                                   sizeof(double));
				        lattice.genAdjMat(adjMat);
				        FileIO::writeOutAdjMat(buffer, "graph", adjMat, uids, lattice.getVortices().size(), i);
				        free(adjMat);
				        free(uids);
				        lattice.getVortices().clear();
				        lattice.getEdges().clear();
				        //exit(0);
			        }

			        FileIO::writeOutVortex(buffer, "vort_arr", vortCoords, num_vortices[0], i);
			        printf("Located %d vortices\n", num_vortices[0]);
			        printf("Sigma=%e\n", vortOLSigma);
			        free(vortexLocation);
			        num_vortices[1] = num_vortices[0];
			        memcpy(vortCoordsP, vortCoords, sizeof(int2) * num_vortices[0]);
			        //exit(1);
			        break;
				case 3:
					fileName = "wfc_ev_ramp";
			        break;
				default:
					break;
			}
			if (write_it) {
				FileIO::writeOut(buffer, fileName, wfc, xDim * yDim, i);
			}
			//printf("Energy[t@%d]=%E\n",i,energy_angmom(gpuPositionOp, gpuMomentumOp, dx, dy, gpuWfc,gstate));
/*			hipMemcpy(V_gpu, V, sizeof(double)*xDim*yDim, hipMemcpyHostToDevice);
			hipMemcpy(K_gpu, K, sizeof(double)*xDim*yDim, hipMemcpyHostToDevice);
			hipMemcpy(V_gpu, , sizeof(double)*xDim*yDim, hipMemcpyHostToDevice);
			hipMemcpy(K_gpu, K, sizeof(double)*xDim*yDim, hipMemcpyHostToDevice);
*/		}
	
	/** ** ####################################################################################################### ** **/
	/** ** ####################################################################################################### ** **/
	/** ** 							More F'n' Dragons!				       ** **/
	/** ** ####################################################################################################### ** **/
		if(i % ((int)t_kick+1) == 0 && num_kick<=6 && gstate==1 && kick_it == 1 ){
			hipMemcpy(V_gpu, EV_opt, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
			++num_kick;
		}
	/** ** ####################################################################################################### ** **/

		/*
		 * U_r(dt/2)*wfc
		 */ 
		if(nonlin == 1){
			cMultDensity<<<grid,threads>>>(gpuPositionOp,gpuWfc,gpuWfc,0.5*Dt,mass,omegaZ,gstate,N*interaction);
		}
		else {
			cMult<<<grid,threads>>>(gpuPositionOp,gpuWfc,gpuWfc);
		}
				
		/*
		 * U_p(dt)*fft2(wfc)
		 */		
		result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD);
		scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc); //Normalise
		cMult<<<grid,threads>>>(gpuMomentumOp,gpuWfc,gpuWfc);
		result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
		scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc); //Normalise
		
		/*
		 * U_r(dt/2)*wfc
		 */	
		if(nonlin == 1){
			cMultDensity<<<grid,threads>>>(gpuPositionOp,gpuWfc,gpuWfc,Dt*0.5,mass,omegaZ,gstate,N*interaction);
		}
		else {
			cMult<<<grid,threads>>>(gpuPositionOp,gpuWfc,gpuWfc);
		}
		if( kick_it !=0 && ( (i % (int)(t_kick+1) == 0 && num_kick<=6 && gstate==1) || (kick_it >= 1 && i==0)) ){
			hipMemcpy(V_gpu, EV, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
			printf("Got here: Cuda memcpy EV into GPU\n");
		}
		/**************************************************************/
		/* Angular momentum xPy-yPx   */
		if(lz == 1){
			switch(i%2 | (gstate<<1)){
				case 0: //Groundstate solver, even step
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); // wfc_xPy
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				angularOp<<<grid,threads>>>(omega_0, Dt, gpuWfc, (double*) gpu1dxPy, gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
			
				result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD); //2D forward
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); //1D inverse to wfc_yPx
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				angularOp<<<grid,threads>>>(omega_0, Dt, gpuWfc, (double*) gpu1dyPx, gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); // wfc_PxPy
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); //2D Inverse
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
				break;
				
				case 1:	//Groundstate solver, odd step
				result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD); //2D forward
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); //1D inverse to wfc_yPx
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				angularOp<<<grid,threads>>>(omega_0, Dt, gpuWfc, (double*) gpu1dyPx, gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); // wfc_PxPy
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); //2D Inverse
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
				
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); // wfc_xPy
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				angularOp<<<grid,threads>>>(omega_0, Dt, gpuWfc, (double*) gpu1dxPy, gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				break;
				
				case 2: //Real time evolution, even step
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); // wfc_xPy
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				cMult<<<grid,threads>>>(gpuWfc, (hipfftDoubleComplex*) gpu1dxPy, gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
			
				result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD); //2D forward
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); //1D inverse to wfc_yPx
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				cMult<<<grid,threads>>>(gpuWfc, (hipfftDoubleComplex*) gpu1dyPx, gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); // wfc_PxPy
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); //2D Inverse
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
				break;
				
				case 3:	//Real time evolution, odd step
				result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD); //2D forward
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); //1D inverse to wfc_yPx
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				cMult<<<grid,threads>>>(gpuWfc, (hipfftDoubleComplex*) gpu1dyPx, gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); // wfc_PxPy
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); //2D Inverse
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
				
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); // wfc_xPy
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				cMult<<<grid,threads>>>(gpuWfc, (hipfftDoubleComplex*) gpu1dxPy, gpuWfc);
				result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
				scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
				break;
			
			}
		}
		/**************************************************************/
	
		if(gstate==0){
			parSum(gpuWfc, gpuParSum, xDim, yDim, threads);
		}
	}
	return 0;
}

/*
 * Used to perform parallel summation on WFC for normalisation.
 */
void parSum(double2* gpuWfc, double2* gpuParSum, int xDim, int yDim, int threads){
		int grid_tmp = xDim*yDim;
		int block = grid_tmp/threads;
		int thread_tmp = threads;
		int pass = 0;
		while((double)grid_tmp/threads > 1.0){
			if(grid_tmp == xDim*yDim){
				multipass<<<block,threads,threads*sizeof(double2)>>>(&gpuWfc[0],&gpuParSum[0],pass); 
			}
			else{
				multipass<<<block,thread_tmp,thread_tmp*sizeof(double2)>>>(&gpuParSum[0],&gpuParSum[0],pass);
			}
			grid_tmp /= threads;
			block = (int) ceil((double)grid_tmp/threads);
			pass++;
		}
		thread_tmp = grid_tmp;
		multipass<<<1,thread_tmp,thread_tmp*sizeof(double2)>>>(&gpuParSum[0],&gpuParSum[0], pass);
		scalarDiv_wfcNorm<<<grid,threads>>>(gpuWfc, dx*dy, gpuParSum, gpuWfc);
}

/**
** Matches the optical lattice to the vortex lattice. Moire super-lattice project.
**/
void optLatSetup(struct Vtx::Vortex centre, double* V, struct Vtx::Vortex *vArray, int num_vortices, double theta_opt, double intensity, double* v_opt, double *x, double *y){
	int i,j;
	double sepMin = Tracker::vortSepAvg(vArray,centre,num_vortices);
	sepMin = sepMin*(1 + sepMinEpsilon);
	appendData(&params,"Vort_sep",(double)sepMin);
	/*
	* Defining the necessary k vectors for the optical lattice
	*/
	double k_mag = ((2*PI/(sepMin*dx))/2)*(2/sqrt(3)); // Additional /2 as a result of lambda/2 period
	double2* k = (double2*) malloc(sizeof(double2)*3);
	appendData(&params,"kmag",(double)k_mag);
	k[0].x = k_mag * cos(0*PI/3 + theta_opt);
	k[0].y = k_mag * sin(0*PI/3 + theta_opt);
	k[1].x = k_mag * cos(2*PI/3 + theta_opt);
	k[1].y = k_mag * sin(2*PI/3 + theta_opt);
	k[2].x = k_mag * cos(4*PI/3 + theta_opt);
	k[2].y = k_mag * sin(4*PI/3 + theta_opt);
	
	double2 *r_opt = (double2*) malloc(sizeof(double2)*xDim);

/*	for (int ii = 0; ii < xDim; ++ii){
		r_opt[ii].x = 0.0 + (xDim/sepMin)*PI*(ii-centre.coords.x)/(xDim-1);
		r_opt[ii].y = 0.0 + (xDim/sepMin)*PI*(ii-centre.coords.y)/(yDim-1);
	}
*/
	FileIO::writeOut(buffer,"r_opt",r_opt,xDim,0);
	appendData(&params,"k[0].x",(double)k[0].x);
	appendData(&params,"k[0].y",(double)k[0].y);
	appendData(&params,"k[1].x",(double)k[1].x);
	appendData(&params,"k[1].y",(double)k[1].y);
	appendData(&params,"k[2].x",(double)k[2].x);
	appendData(&params,"k[2].y",(double)k[2].y);

	double x_shift = dx*(9+(0.5*xDim-1) - centre.coords.x);//sin(theta_opt)*(sepMin);
	double y_shift = dy*(0+(0.5*yDim-1) - centre.coords.y);//cos(theta_opt)*(sepMin);

	printf("Xs=%e\nYs=%e\n",x_shift,y_shift);

	//#pragma omp parallel for private(j)
	for ( j=0; j<yDim; ++j ){
		for ( i=0; i<xDim; ++i ){
			v_opt[j*xDim + i] = intensity*(
					    	  pow( ( cos( k[0].x*( x[i] + x_shift ) + k[0].y*( y[j] + y_shift ) ) ), 2)
					  	+ pow( ( cos( k[1].x*( x[i] + x_shift ) + k[1].y*( y[j] + y_shift ) ) ), 2)
					  	+ pow( ( cos( k[2].x*( x[i] + x_shift ) + k[2].y*( y[j] + y_shift ) ) ), 2)
			/*		    	  pow( abs( cos( k[0].x*( r_opt[i].x + x_shift ) + k[0].y*( r_opt[j].y + y_shift ) ) ), 2)
					  	+ pow( abs( cos( k[1].x*( r_opt[i].x + x_shift ) + k[1].y*( r_opt[j].y + y_shift ) ) ), 2)
					  	+ pow( abs( cos( k[2].x*( r_opt[i].x + x_shift ) + k[2].y*( r_opt[j].y + y_shift ) ) ), 2)
			*/		    );
			EV_opt[(j*xDim + i)].x=cos( -(V[(j*xDim + i)] + v_opt[j*xDim + i])*(dt/(2*HBAR)));
			EV_opt[(j*xDim + i)].y=sin( -(V[(j*xDim + i)] + v_opt[j*xDim + i])*(dt/(2*HBAR)));
		}
	}
}

/**
** Calculates energy and angular momentum of current state. Implementation not fully finished.
**/
double energy_angmom(double *Energy, double* Energy_gpu, double2 *V_op, double2 *K_op, double dx, double dy, double2 *gpuWfc, int gState){
	double renorm_factor_2d=1.0/pow(xDim*yDim,0.5);
	double result=0;

	for (int i=0; i < xDim*yDim; ++i){
		Energy[i] = 0.0; 
	}
	
	
/*	hipMalloc((void**) &energy_gpu, sizeof(double2) * xDim*yDim);

	energyCalc<<<grid,threads>>>( gpuWfc, V_op, 0.5*dt, energy_gpu, gState,1,i 0.5*sqrt(omegaZ/mass));
	result = hipfftExecZ2Z( plan_2d, gpuWfc, gpuWfc, HIPFFT_FORWARD );
	scalarDiv<<<grid,threads>>>( gpuWfc, renorm_factor_2d, gpuWfc ); //Normalise

	energyCalc<<<grid,threads>>>( gpuWfc, K_op, dt, energy_gpu, gState,0, 0.5*sqrt(omegaZ/mass));
	result = hipfftExecZ2Z( plan_2d, gpuWfc, gpuWfc, HIPFFT_BACKWARD );
	scalarDiv<<<grid,threads>>>( gpuWfc, renorm_factor_2d, gpuWfc ); //Normalise
	
	err=hipMemcpy(energy, energy_gpu, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyDeviceToHost);
	
	for(int i=0; i<xDim*yDim; i++){
		result += energy[i].x;
		//printf("En=%E\n",result*dx*dy);
	}
	return result*dx*dy;
*/
	
}


//###################################################################################################################
//###################################################################################################################

/*
 * Used to perform parallel summation using templates from c++
 */
template<typename T> void parSum(T *gpuToSumArr, T *gpuParSum, int xDim, int yDim, int threads){
                int grid_tmp = xDim*yDim;
                int block = grid_tmp/threads;
                int thread_tmp = threads;
                int pass = 0;
                while((double)grid_tmp/threads > 1.0){
                        if(grid_tmp == xDim*yDim){
                                multipass<<<block,threads,threads*sizeof(T)>>>(&gpuToSumArr[0],&gpuParSum[0],pass);
                        }
                        else{
                                multipass<<<block,thread_tmp,thread_tmp*sizeof(T)>>>(&gpuParSum[0],&gpuParSum[0],pass);
                        }
                        grid_tmp /= threads;
                        block = (int) ceil((double)grid_tmp/threads);
                        pass++;
                }
                thread_tmp = grid_tmp;
                multipass<<<1,thread_tmp,thread_tmp*sizeof(double2)>>>(&gpuParSum[0],&gpuParSum[0], pass);
                scalarDiv_wfcNorm<<<grid,threads>>>(gpuToSumArr, dx*dy, gpuParSum, gpuToSumArr);
}
//###################################################################################################################
//###################################################################################################################
int parseArgs(int argc, char** argv){
	int opt;
	while ((opt = getopt (argc, argv, "d:x:y:w:G:g:e:T:t:n:p:r:o:L:l:s:i:P:X:Y:O:k:W:U:V:S:a:")) != -1) {
		switch (opt)
		{
			case 'x':
				xDim = atoi(optarg);
				printf("Argument for x is given as %d\n",xDim);
				appendData(&params,"xDim",(double)xDim);
				break;
			case 'y':
				yDim = atoi(optarg);
				printf("Argument for y is given as %d\n",yDim);
				appendData(&params,"yDim",(double)yDim);
				break;
			case 'w':
				omega = atof(optarg);
				printf("Argument for OmegaRotate is given as %E\n",omega);
				appendData(&params,"omega",omega);
				break;
			case 'G':
				gammaY = atof(optarg);
				printf("Argument for gamma is given as %E\n",gammaY);
				appendData(&params,"gammaY",gammaY);
				break;
			case 'g':
				gsteps = atof(optarg);
				printf("Argument for Groundsteps is given as %ld\n",gsteps);
				appendData(&params,"gsteps",gsteps);
				break;
			case 'e':
				esteps = atof(optarg);
				printf("Argument for EvSteps is given as %ld\n",esteps);
				appendData(&params,"esteps",esteps);
				break;
			case 'T':
				gdt = atof(optarg);
				printf("Argument for groundstate Timestep is given as %E\n",gdt);
				appendData(&params,"gdt",gdt);
				break;
			case 't':
				dt = atof(optarg);
				printf("Argument for Timestep is given as %E\n",dt);
				appendData(&params,"dt",dt);
				break;
			case 'd':
				device = atoi(optarg);
				printf("Argument for device is given as %d\n",device);
				appendData(&params,"device",device);
				break;
			case 'n':
				atoms = atof(optarg);
				printf("Argument for atoms is given as %ld\n",atoms);
				appendData(&params,"atoms",atoms);
				break;
			case 'r':
				read_wfc  = atoi(optarg);
				printf("Argument for ReadIn is given as %d\n",read_wfc);
				appendData(&params,"read_wfc",(double)read_wfc);
				break;
			case 'p':
				print = atoi(optarg);
				printf("Argument for Printout is given as %d\n",print);
				appendData(&params,"print_out",(double)print);
				break;
			case 'L':
				l = atof(optarg);
				printf("Vortex winding is given as : %E\n",l);
				appendData(&params,"winding",l);
				break;
			case 'l':
				ang_mom = atoi(optarg);
				printf("Angular Momentum mode engaged: %d\n",ang_mom);
				appendData(&params,"corotating",(double)ang_mom);
				break;
			case 's':
				gpe = atoi(optarg);
				printf("Non-linear mode engaged: %d\n",gpe);
				appendData(&params,"gpe",gpe);
				break;
			case 'o':
				omegaZ = atof(optarg);
				printf("Argument for OmegaZ is given as %E\n",omegaZ);
				appendData(&params,"omegaZ",omegaZ);
				break;
			case 'i':
				interaction = atof(optarg);
				printf("Argument for interaction scaling is %E\n",interaction);
				appendData(&params,"int_scaling",interaction);
				break;
			case 'P':
				laser_power = atof(optarg);
				printf("Argument for laser power is %E\n",laser_power);
				appendData(&params,"laser_power",laser_power);
				break;
			case 'X':
				omegaX = atof(optarg);
				printf("Argument for omegaX is %E\n",omegaX);
				appendData(&params,"omegaX",omegaX);
				break;
			case 'Y':
				omegaY = atof(optarg);
				printf("Argument for omegaY is %E\n",omegaY);
				appendData(&params,"omegaY",omegaY);
				break;
			case 'O':
				angle_sweep = atof(optarg);
				printf("Argument for angle_sweep is %E\n",angle_sweep);
				appendData(&params,"angle_sweep",angle_sweep);
				break;
			case 'k':
				kick_it = atoi(optarg);
				printf("Argument for kick_it is %i\n",kick_it);
				appendData(&params,"kick_it",kick_it);
				break;
			case 'W':
				write_it = atoi(optarg);
				printf("Argument for write_it is %i\n",write_it);
				appendData(&params,"write_it",write_it);
				break;
			case 'U':
				x0_shift = atof(optarg);
				printf("Argument for x0_shift is %lf\n",x0_shift);
				appendData(&params,"x0_shift",x0_shift);
				break;
			case 'V':
				y0_shift = atof(optarg);
				printf("Argument for y0_shift is %lf\n",y0_shift);
				appendData(&params,"y0_shift",y0_shift);
				break;
			case 'S':
				sepMinEpsilon = atof(optarg);
				printf("Argument for sepMinEpsilon is %lf\n",sepMinEpsilon);
				appendData(&params,"sepMinEpsilon",sepMinEpsilon);
				break;
			case 'a':
				graph = atoi(optarg);
				printf("Argument for graph is %d\n",graph);
				appendData(&params,"graph",graph);
				break;
			case '?':
				if (optopt == 'c') {
					fprintf (stderr, "Option -%c requires an argument.\n", optopt);
				} else if (isprint (optopt)) {
					fprintf (stderr, "Unknown option `-%c'.\n", optopt);
				} else {
					fprintf (stderr,"Unknown option character `\\x%x'.\n",optopt);
				}
				return -1;
			default:
				abort ();
		}
	}
	return 0;
}

void delta_define(double *x, double *y, double x0, double y0, double *delta){
	for (unsigned int i=0; i<xDim; ++i){
		for (unsigned int j=0; j<yDim; ++j){
			delta[j*xDim + i] = 1e6*HBAR*exp( -( pow( x[i] - x0, 2)  +  pow( y[j] - y0, 2) )/(5*dx*dx) );
			EV_opt[(j*xDim + i)].x=cos( -(V[(j*xDim + i)] + delta[j*xDim + i])*(dt/(2*HBAR)));
			EV_opt[(j*xDim + i)].y=sin( -(V[(j*xDim + i)] + delta[j*xDim + i])*(dt/(2*HBAR)));
		}
	}
}


int main(int argc, char **argv){
	
	time_t start,fin;
	time(&start);
	printf("Start: %s\n", ctime(&start));
	initArr(&params,32);
	//appendData(&params,ctime(&start),0.0);
	parseArgs(argc,argv);
	hipSetDevice(device);
	//************************************************************//
	/*
	* Initialise the Params data structure to track params and variables
	*/
	//************************************************************//
	//paramS = (Params *) malloc(sizeof(Params));
	//strcpy(paramS->data,"INIT");
	//paramS->next=NULL;

	initialise(omegaX,omegaY,atoms);
	timeTotal = 0.0;
	//************************************************************//
	/*
	* Groundstate finder section
	*/
	//************************************************************//
	FileIO::writeOutParam(buffer, params, "Params.dat");
	if(read_wfc == 1){
		printf("Loading wavefunction...");
		wfc=FileIO::readIn("wfc_load","wfci_load",xDim, yDim);
		printf("Wavefunction loaded.\n");
	}
	
	double2 ph;
	double x_0,y_0;
	x_0 = 0;//(0.5*xDim)*dx;
	y_0 = 0;//(0.5*yDim)*dy;
/*	for(int i=0; i < xDim; i++ ){
		for(int j=0; j < yDim; j++ ){
			ph.x = cos( fmod( 0*atan2( y[j] - y_0, x[i] - x_0 ), 2*PI) );
			ph.y = -sin( fmod( 0*atan2( y[j] - y_0, x[i] - x_0 ), 2*PI) );
			wfc[(i*yDim + j)] = Minions::complexMult( wfc[(i*yDim + j)], ph );
		}
	}
	printf("l=%e\n",l);
*/	if(gsteps > 0){
		err=hipMemcpy(K_gpu, GK, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		err=hipMemcpy(V_gpu, GV, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		err=hipMemcpy(xPy_gpu, xPy, sizeof(double)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		err=hipMemcpy(yPx_gpu, yPx, sizeof(double)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		err=hipMemcpy(wfc_gpu, wfc, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		
		evolve(wfc_gpu, K_gpu, V_gpu, yPx_gpu, xPy_gpu, par_sum, xDim*yDim, gsteps, threads, 0, ang_mom, gpe, print, atoms, 0);
		hipMemcpy(wfc, wfc_gpu, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyDeviceToHost);
	}

	free(GV); free(GK); free(xPy); free(yPx);

	//************************************************************//
	/*
	* Evolution
	*/
	//************************************************************//
	if(esteps > 0){
		err=hipMemcpy(xPy_gpu, ExPy, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		err=hipMemcpy(yPx_gpu, EyPx, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		err=hipMemcpy(xPy_gpu, ExPy, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		err=hipMemcpy(yPx_gpu, EyPx, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		err=hipMemcpy(K_gpu, EK, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		err=hipMemcpy(V_gpu, EV, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
		err=hipMemcpy(wfc_gpu, wfc, sizeof(hipfftDoubleComplex)*xDim*yDim, hipMemcpyHostToDevice);
		if(err!=hipSuccess)
			exit(1);
			
		//delta_define(x, y, (523.6667 - 512 + x0_shift)*dx, (512.6667 - 512  + y0_shift)*dy, V_opt);
		FileIO::writeOutDouble(buffer,"V_opt",V_opt,xDim*yDim,0);
		evolve(wfc_gpu, K_gpu, V_gpu, yPx_gpu, xPy_gpu, par_sum, xDim*yDim, esteps, threads, 1, ang_mom, gpe, print, atoms, 0);
	
	}
	free(EV); free(EK); free(ExPy); free(EyPx);
	free(x);free(y);
	hipFree(wfc_gpu); hipFree(K_gpu); hipFree(V_gpu); hipFree(yPx_gpu); hipFree(xPy_gpu); hipFree(par_sum);

	time(&fin);
	//appendData(&params,ctime(&fin),0.0);
	printf("Finish: %s\n", ctime(&fin));
	printf("Total time: %ld seconds\n ",(long)fin-start);
	//appendData(&params,"t_duration",fin-start);
	return 0;
}
