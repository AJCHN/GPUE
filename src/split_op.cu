#include "hip/hip_runtime.h"
/*
* split_op.cu - GPUE: Split Operator based GPU solver for Nonlinear 
Schrodinger Equation, Copyright (C) 2011-2015, Lee J. O'Riordan 
<loriordan@gmail.com>, Tadhg Morgan, Neil Crowley. All rights reserved.

Redistribution and use in source and binary forms, with or without 
modification, are permitted provided that the following conditions are 
met:

1. Redistributions of source code must retain the above copyright 
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright 
notice, this list of conditions and the following disclaimer in the 
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its 
contributors may be used to endorse or promote products derived from 
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT 
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR 
PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF 
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING 
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS 
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "../include/split_op.h"
#include "../include/kernels.h"
#include "../include/constants.h"
#include "../include/fileIO.h"
#include "../include/tracker.h"
#include "../include/minions.h"
#include "../include/parser.h"

#include "../include/lattice.h"
#include "../include/node.h"
#include "../include/edge.h"
#include "../include/manip.h"
#include "../include/vort.h"
#include <string>
#include <iostream>

unsigned int LatticeGraph::Edge::suid = 0;
unsigned int LatticeGraph::Node::suid = 0;

char buffer[100];
int verbose; //Print more info. Not curently implemented.
int device; //GPU ID choice.
int kick_it; //Kicking mode: 0 = off, 1 = multiple, 2 = single
int graph=0; //Generate graph from vortex lattice.
double gammaY; //Aspect ratio of trapping geometry.
double omega; //Rotation rate of condensate
double timeTotal;
double angle_sweep; //Rotation angle of condensate relative to x-axis
Params *paramS;
//Array params;
double x0_shift, y0_shift; //Optical lattice shift parameters.
double Rxy; //Condensate scaling factor.
double a0x, a0y; //Harmonic oscillator length in x and y directions
double sepMinEpsilon=0.0; //Minimum separation for epsilon.

/*
 * Checks CUDA routines have exitted correctly.
 */
int isError(int result, char* c){
    if(result!=0){
        printf("Error has occurred for method %s with return type %d\n",
               c,result);
        exit(result);
    }
    return result;
}

int initialise(Grid &par){

    double omegaX = par.dval("omegaX");
    double omegaY = par.dval("omegaY");
    int N = par.ival("atoms");
    unsigned int xD=1,yD=1,zD=1;
    threads = 128;

    // number of blocks in simulation
    unsigned int b = xDim*yDim/threads;

    // largest number of elements
    unsigned long long maxElements = 65536*65536ULL; 

    if( b < (1<<16) ){
        xD = b;
    }
    else if( (b >= (1<<16) ) && (b <= (maxElements)) ){
        int t1 = log(b)/log(2);
        float t2 = (float) t1/2;
        t1 = (int) t2;
        if(t2 > (float) t1){
            xD <<= t1;
            yD <<= (t1 + 1);
        }
        else if(t2 == (float) t1){
            xD <<= t1;
            yD <<= t1;
        }
    }
    else{
        printf("Outside range of supported indexing");
        exit(-1);
    }
    printf("Compute grid dimensions chosen as X=%d    Y=%d\n",xD,yD);
    
    grid.x=xD; 
    grid.y=yD; 
    grid.z=zD; 
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
    
    int i,j; //Used in for-loops for indexing
    
    unsigned int gSize = xDim*yDim;
    double xOffset, yOffset;
    xOffset=0.0;//5.0e-6;
    yOffset=0.0;//5.0e-6;
    
    mass = 1.4431607e-25; //Rb 87 mass, kg
    par.store("Mass",mass);
    a_s = 4.67e-9;
    par.store("a_s",a_s);

    double sum = 0.0;

    a0x = sqrt(HBAR/(2*mass*omegaX));
    a0y = sqrt(HBAR/(2*mass*omegaY));
    par.store("a0x",a0x);
    par.store("a0y",a0y);
    
    Rxy = pow(15,0.2)*pow(N*a_s*sqrt(mass*omegaZ/HBAR),0.2);
    par.store("Rxy",Rxy);
    double bec_length = sqrt( HBAR/(mass*sqrt( omegaX*omegaX * 
                                               ( 1 - omega*omega) ) ));
    xMax = 6*Rxy*a0x; //10*bec_length; //6*Rxy*a0x;
    yMax = 6*Rxy*a0y; //10*bec_length;
    par.store("xMax",xMax);
    par.store("yMax",yMax);

    double pxMax, pyMax;
    pxMax = (PI/xMax)*(xDim>>1);
    pyMax = (PI/yMax)*(yDim>>1);
    par.store("pyMax",pyMax);
    par.store("pxMax",pxMax);
    
    dx = xMax/(xDim>>1);
    dy = yMax/(yDim>>1);
    par.store("dx",dx);
    par.store("dy",dy);
    
    double dpx, dpy;
    dpx = PI/(xMax);
    dpy = PI/(yMax);
    par.store("dpx",dpx);
    par.store("dpy",dpy);

    //printf("a0x=%e  a0y=%e \n dx=%e   dx=%e\n R_xy=%e\n",a0x,a0y,dx,dy,Rxy);
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
    
    //double *x,*y,*xp,*yp;
    x = (double *) malloc(sizeof(double) * xDim);
    y = (double *) malloc(sizeof(double) * yDim);
    xp = (double *) malloc(sizeof(double) * xDim);
    yp = (double *) malloc(sizeof(double) * yDim);

    /*
     * R-space and K-space grids
     */
    for(i=0; i<xDim/2; ++i){
        x[i] = -xMax + (i+1)*dx;        
        x[i + (xDim/2)] = (i+1)*dx;
        
        y[i] = -yMax + (i+1)*dy;        
        y[i + (yDim/2)] = (i+1)*dy;
        
        xp[i] = (i+1)*dpx;
        xp[i + (xDim/2)] = -pxMax + (i+1)*dpx;
        
        yp[i] = (i+1)*dpy;
        yp[i + (yDim/2)] = -pyMax + (i+1)*dpy;
    }
    
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
    
    /* Initialise wavefunction, momentum, position, angular momentum, 
       imaginary and real-time evolution operators . */
    Energy = (double*) malloc(sizeof(double) * gSize);
    r = (double *) malloc(sizeof(double) * gSize);
    Phi = (double *) malloc(sizeof(double) * gSize);
    wfc = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    wfc_backup = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * 
                                               (gSize/threads));
    K = (double *) malloc(sizeof(double) * gSize);
    V = (double *) malloc(sizeof(double) * gSize);
    V_opt = (double *) malloc(sizeof(double) * gSize);
    GK = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    GV = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EK = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EV = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EV_opt = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    xPy = (double *) malloc(sizeof(double) * gSize);
    yPx = (double *) malloc(sizeof(double) * gSize);
    ExPy = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EyPx = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EappliedField = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * 
                                                         gSize);
    
    /* Initialise wfc, EKp, and EVr buffers on GPU */
    hipMalloc((void**) &Energy_gpu, sizeof(double) * gSize);
    hipMalloc((void**) &wfc_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &Phi_gpu, sizeof(double) * gSize);
    hipMalloc((void**) &K_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &V_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &xPy_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &yPx_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &par_sum, sizeof(hipfftDoubleComplex) * (gSize/threads));
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    #ifdef __linux
    int cores = omp_get_num_procs();
    par.store("Cores_Total",cores);

    // Assuming dev system specifics (Xeon with HT -> cores detected / 2)
    par.store("Cores_Max",cores/2);
    omp_set_num_threads(cores/2);
    #pragma omp parallel for private(j)
    #endif
    for( i=0; i < xDim; i++ ){
        for( j=0; j < yDim; j++ ){
            Phi[(i*yDim + j)] = fmod(l*atan2(y[j], x[i]),2*PI);
            
            wfc[(i*yDim + j)].x = exp(-( pow((x[i])/(Rxy*a0x),2) + 
                                         pow((y[j])/(Rxy*a0y),2) ) ) *
                                  cos(Phi[(i*xDim + j)]);
            wfc[(i*yDim + j)].y = -exp(-( pow((x[i])/(Rxy*a0x),2) + 
                                          pow((y[j])/(Rxy*a0y),2) ) ) *
                                  sin(Phi[(i*xDim + j)]);
                
            V[(i*yDim + j)] = 0.5*mass*( pow(omegaX*(x[i]+xOffset),2) + 
                                         pow(gammaY*omegaY*(y[j]+yOffset),2) );
            K[(i*yDim + j)] = (HBAR*HBAR/(2*mass))*(xp[i]*xp[i] + yp[j]*yp[j]);

            GV[(i*yDim + j)].x = exp( -V[(i*xDim + j)]*(gdt/(2*HBAR)));
            GK[(i*yDim + j)].x = exp( -K[(i*xDim + j)]*(gdt/HBAR));
            GV[(i*yDim + j)].y = 0.0;
            GK[(i*yDim + j)].y = 0.0;
            
            xPy[(i*yDim + j)] = x[i]*yp[j];
            yPx[(i*yDim + j)] = -y[j]*xp[i];
            
            EV[(i*yDim + j)].x=cos( -V[(i*xDim + j)]*(dt/(2*HBAR)));
            EV[(i*yDim + j)].y=sin( -V[(i*xDim + j)]*(dt/(2*HBAR)));
            EK[(i*yDim + j)].x=cos( -K[(i*xDim + j)]*(dt/HBAR));
            EK[(i*yDim + j)].y=sin( -K[(i*xDim + j)]*(dt/HBAR));
            
            ExPy[(i*yDim + j)].x=cos(-omega*omegaX*xPy[(i*xDim + j)]*dt);
            ExPy[(i*yDim + j)].y=sin(-omega*omegaX*xPy[(i*xDim + j)]*dt);
            EyPx[(i*yDim + j)].x=cos(-omega*omegaX*yPx[(i*xDim + j)]*dt);
            EyPx[(i*yDim + j)].y=sin(-omega*omegaX*yPx[(i*xDim + j)]*dt);
    
            sum+=sqrt(wfc[(i*xDim + j)].x*wfc[(i*xDim + j)].x + 
                      wfc[(i*xDim + j)].y*wfc[(i*xDim + j)].y);
        }
    }
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
    //hdfWriteDouble(xDim, V, 0, "V_0"); //HDF COMING SOON!
    //hdfWriteComplex(xDim, wfc, 0, "wfc_0");
    FileIO::writeOutDouble(buffer,"V",V,xDim*yDim,0);
    //FileIO::writeOutDouble(buffer,"V_opt",V_opt,xDim*yDim,0);
    FileIO::writeOutDouble(buffer,"K",K,xDim*yDim,0);
    FileIO::writeOutDouble(buffer,"xPy",xPy,xDim*yDim,0);
    FileIO::writeOutDouble(buffer,"yPx",yPx,xDim*yDim,0);
    FileIO::writeOut(buffer,"WFC",wfc,xDim*yDim,0);
    FileIO::writeOut(buffer,"ExPy",ExPy,xDim*yDim,0);
    FileIO::writeOut(buffer,"EyPx",EyPx,xDim*yDim,0);
    FileIO::writeOutDouble(buffer,"Phi",Phi,xDim*yDim,0);
    FileIO::writeOutDouble(buffer,"r",r,xDim*yDim,0);
    FileIO::writeOutDouble(buffer,"x",x,xDim,0);
    FileIO::writeOutDouble(buffer,"y",y,yDim,0);
    FileIO::writeOutDouble(buffer,"px",xp,xDim,0);
    FileIO::writeOutDouble(buffer,"py",yp,yDim,0);
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    //free(V); 
    free(K); free(r); //free(Phi);

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    sum=sqrt(sum*dx*dy);
    //#pragma omp parallel for reduction(+:sum) private(j)
    for (i = 0; i < xDim; i++){
        for (j = 0; j < yDim; j++){
            wfc[(i*yDim + j)].x = (wfc[(i*yDim + j)].x)/(sum);
            wfc[(i*yDim + j)].y = (wfc[(i*yDim + j)].y)/(sum);
        }
    }
    
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
    
    result = hipfftPlan2d(&plan_2d, xDim, yDim, HIPFFT_Z2Z);
    if(result != HIPFFT_SUCCESS){
        printf("Result:=%d\n",result);
        printf("Error: Could not execute hipfftPlan2d(%s ,%d, %d).\n", "plan_2d",
                (unsigned int)xDim, (unsigned int)yDim);
        return -1;
    }

    result = hipfftPlan1d(&plan_1d, xDim, HIPFFT_Z2Z, yDim);
    if(result != HIPFFT_SUCCESS){
        printf("Result:=%d\n",result);
        printf("Error: Could not execute hipfftPlan3d(%s ,%d ,%d ).\n", 
               "plan_1d", (unsigned int)xDim, (unsigned int)yDim);
        return -1;
    }
    
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
    
    return 0;
}

int evolve( hipfftDoubleComplex *gpuWfc, hipfftDoubleComplex *gpuMomentumOp,
            hipfftDoubleComplex *gpuPositionOp, void *gpu1dyPx, void *gpu1dxPy,
            hipfftDoubleComplex *gpuParSum, int numSteps,
            int threads, unsigned int gstate, unsigned int ramp, Grid &par){

    // Re-establishing variables from parsed Grid class
    int N = par.ival("atoms");
    int printSteps = par.ival("print");
    int nonlin = par.ival("gpe");
    int lz = par.ival("ang_mom");
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    int gridSize = xDim * yDim;

    // Because no two operations are created equally. 
    // Multiplication is faster than divisions.
    double renorm_factor_2d=1.0/pow(gridSize,0.5);
    double renorm_factor_1d=1.0/pow(xDim,0.5);

    clock_t begin, end;
    double time_spent;
    double Dt;
    if(gstate==0){
        Dt = gdt;
        printf("Timestep for groundstate solver set as: %E\n",Dt);
    }
    else{
        Dt = dt;
        printf("Timestep for evolution set as: %E\n",Dt);
    }
    begin = clock();
    double omega_0=omega*omegaX;

    #if 0 
    /** Determines the initial average density at the condensate central 
    * position and calculates a value for the healing length from this. Used 
    * thereafter as the lower limit for distances between vortices. **/
    int gridSum = 1<<6;
    double *densitySubset = (double*) malloc(sizeof(double)*gridSum);
    #pragma omp parallel for private(k)
    for (int j=0; j<gridSum; ++j){
        for (int k=0; k<gridSum; ++k){
            densitySubset[j*gridSum + k] = Minions::psi2(wfc[ ( (yDim/2) - 
                (gridSum/2) + j )*yDim  + ( (xDim/2)  - (gridSum/2) + k )]);
        }
    }
    // defined central condensate density
    xi = 1/sqrt(8*PI*a_s*Minions::sumAvg(densitySubset,gridSum)/(dx*dy));
    printf("Avg healing length at centre=%E\n",xi);
    #endif

    /** ** ############################################################## ** **/
    /** **         HERE BE DRAGONS OF THE MOST DANGEROUS KIND!            ** **/
    /** ** ############################################################## ** **/

    // Double buffering and will attempt to thread free and calloc operations to
    // hide time penalty. Or may not bother.
    int num_vortices[2] = {0,0};

    // binary matrix of size xDim*yDim, 
    // 1 for vortex at specified index, 0 otherwise
    int* vortexLocation;
    int* olMaxLocation = (int*) calloc(xDim*yDim,sizeof(int));

    struct Vtx::Vortex central_vortex; //vortex closest to the central position

    // Angle of vortex lattice. Add to optical lattice for alignment.
    double vort_angle;

    // array of vortex coordinates from vortexLocation 1's
    struct Vtx::Vortex *vortCoords = NULL;

    //Previous array of vortex coordinates from vortexLocation 1's
    struct Vtx::Vortex *vortCoordsP = NULL;

    LatticeGraph::Lattice lattice; //Vortex lattice graph.
    double* adjMat;
    
    double vortOLSigma=0.0;
    double sepAvg = 0.0;
    
    int num_kick = 0;
    double t_kick = (2*PI/omega_0)/(6*Dt);
    
    for(int i=0; i < numSteps; ++i){
        if ( ramp == 1 ){
            //Adjusts omega for the appropriate trap frequency.
            omega_0=omegaX*((omega-0.39)*((double)i/(double)(numSteps)) + 0.39);
        }

        // Print-out at pre-determined rate.
        // Vortex & wfc analysis performed here also.
        if(i % printSteps == 0) { 
            printf("Step: %d    Omega: %lf\n", i, omega_0 / omegaX);
            hipMemcpy(wfc, gpuWfc, sizeof(hipfftDoubleComplex) * xDim * yDim, 
                       hipMemcpyDeviceToHost);
            end = clock();
            time_spent = (double) (end - begin) / CLOCKS_PER_SEC;
            printf("Time spent: %lf\n", time_spent);
            std::string fileName = "";
            printf("ramp=%d        gstate=%d    rg=%d        \n", 
                   ramp, gstate, ramp | (gstate << 1));
            switch (ramp | (gstate << 1)) {
                case 0: //Groundstate solver, constant Omega value.
                    fileName = "wfc_0_const";
                    break;
                case 1: //Groundstate solver, ramped Omega value.
                    fileName = "wfc_0_ramp";
                    break;
                case 2: //Real-time evolution, constant Omega value.
                    fileName = "wfc_ev";
                    vortexLocation = (int *) calloc(xDim * yDim, sizeof(int));
                    num_vortices[0] = Tracker::findVortex(vortexLocation, 
                                                         wfc, 1e-4, xDim, x, i);

                    // If initial step, locate vortices, least-squares to find
                    // exact centre, calculate lattice angle, generate optical 
                    // lattice.
                    if (i == 0) {
                        vortCoords = (struct Vtx::Vortex *) malloc(
                                sizeof(struct Vtx::Vortex) * 
                                (2 * num_vortices[0]));
                        vortCoordsP = (struct Vtx::Vortex *) malloc(
                                sizeof(struct Vtx::Vortex) * 
                                (2 * num_vortices[0]));
                        Tracker::vortPos(vortexLocation, vortCoords, xDim, wfc);
                        Tracker::lsFit(vortCoords, wfc, num_vortices[0], xDim);
                        central_vortex = Tracker::vortCentre(vortCoords, 
                                                             num_vortices[0], 
                                                             xDim);
                        vort_angle = Tracker::vortAngle(vortCoords, 
                                                        central_vortex, 
                                                        num_vortices[0]);
                        par.store("Vort_angle", vort_angle);
                        optLatSetup(central_vortex, V, vortCoords, 
                                    num_vortices[0], 
                                    vort_angle + PI * angle_sweep / 180.0,
                                    laser_power * HBAR * sqrt(omegaX * omegaY),
                                    V_opt, x, y, par);
                        sepAvg = Tracker::vortSepAvg(vortCoords, central_vortex,                                                     num_vortices[0]);
                        if (kick_it == 2) {
                            printf("Kicked it 1\n");
                            hipMemcpy(V_gpu, EV_opt, 
                                       sizeof(hipfftDoubleComplex) * xDim * yDim,
                                       hipMemcpyHostToDevice);
                        }
                        FileIO::writeOutDouble(buffer, "V_opt_1", V_opt, 
                                               xDim * yDim, 0);
                        FileIO::writeOut(buffer, "EV_opt_1", EV_opt, 
                                         xDim * yDim, 0);
                        par.store("Central_vort_x", 
                                  (double) central_vortex.coords.x);
                        par.store("Central_vort_y", 
                                  (double) central_vortex.coords.y);
                        par.store("Central_vort_winding", 
                                  (double) central_vortex.wind);
                        par.store("Num_vort", (double) num_vortices[0]);
                        FileIO::writeOutParam(buffer, par, "Params.dat");
                    }
                    else if (num_vortices[0] > num_vortices[1]) {
                        printf("Number of vortices increased from %d to %d\n", 
                               num_vortices[1], num_vortices[0]);
                        Tracker::vortPos(vortexLocation, vortCoords, xDim, wfc);
                        Tracker::lsFit(vortCoords, wfc, num_vortices[0], xDim);
                    }
                    else {
                        Tracker::vortPos(vortexLocation, vortCoords, xDim, wfc);
                        Tracker::lsFit(vortCoords, wfc, num_vortices[0], xDim);
                        Tracker::vortArrange(vortCoords, vortCoordsP, 
                                             num_vortices[0]);
                    }

                    if (graph == 1) {

                        for (int ii = 0; ii < num_vortices[0]; ++ii) {
                            std::shared_ptr<LatticeGraph::Node> 
                                n(new LatticeGraph::Node(vortCoords[ii]));
                            lattice.addVortex(std::move(n));
                        }
                        unsigned int *uids = (unsigned int *) malloc(
                                sizeof(unsigned int) *
                                lattice.getVortices().size());
                        for (size_t a=0; a < lattice.getVortices().size(); ++a){
                            uids[a] = lattice.getVortexIdx(a)->getUid();
                        }
                        if(i==0) {
                            //Lambda for vortex annihilation/creation.
                            auto killIt=[&](int idx) {
                                WFC::phaseWinding(Phi, 1, x, y, dx, dy, 
                                                  lattice.getVortexUid(idx)->
                                                      getData().coordsD.x,
                                                  lattice.getVortexUid(idx)->
                                                      getData().coordsD.y,xDim);
                                hipMemcpy(Phi_gpu, Phi, 
                                           sizeof(double) * xDim * yDim, 
                                           hipMemcpyHostToDevice);
                                cMultPhi <<<grid, threads>>> (gpuWfc, Phi_gpu, 
                                                              gpuWfc);
                            };
                            //killIt(44); //Kills vortex with UID 44


                        }
                        lattice.createEdges(1.5 * 2e-5 / dx);
                        adjMat = (double *)calloc(lattice.getVortices().size() *
                                                  lattice.getVortices().size(),
                                                   sizeof(double));
                        lattice.genAdjMat(adjMat);
                        FileIO::writeOutAdjMat(buffer, "graph", adjMat, uids, 
                                               lattice.getVortices().size(), i);
                        free(adjMat);
                        free(uids);
                        lattice.getVortices().clear();
                        lattice.getEdges().clear();
                        //exit(0);
                    }

                    FileIO::writeOutVortex(buffer, "vort_arr", vortCoords, 
                                           num_vortices[0], i);
                    printf("Located %d vortices\n", num_vortices[0]);
                    printf("Sigma=%e\n", vortOLSigma);
                    free(vortexLocation);
                    num_vortices[1] = num_vortices[0];
                    memcpy(vortCoordsP, vortCoords, 
                           sizeof(int2) * num_vortices[0]);
                    //exit(1);
                    break;


                case 3:
                    fileName = "wfc_ev_ramp";
                    break;
                default:
                    break;
            }
            if (write_it) {
                FileIO::writeOut(buffer, fileName, wfc, xDim * yDim, i);
            }
            // printf("Energy[t@%d]=%E\n",i,energy_angmom(gpuPositionOp, 
            //        gpuMomentumOp, dx, dy, gpuWfc,gstate));
/*            hipMemcpy(V_gpu, V, sizeof(double)*xDim*yDim, 
                         hipMemcpyHostToDevice);
            hipMemcpy(K_gpu, K, sizeof(double)*xDim*yDim, 
                       hipMemcpyHostToDevice);
            hipMemcpy(V_gpu, , sizeof(double)*xDim*yDim, 
                       hipMemcpyHostToDevice);
            hipMemcpy(K_gpu, K, sizeof(double)*xDim*yDim, 
                       hipMemcpyHostToDevice);
*/        
        }
    
    /** ** ############################################################## ** **/
    /** **                       More F'n' Dragons!                       ** **/
    /** ** ############################################################## ** **/
        if(i%((int)t_kick+1) == 0 && num_kick<=6 && gstate==1 && kick_it == 1 ){
            hipMemcpy(V_gpu, EV_opt, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
            ++num_kick;
        }
    /** ** ############################################################## ** **/

        /*
         * U_r(dt/2)*wfc
         */ 
        if(nonlin == 1){
            cMultDensity<<<grid,threads>>>(gpuPositionOp,gpuWfc,gpuWfc,0.5*Dt,
                                           mass,omegaZ,gstate,N*interaction);
        }
        else {
            cMult<<<grid,threads>>>(gpuPositionOp,gpuWfc,gpuWfc);
        }
                
        /*
         * U_p(dt)*fft2(wfc)
         */        
        result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD);
        scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc); //Normalise
        cMult<<<grid,threads>>>(gpuMomentumOp,gpuWfc,gpuWfc);
        result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
        scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc); //Normalise
        
        /*
         * U_r(dt/2)*wfc
         */    
        if(nonlin == 1){
            cMultDensity<<<grid,threads>>>(gpuPositionOp,gpuWfc,gpuWfc,Dt*0.5,
                                           mass,omegaZ,gstate,N*interaction);
        }
        else {
            cMult<<<grid,threads>>>(gpuPositionOp,gpuWfc,gpuWfc);
        }
        if( (i % (int)(t_kick+1) == 0 && num_kick<=6 && gstate==1) || 
            (kick_it >= 1 && i==0) ){
            hipMemcpy(V_gpu, EV, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
            printf("Got here: Cuda memcpy EV into GPU\n");
        }
        /**************************************************************/
        /* Angular momentum xPy-yPx   */
        if(lz == 1){
            switch(i%2 | (gstate<<1)){
                case 0: //Groundstate solver, even step

                    // wfc_xPy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                    angularOp<<<grid,threads>>>(omega_0, Dt, gpuWfc, 
                                                (double*) gpu1dxPy, gpuWfc);
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
    
                    // 2D forward
                    result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
    
                    // 1D inverse to wfc_yPx
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                    angularOp<<<grid,threads>>>(omega_0, Dt, gpuWfc, 
                                                (double*) gpu1dyPx, gpuWfc);
    
                    // wfc_PxPy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
    
                    // 2D Inverse
                    result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
                    break;
                
                case 1:    //Groundstate solver, odd step

                    // 2D forward
                    result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);

                    // 1D inverse to wfc_yPx
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                    angularOp<<<grid,threads>>>(omega_0, Dt, gpuWfc, 
                                                (double*) gpu1dyPx, gpuWfc);

                    // wfc_PxPy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);

                    // 2D inverse
                    result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
                    
                    // wfc_xPy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                    angularOp<<<grid,threads>>>(omega_0, Dt, gpuWfc, 
                                                (double*) gpu1dxPy, gpuWfc);
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                    break;
                
                case 2: //Real time evolution, even step

                    // wfc_xPy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dxPy, gpuWfc);
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                
                    //2D forward
                    result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD);
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);

                    // 1D inverses to wfc_yPx
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dyPx, gpuWfc);

                    // wfc_PxPy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);

                    // 2D Inverse
                    result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
                    break;
                
                case 3:    //Real time evolution, odd step

                    // 2D forward
                    result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);

                    // 1D inverse to wfc_yPx
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dyPx, gpuWfc);

                    // wfc_PxPy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD);
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);

                    // 2D inverse
                    result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
                    
                    // wfc_xPy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dxPy, gpuWfc);
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarDiv<<<grid,threads>>>(gpuWfc,renorm_factor_1d,gpuWfc);
                    break;
            
            }
        }
        /**************************************************************/
    
        if(gstate==0){
            parSum(gpuWfc, gpuParSum, xDim, yDim, threads);
        }
    }
    return 0;
}

/*
 * Used to perform parallel summation on WFC for normalisation.
 */
void parSum(double2* gpuWfc, double2* gpuParSum, int xDim, int yDim, 
            int threads){
        int grid_tmp = xDim*yDim;
        int block = grid_tmp/threads;
        int thread_tmp = threads;
        int pass = 0;
        while((double)grid_tmp/threads > 1.0){
            if(grid_tmp == xDim*yDim){
                multipass<<<block,threads,threads*sizeof(double2)>>>(&gpuWfc[0],
                    &gpuParSum[0],pass); 
            }
            else{
                multipass<<<block,thread_tmp,thread_tmp*sizeof(double2)>>>(
                    &gpuParSum[0],&gpuParSum[0],pass);
            }
            grid_tmp /= threads;
            block = (int) ceil((double)grid_tmp/threads);
            pass++;
        }
        thread_tmp = grid_tmp;
        multipass<<<1,thread_tmp,thread_tmp*sizeof(double2)>>>(&gpuParSum[0],
            &gpuParSum[0], pass);
        scalarDiv_wfcNorm<<<grid,threads>>>(gpuWfc, dx*dy, gpuParSum, gpuWfc);
}

/**
** Matches the optical lattice to the vortex lattice. 
** Moire super-lattice project.
**/
void optLatSetup(struct Vtx::Vortex centre, double* V, 
                 struct Vtx::Vortex *vArray, int num_vortices, double theta_opt,
                 double intensity, double* v_opt, double *x, double *y,
                 Grid &par){
    int i,j;
    double sepMin = Tracker::vortSepAvg(vArray,centre,num_vortices);
    sepMin = sepMin*(1 + sepMinEpsilon);
    par.store("Vort_sep",(double)sepMin);
    /*
    * Defining the necessary k vectors for the optical lattice
    */

    // Additional /2 as a result of lambda/2 period
    double k_mag = ((2*PI/(sepMin*dx))/2)*(2/sqrt(3));
    double2* k = (double2*) malloc(sizeof(double2)*3);
    par.store("kmag",(double)k_mag);
    k[0].x = k_mag * cos(0*PI/3 + theta_opt);
    k[0].y = k_mag * sin(0*PI/3 + theta_opt);
    k[1].x = k_mag * cos(2*PI/3 + theta_opt);
    k[1].y = k_mag * sin(2*PI/3 + theta_opt);
    k[2].x = k_mag * cos(4*PI/3 + theta_opt);
    k[2].y = k_mag * sin(4*PI/3 + theta_opt);
    
    double2 *r_opt = (double2*) malloc(sizeof(double2)*xDim);

/*    for (int ii = 0; ii < xDim; ++ii){
        r_opt[ii].x = 0.0 + (xDim/sepMin)*PI*(ii-centre.coords.x)/(xDim-1);
        r_opt[ii].y = 0.0 + (xDim/sepMin)*PI*(ii-centre.coords.y)/(yDim-1);
    }
*/
    FileIO::writeOut(buffer,"r_opt",r_opt,xDim,0);
    par.store("k[0].x",(double)k[0].x);
    par.store("k[0].y",(double)k[0].y);
    par.store("k[1].x",(double)k[1].x);
    par.store("k[1].y",(double)k[1].y);
    par.store("k[2].x",(double)k[2].x);
    par.store("k[2].y",(double)k[2].y);

    // sin(theta_opt)*(sepMin);
    double x_shift = dx*(9+(0.5*xDim-1) - centre.coords.x);

    // cos(theta_opt)*(sepMin);
    double y_shift = dy*(0+(0.5*yDim-1) - centre.coords.y);

    printf("Xs=%e\nYs=%e\n",x_shift,y_shift);

    //#pragma omp parallel for private(j)
    for ( j=0; j<yDim; ++j ){
        for ( i=0; i<xDim; ++i ){
            v_opt[j*xDim + i] = intensity*(
                                pow( ( cos( k[0].x*( x[i] + x_shift ) + 
                                       k[0].y*( y[j] + y_shift ) ) ), 2) +
                                pow( ( cos( k[1].x*( x[i] + x_shift ) + 
                                       k[1].y*( y[j] + y_shift ) ) ), 2) +
                                pow( ( cos( k[2].x*( x[i] + x_shift ) + 
                                       k[2].y*( y[j] + y_shift ) ) ), 2)
/*                  
                              + pow( abs( cos( k[0].x*( r_opt[i].x + x_shift ) +
                                     k[0].y*( r_opt[j].y + y_shift ) ) ), 2)
                              + pow( abs( cos( k[1].x*( r_opt[i].x + x_shift ) +
                                     k[1].y*( r_opt[j].y + y_shift ) ) ), 2)
                              + pow( abs( cos( k[2].x*( r_opt[i].x + x_shift ) +
                                     k[2].y*( r_opt[j].y + y_shift ) ) ), 2)
*/              );
            EV_opt[(j*xDim + i)].x=cos( -(V[(j*xDim + i)] + 
                                   v_opt[j*xDim + i])*(dt/(2*HBAR)));
            EV_opt[(j*xDim + i)].y=sin( -(V[(j*xDim + i)] + 
                                   v_opt[j*xDim + i])*(dt/(2*HBAR)));
        }
    }
}

/**
** Calculates energy and angular momentum of current state. Implementation not fully finished.
**/
double energy_angmom(double *Energy, double* Energy_gpu, double2 *V_op, 
                     double2 *K_op, double dx, double dy, double2 *gpuWfc, 
                     int gState){
    double renorm_factor_2d=1.0/pow(xDim*yDim,0.5);
    double result=0;

    for (int i=0; i < xDim*yDim; ++i){
        Energy[i] = 0.0; 
    }
    
    
/*    hipMalloc((void**) &energy_gpu, sizeof(double2) * xDim*yDim);

    energyCalc<<<grid,threads>>>( gpuWfc, V_op, 0.5*dt, energy_gpu, gState,1,
                                  i 0.5*sqrt(omegaZ/mass));
    result = hipfftExecZ2Z( plan_2d, gpuWfc, gpuWfc, HIPFFT_FORWARD );
    scalarDiv<<<grid,threads>>>( gpuWfc, renorm_factor_2d, gpuWfc ); //Normalise

    energyCalc<<<grid,threads>>>( gpuWfc, K_op, dt, energy_gpu, gState,0, 
                                  0.5*sqrt(omegaZ/mass));
    result = hipfftExecZ2Z( plan_2d, gpuWfc, gpuWfc, HIPFFT_BACKWARD );
    scalarDiv<<<grid,threads>>>( gpuWfc, renorm_factor_2d, gpuWfc ); //Normalise
    
    err=hipMemcpy(energy, energy_gpu, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                   hipMemcpyDeviceToHost);
    
    for(int i=0; i<xDim*yDim; i++){
        result += energy[i].x;
        //printf("En=%E\n",result*dx*dy);
    }
*/
    return result*dx*dy;
    
}


//##############################################################################
//##############################################################################

/*
 * Used to perform parallel summation using templates from c++
 */
template<typename T> void parSum(T *gpuToSumArr, T *gpuParSum, int xDim, 
                                 int yDim, int threads){
    int grid_tmp = xDim*yDim;
    int block = grid_tmp/threads;
    int thread_tmp = threads;
    int pass = 0;
    while((double)grid_tmp/threads > 1.0){
        if(grid_tmp == xDim*yDim){
            multipass<<<block,threads,threads*sizeof(T)>>>(
                &gpuToSumArr[0],&gpuParSum[0],pass);
             }
        else{
            multipass<<<block,thread_tmp,thread_tmp*sizeof(T)>>>(
                &gpuParSum[0],&gpuParSum[0],pass);
        }
        grid_tmp /= threads;
        block = (int) ceil((double)grid_tmp/threads);
        pass++;
    }
    thread_tmp = grid_tmp;
    multipass<<<1,thread_tmp,thread_tmp*sizeof(double2)>>>(&gpuParSum[0],
                                                           &gpuParSum[0], pass);
    scalarDiv_wfcNorm<<<grid,threads>>>(gpuToSumArr, dx*dy, gpuParSum, 
                                        gpuToSumArr);
}
//##############################################################################
//##############################################################################

void delta_define(double *x, double *y, double x0, double y0, double *delta){
    for (int i=0; i<xDim; ++i){
        for (int j=0; j<yDim; ++j){
            delta[j*xDim + i] = 1e6*HBAR*exp( -( pow( x[i] - x0, 2) + 
                                pow( y[j] - y0, 2) )/(5*dx*dx) );
            EV_opt[(j*xDim + i)].x = cos( -(V[(j*xDim + i)] + 
                                     delta[j*xDim + i])*(dt/(2*HBAR)));
            EV_opt[(j*xDim + i)].y = sin( -(V[(j*xDim + i)] + 
                                     delta[j*xDim + i])*(dt/(2*HBAR)));
        }
    }
}


int main(int argc, char **argv){
    
    time_t start,fin;
    time(&start);
    printf("Start: %s\n", ctime(&start));
    //initArr(&params,32);
    //appendData(&params,ctime(&start),0.0);
    Grid par = parseArgs(argc,argv);
    hipSetDevice(device);
    //************************************************************//
    /*
    * Initialise the Params data structure to track params and variables
    */
    //************************************************************//
    //paramS = (Params *) malloc(sizeof(Params));
    //strcpy(paramS->data,"INIT");
    //paramS->next=NULL;

    initialise(par);
    timeTotal = 0.0;
    //************************************************************//
    /*
    * Groundstate finder section
    */
    //************************************************************//
    FileIO::writeOutParam(buffer, par, "Params.dat");
    if(read_wfc == 1){
        printf("Loading wavefunction...");
        wfc=FileIO::readIn("wfc_load","wfci_load",xDim, yDim);
        printf("Wavefunction loaded.\n");
    }
    
/*
    double x_0,y_0;
    x_0 = 0;//(0.5*xDim)*dx;
    y_0 = 0;//(0.5*yDim)*dy;
    for(int i=0; i < xDim; i++ ){
        for(int j=0; j < yDim; j++ ){
            ph.x = cos( fmod( 0*atan2( y[j] - y_0, x[i] - x_0 ), 2*PI) );
            ph.y = -sin( fmod( 0*atan2( y[j] - y_0, x[i] - x_0 ), 2*PI) );
            wfc[(i*yDim + j)] = Minions::complexMult( wfc[(i*yDim + j)], ph );
        }
    }
    printf("l=%e\n",l);
*/
    if(gsteps > 0){
        err=hipMemcpy(K_gpu, GK, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        err=hipMemcpy(V_gpu, GV, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        err=hipMemcpy(xPy_gpu, xPy, sizeof(double)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        err=hipMemcpy(yPx_gpu, yPx, sizeof(double)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        err=hipMemcpy(wfc_gpu, wfc, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        
        evolve(wfc_gpu, K_gpu, V_gpu, yPx_gpu, xPy_gpu, par_sum, 
               par.ival("gsteps"), 128, 0, 0, par);
        hipMemcpy(wfc, wfc_gpu, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                   hipMemcpyDeviceToHost);
    }

    free(GV); free(GK); free(xPy); free(yPx);

    //************************************************************//
    /*
    * Evolution
    */
    //************************************************************//
    if(esteps > 0){
        err=hipMemcpy(xPy_gpu, ExPy, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        err=hipMemcpy(yPx_gpu, EyPx, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        err=hipMemcpy(xPy_gpu, ExPy, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        err=hipMemcpy(yPx_gpu, EyPx, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        err=hipMemcpy(K_gpu, EK, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        err=hipMemcpy(V_gpu, EV, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
        err=hipMemcpy(wfc_gpu, wfc, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            exit(1);
            
        // delta_define(x, y, (523.6667 - 512 + x0_shift)*dx, 
        //              (512.6667 - 512 + y0_shift)*dy, V_opt);
        FileIO::writeOutDouble(buffer,"V_opt",V_opt,xDim*yDim,0);
        evolve(wfc_gpu, K_gpu, V_gpu, yPx_gpu, xPy_gpu, par_sum, 
               par.ival("esteps"), 128, 1, 0, par);
    
    }
    free(EV); free(EK); free(ExPy); free(EyPx);
    free(x);free(y);
    hipFree(wfc_gpu); hipFree(K_gpu); hipFree(V_gpu); hipFree(yPx_gpu); 
    hipFree(xPy_gpu); hipFree(par_sum);

    time(&fin);
    //appendData(&params,ctime(&fin),0.0);
    printf("Finish: %s\n", ctime(&fin));
    printf("Total time: %ld seconds\n ",(long)fin-start);
    //appendData(&params,"t_duration",fin-start);
    return 0;
}
