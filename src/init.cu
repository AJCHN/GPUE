#include "hip/hip_runtime.h"

#include "../include/init.h"

int init(Op &opr, Grid &par, Wave &wave){

    set_fns(par, opr, wave);

    // Re-establishing variables from parsed Grid class
    // Initializes uninitialized variables to 0 values
    std::string data_dir = par.sval("data_dir");
    int dimnum = par.ival("dimnum");
    int N = par.ival("atoms");
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    int zDim = par.ival("zDim");
    bool write_file = par.bval("write_file");
    dim3 threads;
    unsigned int gSize = xDim*yDim;
    if (dimnum == 3){
        gSize *= zDim;
    }
    double omega = par.dval("omega");
    double gdt = par.dval("gdt");
    double dt = par.dval("dt");
    double omegaX = par.dval("omegaX");
    double omegaY = par.dval("omegaY");
    double omegaZ = par.dval("omegaZ");
    double gammaY = par.dval("gammaY"); //Aspect ratio of trapping geometry.
    double l = par.dval("winding");
    double box_size = par.dval("box_size");
    double *Energy;
    double *r;
    double *V_opt;
    double *Bz;
    double *Bx;
    double *By;
    double *Energy_gpu;
    hipfftDoubleComplex *wfc;
    if (par.bval("read_wfc") == true){
        wfc = par.cufftDoubleComplexval("wfc");
    }
    hipfftDoubleComplex *EV_opt;
    hipfftDoubleComplex *wfc_backup;
    hipfftDoubleComplex *EappliedField;
    hipfftDoubleComplex *par_sum;
    hipMalloc((void**) &par_sum, sizeof(double2)*gSize);

    std::cout << "gSize is: " << gSize << '\n';
    hipfftResult result;
    hipfftHandle plan_1d;
    hipfftHandle plan_2d;
    hipfftHandle plan_3d;
    hipfftHandle plan_other2d;
    hipfftHandle plan_dim2;
    hipfftHandle plan_dim3;

    std::string buffer;
    double Rxy; //Condensate scaling factor.
    double a0x, a0y, a0z; //Harmonic oscillator length in x and y directions

    generate_grid(par);

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    double mass = 1.4431607e-25; //Rb 87 mass, kg
    par.store("mass",mass);
    double a_s = 4.76e-9;
    par.store("a_s",a_s);

    double sum = 0.0;

    a0x = sqrt(HBAR/(2*mass*omegaX));
    a0y = sqrt(HBAR/(2*mass*omegaY));
    a0z = sqrt(HBAR/(2*mass*omegaZ));
    par.store("a0x",a0x);
    par.store("a0y",a0y);
    par.store("a0z",a0z);

    // Let's go ahead and define the gDensConst here
    // N*4*HBAR*HBAR*PI*(4.67e-9/mass)*sqrt(mass*(omegaZ)/(2*PI*HBAR)
    double gDenConst = N*4*HBAR*HBAR*PI*(4.67e-9/mass);
    if (dimnum == 2){
        gDenConst*= sqrt(mass*(omegaZ)/(2*PI*HBAR));
    }
    par.store("gDenConst", gDenConst);

    Rxy = pow(15,0.2)*pow(N*a_s*sqrt(mass*omegaZ/HBAR),0.2);
    par.store("Rxy",Rxy);

    //std::cout << "Rxy is: " << Rxy << '\n';
    double xMax, yMax, zMax;
    if (box_size > 0){
        xMax = box_size;
        yMax = box_size;
        zMax = box_size;
    }
    else{
        xMax = 6*Rxy*a0x;
        yMax = 6*Rxy*a0y;
        zMax = 6*Rxy*a0z;
    }
    par.store("xMax",xMax);
    par.store("yMax",yMax);
    par.store("zMax",zMax);

    double pxMax, pyMax, pzMax;
    pxMax = (PI/xMax)*(xDim>>1);
    pyMax = (PI/yMax)*(yDim>>1);
    pzMax = (PI/zMax)*(zDim>>1);
    par.store("pyMax",pyMax);
    par.store("pxMax",pxMax);
    par.store("pzMax",pzMax);

    double dx = xMax/(xDim>>1);
    double dy = yMax/(yDim>>1);
    double dz = zMax/(zDim>>1);
    if (dimnum == 2){
        dz = 1;
    }
    par.store("dx",dx);
    par.store("dy",dy);
    par.store("dz",dz);

    double dpx, dpy, dpz;
    dpx = PI/(xMax);
    dpy = PI/(yMax);
    dpz = PI/(zMax);
    //std::cout << "yMax is: " << yMax << '\t' << "xMax is: " << xMax << '\n';
    //std::cout << "dpx and dpy are:" << '\n';
    //std::cout << dpx << '\t' << dpy << '\n';
    par.store("dpx",dpx);
    par.store("dpy",dpy);
    par.store("dpz",dpz);

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    /* Initialise wavefunction, momentum, position, angular momentum,
       imaginary and real-time evolution operators . */
    Energy = (double*) malloc(sizeof(double) * gSize);
    r = (double *) malloc(sizeof(double) * gSize);
    V_opt = (double *) malloc(sizeof(double) * gSize);
    EV_opt = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    Bz = (double *) malloc(sizeof(double) * gSize);
    Bx = (double *) malloc(sizeof(double) * gSize);
    By = (double *) malloc(sizeof(double) * gSize);
    EappliedField = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) *
                                                         gSize);

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    #ifdef __linux
    int cores = omp_get_num_procs();
    par.store("Cores_Total",cores);

    // Assuming dev system specifics (Xeon with HT -> cores detected / 2)
    par.store("Cores_Max",cores/2);
    omp_set_num_threads(cores/2);

    //#pragma omp parallel for private(j)
    #endif

    par.store("gSize", xDim*yDim*zDim);
    generate_fields(par);
    double *K = par.dsval("K");
    double *Ax = par.dsval("Ax");
    double *Ay = par.dsval("Ay");
    double *Az = par.dsval("Az");
    double *V = par.dsval("V");

    double *pAx = par.dsval("pAx");
    double *pAy = par.dsval("pAy");
    double *pAz = par.dsval("pAz");

    double *x = par.dsval("x");
    double *y = par.dsval("y");
    double *z = par.dsval("z");

    double2 *GpAx = par.cufftDoubleComplexval("GpAx");
    double2 *GpAy = par.cufftDoubleComplexval("GpAy");
    double2 *GpAz = par.cufftDoubleComplexval("GpAz");
    double2 *EpAx = par.cufftDoubleComplexval("EpAx");
    double2 *EpAy = par.cufftDoubleComplexval("EpAy");
    double2 *EpAz = par.cufftDoubleComplexval("EpAz");

    double2 *GV = par.cufftDoubleComplexval("GV");
    double2 *EV = par.cufftDoubleComplexval("EV");
    double2 *GK = par.cufftDoubleComplexval("GK");
    double2 *EK = par.cufftDoubleComplexval("EK");

    wfc = par.cufftDoubleComplexval("wfc");

    int index = 0;
    for(int i=0; i < gSize; i++ ){
        sum+=sqrt(wfc[i].x*wfc[i].x + wfc[i].y*wfc[i].y);
    }

    if (write_file){
        if (dimnum == 2){
            Bz = curl2d(par, Ax, Ay);
        }
        if (dimnum == 3){
            std::cout << "Calculating the 3d curl..." << '\n';
                    Bx = curl3d_x(par, Ax, Ay, Az);
                    By = curl3d_y(par, Ax, Ay, Az);
                    Bz = curl3d_z(par, Ax, Ay, Az);
                    std::cout << "Finished calculating Curl" << '\n';
        }
        std::cout << "writing initial variables to file..." << '\n';
        //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
        //hdfWriteDouble(xDim, V, 0, "V_0"); //HDF COMING SOON!
        //hdfWriteComplex(xDim, wfc, 0, "wfc_0");
        FileIO::writeOutDouble(buffer, data_dir + "V",V,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "K",K,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "pAy",pAy,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "pAx",pAx,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "Ax",Ax,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "Ay",Ay,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "Az",Az,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "x",x,xDim,0);
        FileIO::writeOutDouble(buffer, data_dir + "y",y,yDim,0);
        FileIO::writeOutDouble(buffer, data_dir + "z",z,zDim,0);
        FileIO::writeOutDouble(buffer, data_dir + "Bz",Bz,gSize,0);
        if (dimnum == 3){
            FileIO::writeOutDouble(buffer, data_dir + "Bx",Bx,gSize,0);
            FileIO::writeOutDouble(buffer, data_dir + "By",By,gSize,0);
        }
        FileIO::writeOut(buffer, data_dir + "WFC",wfc,gSize,0);
        FileIO::writeOut(buffer, data_dir + "EpAz",EpAz,gSize,0);
        FileIO::writeOut(buffer, data_dir + "EpAy",EpAy,gSize,0);
        FileIO::writeOut(buffer, data_dir + "EpAx",EpAx,gSize,0);
        FileIO::writeOut(buffer, data_dir + "GK",GK,gSize,0);
        FileIO::writeOut(buffer, data_dir + "GV",GV,gSize,0);
        FileIO::writeOut(buffer, data_dir + "GpAx",GpAx,gSize,0);
        FileIO::writeOut(buffer, data_dir + "GpAy",GpAy,gSize,0);
        FileIO::writeOut(buffer, data_dir + "GpAz",GpAz,gSize,0);
    }

    if (par.bval("read_wfc") == false){
        sum=sqrt(sum*dx*dy*dz);
        for (int i = 0; i < gSize; i++){
            wfc[i].x = (wfc[i].x)/(sum);
            wfc[i].y = (wfc[i].y)/(sum);
        }
    }

    result = hipfftPlan2d(&plan_2d, xDim, yDim, HIPFFT_Z2Z);
    if(result != HIPFFT_SUCCESS){
        printf("Result:=%d\n",result);
        printf("Error: Could not execute hipfftPlan2d(%s, %d, %d).\n", "plan_2d",
                (unsigned int)xDim, (unsigned int)yDim);
        return -1;
    }
    generate_plan_other2d(&plan_other2d, par);

    generate_plan_other3d(&plan_1d, par, 0);
    generate_plan_other3d(&plan_dim2, par, 1);
    generate_plan_other3d(&plan_dim3, par, 2);
    result = hipfftPlan3d(&plan_3d, xDim, yDim, zDim, HIPFFT_Z2Z);
    if(result != HIPFFT_SUCCESS){
        printf("Result:=%d\n",result);
        printf("Error: Could not execute hipfftPlan3d(%s, %d, %d, %d).\n", 
                "plan_3d",
                (unsigned int)xDim, (unsigned int)yDim, (unsigned int) zDim);
        return -1;
    }

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    //std::cout << GV[0].x << '\t' << GK[0].x << '\t'
    //          << pAy[0] << '\t' << pAx[0] << '\n';

    //std::cout << "storing variables..." << '\n';

    // Storing variables that have been initialized
    // Re-establishing variables from parsed Grid class
    // Initializes uninitialized variables to 0 values
    par.store("Energy", Energy);
    par.store("r", r);
    par.store("Energy_gpu", Energy_gpu);
    par.store("wfc", wfc);
    par.store("EV_opt", EV_opt);
    par.store("V_opt", V_opt);
    par.store("wfc_backup", wfc_backup);
    par.store("EappliedField", EappliedField);
    par.store("par_sum", par_sum);

    par.store("result", result);
    par.store("plan_1d", plan_1d);
    par.store("plan_2d", plan_2d);
    par.store("plan_other2d", plan_other2d);
    par.store("plan_3d", plan_3d);
    par.store("plan_dim2", plan_dim2);
    par.store("plan_dim3", plan_dim3);

    std::cout << "variables stored" << '\n';

    return 0;
}

// initializing all variables for 3d
int init_3d(Op &opr, Grid &par, Wave &wave){

    int max_threads = 128;

    // Setting functions for operators
    set_fns(par, opr, wave);
    //par.set_fns();
    //par.set_fns();

    // Re-establishing variables from parsed Grid class
    // Initializes uninitialized variables to 0 values
    std::string data_dir = par.sval("data_dir");
    int N = par.ival("atoms");
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    int zDim = par.ival("zDim");
    dim3 threads(max_threads,1,1);
    bool write_file = par.bval("write_file");
    unsigned int gSize = xDim*yDim*zDim;
    double omega = par.dval("omega");
    double gdt = par.dval("gdt");
    double dt = par.dval("dt");
    double omegaX = par.dval("omegaX");
    double omegaY = par.dval("omegaY");
    double omegaZ = par.dval("omegaZ");
    double box_size = par.dval("box_size");
    double gammaY = par.dval("gammaY"); //Aspect ratio of trapping geometry.
    double l = par.dval("winding");
    double *x;
    double *y;
    double *z;
    double *xp;
    double *yp;
    double *zp;
    double *Energy;
    double *r;
    double *V;
    double *V_opt;
    double *Phi;
    double *Phi_gpu;
    double *K;
    double *pAy;
    double *pAx;
    double *pAz;
    double *Ax;
    double *Ay;
    double *Az;
    double *pAy_gpu;
    double *pAx_gpu;
    double *pAz_gpu;
    double *Energy_gpu;
    hipfftDoubleComplex *wfc;
    if (par.bval("read_wfc") == true){
        wfc = par.cufftDoubleComplexval("wfc");
    }
    hipfftDoubleComplex *V_gpu;
    hipfftDoubleComplex *EV_opt;
    hipfftDoubleComplex *wfc_backup;
    hipfftDoubleComplex *GK;
    hipfftDoubleComplex *GV;
    hipfftDoubleComplex *GpAx;
    hipfftDoubleComplex *GpAy;
    hipfftDoubleComplex *GpAz;
    hipfftDoubleComplex *EV;
    hipfftDoubleComplex *EK;
    hipfftDoubleComplex *EpAy;
    hipfftDoubleComplex *EpAx;
    hipfftDoubleComplex *EpAz;
    hipfftDoubleComplex *EappliedField;
    hipfftDoubleComplex *wfc_gpu;
    hipfftDoubleComplex *K_gpu;
    hipfftDoubleComplex *par_sum;

    //std::cout << omegaX << '\t' << omegaY << '\n';
    //std::cout << "xDim is: " << xDim << '\t' <<  "yDim is: " << yDim << '\t'
    //          << "zDim is: " << zDim << '\n';

    hipfftResult result;
    //hipfftHandle plan_1d;
    hipfftHandle plan_3d;

    dim3 grid = par.grid;

    std::string buffer;

    double Rxy; //Condensate scaling factor.
    double a0x, a0y, a0z; //Harmonic oscillator length in x and y directions

    int xD = 1, yD = 1, zD = 1;

    if (xDim <= max_threads){
        threads.x = xDim;
        threads.y = 1;
        threads.z = 1;

        xD = 1;
        yD = yDim;
        zD = zDim;
    }
    else{
        int count = 0;
        int dim_tmp = xDim;
        while (dim_tmp > max_threads){
            count++;
            dim_tmp /= 2;
        }

        std::cout << "count is: " << count << '\n';

        threads.x = dim_tmp;
        threads.y = 1;
        threads.z = 1;
        xD = pow(2,count);
        yD = yDim;
        zD = zDim;
    }

    std::cout << "threads in x are: " << threads.x << '\n';
    std::cout << "dimensions are: " << xD << '\t' << yD << '\t' << zD << '\n';

    grid.x=xD;
    grid.y=yD;
    grid.z=zD;

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    int i, j, k; //Used in for-loops for indexing


    double mass = 1.4431607e-25; //Rb 87 mass, kg
    par.store("mass",mass);
    double a_s = 4.76e-9;
    par.store("a_s",a_s);

    // Let's go ahead and define the gDensConst here
    // N*4*HBAR*HBAR*PI*(4.67e-9/mass)*sqrt(mass*(omegaZ)/(2*PI*HBAR)
    double gDenConst = N*4*HBAR*HBAR*PI*(4.76e-9/mass);
    par.store("gDenConst", gDenConst);


    double sum = 0.0;

    a0x = pow(HBAR/(2*mass*omegaX), 0.5);
    a0y = pow(HBAR/(2*mass*omegaY), 0.5);
    a0z = pow(HBAR/(2*mass*omegaZ), 0.5);
    par.store("a0x",a0x);
    par.store("a0y",a0y);
    par.store("a0z",a0z);

    //std::cout << "a0x and y are: " << a0x << '\t' << a0y << '\n';

    //std::cout << N << '\t' << a_s << '\t' << mass << '\t' << omegaZ << '\n';

    Rxy = pow(15,0.2)*pow(N*a_s*sqrt(mass*omegaZ/HBAR),0.2);
    par.store("Rxy",Rxy);
    double bec_length = sqrt( HBAR/(mass*sqrt( omegaX*omegaX *
                                               ( 1 - omega*omega) ) ));

    //std::cout << "Rxy is: " << Rxy << '\n';
    double xMax = box_size;
    double yMax = box_size;
    double zMax = box_size;
    par.store("xMax",xMax);
    par.store("yMax",yMax);
    par.store("zMax",zMax);

    double pxMax, pyMax, pzMax;
    pxMax = (PI/xMax)*(xDim>>1);
    pyMax = (PI/yMax)*(yDim>>1);
    pzMax = (PI/zMax)*(zDim>>1);
    par.store("pyMax",pyMax);
    par.store("pxMax",pxMax);
    par.store("pzMax",pzMax);

    double dx = xMax/(xDim>>1);
    double dy = yMax/(yDim>>1);
    double dz = zMax/(zDim>>1);
    par.store("dx",dx);
    par.store("dy",dy);
    par.store("dz",dz);

    double dpx, dpy, dpz;
    dpx = PI/(xMax);
    dpy = PI/(yMax);
    dpz = PI/(zMax);
    //std::cout << "yMax is: " << yMax << '\t' << "xMax is: " << xMax << '\n';
    //std::cout << "dpx and dpy are:" << '\n';
    //std::cout << dpx << '\t' << dpy << '\n';
    par.store("dpx",dpx);
    par.store("dpy",dpy);
    par.store("dpz",dpz);

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    //double *x,*y,*xp,*yp;
    x = (double *) malloc(sizeof(double) * xDim);
    y = (double *) malloc(sizeof(double) * yDim);
    z = (double *) malloc(sizeof(double) * zDim);
    xp = (double *) malloc(sizeof(double) * xDim);
    yp = (double *) malloc(sizeof(double) * yDim);
    zp = (double *) malloc(sizeof(double) * zDim);

    //std::cout << "dx and dy are: " << '\n';
    //std::cout << dx << '\t' << dy << '\n';
    // creating x,y,z,xp,yp,zp
    for(i=0; i<xDim/2; ++i){
        x[i] = -xMax + i*dx;
        x[i + (xDim/2)] = i*dx;

        xp[i] = i*dpx;
        xp[i + (xDim/2)] = -pxMax + i*dpx;

    }
    for(i=0; i<yDim/2; ++i){
        y[i] = -yMax + i*dy;
        y[i + (yDim/2)] = i*dy;

        yp[i] = i*dpy;
        yp[i + (yDim/2)] = -pyMax + i*dpy;

    }
    for(i=0; i<zDim/2; ++i){
        z[i] = -zMax + i*dz;
        z[i + (zDim/2)] = i*dz;

        zp[i] = i*dpz;
        zp[i + (zDim/2)] = -pzMax + i*dpz;

    }

    par.store("x", x);
    par.store("y", y);
    par.store("z", z);
    par.store("xp", xp);
    par.store("yp", yp);
    par.store("zp", zp);

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
    Energy = (double*) malloc(sizeof(double) * gSize);
    r = (double *) malloc(sizeof(double) * gSize);
    Phi = (double *) malloc(sizeof(double) * gSize);
    if (par.bval("read_wfc") == false){
        wfc = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    }
    wfc_backup = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) *
                                               (gSize/threads.x));
    K = (double *) malloc(sizeof(double) * gSize);
    V = (double *) malloc(sizeof(double) * gSize);
    V_opt = (double *) malloc(sizeof(double) * gSize);
    GK = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    GV = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    GpAx = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    GpAy = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    GpAz = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EK = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EV = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EV_opt = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    pAy = (double *) malloc(sizeof(double) * gSize);
    pAx = (double *) malloc(sizeof(double) * gSize);
    pAz = (double *) malloc(sizeof(double) * gSize);
    Ax = (double *) malloc(sizeof(double) * gSize);
    Ay = (double *) malloc(sizeof(double) * gSize);
    Az = (double *) malloc(sizeof(double) * gSize);
    EpAy = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EpAx = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EpAz = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);
    EappliedField = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) *
                                                         gSize);

    hipMalloc((void**) &Energy_gpu, sizeof(double) * gSize);
    hipMalloc((void**) &wfc_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &Phi_gpu, sizeof(double) * gSize);
    hipMalloc((void**) &K_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &V_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &pAy_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &pAx_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &pAz_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void**) &par_sum, sizeof(hipfftDoubleComplex)*(gSize/threads.x));
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    //std::cout << "all variables malloc'd" << '\n';

    #ifdef __linux
    int cores = omp_get_num_procs();
    par.store("Cores_Total",cores);

    // Assuming dev system specifics (Xeon with HT -> cores detected / 2)
    par.store("Cores_Max",cores/2);
    omp_set_num_threads(cores/2);
    //std::cout << "GAMMAY IS: " << gammaY << '\n';
    //#pragma omp parallel for private(k)
    #endif
    int index;
    // Setting Ax, Ay, and Az if from file
    if (par.Afn == "file"){
        file_A(par.Axfile, Ax, omega);
        par.store("Ax",Ax);

        file_A(par.Ayfile, Ay, omega);
        par.store("Ay", Ay);

        file_A(par.Azfile, Az, omega);
        par.store("Az", Az);

        std::cout << "finished reading Ax / Ay / Az from file" << '\n';
    }
    for( i=0; i < xDim; i++ ){
        for( j=0; j < yDim; j++ ){
            for( k=0; k < zDim; k++ ){
                index = (i * yDim * zDim) + (j * zDim) + k;
                if (par.Afn == "rotation"){
                    Phi[index] = fmod(l*atan2(y[j], x[i]),2*PI);
                }
                else if (par.Vfn == "torus"){
                    double xOffset = par.dval("x0_shift");
                    double yOffset = par.dval("y0_shift");
                    double rMax = par.dval("xMax");
                    double fudge = par.dval("fudge");
                    double x_loc = x[i] - xOffset;
                    double y_loc = y[j] - yOffset;
                    double radius = sqrt(x_loc*x_loc + y_loc*y_loc)
                                    - 0.5*rMax*fudge;
                    Phi[index] = fmod(l*atan2(z[k], radius),2*PI);
                }
                else{
                    Phi[index] = 0.0;
                }

                if (par.bval("read_wfc") != true){
                    wfc[index] = wave.Wfc_fn(par, Phi[index],i,j,k);
                    sum+=sqrt(wfc[index].x*wfc[index].x + 
                              wfc[index].y*wfc[index].y);
                }
                
                V[index] = opr.V_fn(par, opr, i, j, k);
                K[index] = opr.K_fn(par, opr, i, j, k);
    
                GV[index].x = exp( -V[index]*(gdt/(2*HBAR)));
                GK[index].x = exp( -K[index]*(gdt/HBAR));
                GV[index].y = 0.0;
                GK[index].y = 0.0;

                // Ax and Ay will be calculated here but are used only for
                // debugging. They may be needed later for magnetic field calc
                if (par.Afn != "file"){
                    Ax[index] = opr.Ax_fn(par, opr, i, j, k);
                    Ay[index] = opr.Ay_fn(par, opr, i, j, k);
                    Az[index] = opr.Az_fn(par, opr, i, j, k);
                }
                
                pAy[index] = pAy_fn(par, opr, i, j, k);
                pAx[index] = pAx_fn(par, opr, i, j, k);
                pAz[index] = pAz_fn(par, opr, i, j, k);
    
                GpAx[index].x = exp(-pAx[index]*gdt);
                GpAx[index].y = 0;
                GpAy[index].x = exp(-pAy[index]*gdt);
                GpAy[index].y = 0;
                GpAz[index].x = exp(-pAz[index]*gdt);
                GpAz[index].y = 0;

                EV[index].x=cos( -V[index]*(dt/(2*HBAR)));
                EV[index].y=sin( -V[index]*(dt/(2*HBAR)));
                EK[index].x=cos( -K[index]*(dt/HBAR));
                EK[index].y=sin( -K[index]*(dt/HBAR));

                EpAy[index].x=cos(-pAy[index]*dt);
                EpAy[index].y=sin(-pAy[index]*dt);
                EpAx[index].x=cos(-pAx[index]*dt);
                EpAx[index].y=sin(-pAx[index]*dt);
                EpAz[index].x=cos(-pAz[index]*dt);
                EpAz[index].y=sin(-pAz[index]*dt);

            }
        }
    }

    if (write_file){
        // Calculating the curl 
        std::cout << "Calculating the 3d curl..." << '\n';
        double *Bx = curl3d_x(par, Ax, Ay, Az);
        double *By = curl3d_y(par, Ax, Ay, Az);
        double *Bz = curl3d_z(par, Ax, Ay, Az);
        std::cout << "Finished calculating Curl" << '\n';

        std::cout << "writing initial variables to file..." << '\n';
        //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//
        //hdfWriteDouble(xDim, V, 0, "V_0"); //HDF COMING SOON!
        //hdfWriteComplex(xDim, wfc, 0, "wfc_0");
        //FileIO::writeOutDouble(buffer, data_dir + "V_opt",V_opt,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "V",V,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "K",K,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "pAy",pAy,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "pAx",pAx,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "pAz",pAz,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "Ax",Ax,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "Ay",Ay,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "Az",Az,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "Bz",Bz,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "By",By,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "Bx",By,gSize,0);
        FileIO::writeOut(buffer, data_dir + "WFC",wfc,gSize,0);
        FileIO::writeOut(buffer, data_dir + "EpAy",EpAy,gSize,0);
        FileIO::writeOut(buffer, data_dir + "EpAx",EpAx,gSize,0);
        FileIO::writeOut(buffer, data_dir + "EpAz",EpAz,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "Phi",Phi,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "r",r,gSize,0);
        FileIO::writeOutDouble(buffer, data_dir + "x",x,xDim,0);
        FileIO::writeOutDouble(buffer, data_dir + "y",y,yDim,0);
        FileIO::writeOutDouble(buffer, data_dir + "z",z,zDim,0);
        FileIO::writeOutDouble(buffer, data_dir + "px",xp,xDim,0);
        FileIO::writeOutDouble(buffer, data_dir + "py",yp,yDim,0);
        FileIO::writeOutDouble(buffer, data_dir + "pz",zp,zDim,0);
        FileIO::writeOut(buffer, data_dir + "GK",GK,gSize,0);
        FileIO::writeOut(buffer, data_dir + "GV",GV,gSize,0);
        FileIO::writeOut(buffer, data_dir + "GpAx",GpAx,gSize,0);
        FileIO::writeOut(buffer, data_dir + "GpAy",GpAy,gSize,0);
        FileIO::writeOut(buffer, data_dir + "GpAz",GpAz,gSize,0);
    }

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    //std::cout << "wrote initial variables" << '\n';

    //free(V);
    free(K); free(r); free(Phi);

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    if (par.bval("read_wfc") == false){
        sum=sqrt(sum*dx*dy*dz);
        //#pragma omp parallel for reduction(+:sum) private(j)
        for (i = 0; i < xDim; i++){
            for (j = 0; j < yDim; j++){
                for (k = 0; k < zDim; k++){
                    index = i * yDim * zDim + j * zDim + k;
                    wfc[index].x = (wfc[index].x)/(sum);
                    wfc[index].y = (wfc[index].y)/(sum);
                }
            }
        }
    }

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    //std::cout << "xDim is: " << xDim << '\t' << "yDim is: " << yDim << '\n';
    //std::cout << "plan_2d is: " << plan_2d << '\n';
    result = hipfftPlan3d(&plan_3d, xDim, yDim, zDim, HIPFFT_Z2Z);
    //std::cout << "found result" << '\n';
    if(result != HIPFFT_SUCCESS){
        printf("Result:=%d\n",result);
        printf("Error: Could not execute hipfftPlan3d(%s, %d, %d, %d).\n", 
                "plan_3d",
                (unsigned int)xDim, (unsigned int)yDim, (unsigned int) zDim);
        return -1;
    }

    hipfftHandle plan_1d;
    generate_plan_other3d(&plan_1d, par, 0);
    hipfftHandle plan_dim2;
    generate_plan_other3d(&plan_dim2, par, 1);
    hipfftHandle plan_dim3;
    generate_plan_other3d(&plan_dim3, par, 2);

    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%//

    //std::cout << GV[0].x << '\t' << GK[0].x << '\t'
    //          << pAy[0] << '\t' << pAx[0] << '\n';

    //std::cout << "storing variables..." << '\n';

    // Storing variables that have been initialized
    // Re-establishing variables from parsed Grid class
    // Initializes uninitialized variables to 0 values
    par.store("omega", omega);
    par.store("gdt", gdt);
    par.store("dt", dt);
    par.store("omegaX", omegaX);
    par.store("omegaY", omegaY);
    par.store("omegaZ", omegaZ);
    par.store("dx", dx);
    par.store("dy", dy);
    par.store("dz", dz);
    par.store("xMax", xMax);
    par.store("yMax", yMax);
    par.store("zMax", zMax);
    par.store("winding", l);
    par.store("x", x);
    par.store("y", y);
    par.store("z", z);
    par.store("xp", xp);
    par.store("yp", yp);
    par.store("zp", zp);
    par.store("Energy", Energy);
    par.store("r", r);
    par.store("V", V);
    par.store("V_opt", V_opt);
    par.store("Phi", Phi);
    par.store("Phi_gpu", Phi_gpu);
    par.store("K", K);
    par.store("pAy", pAy);
    par.store("pAx", pAx);
    par.store("pAz", pAz);
    par.store("Energy_gpu", Energy_gpu);
    par.store("atoms", N);
    par.store("wfc", wfc);
    par.store("V_gpu", V_gpu);
    par.store("EV_opt", EV_opt);
    par.store("wfc_backup", wfc_backup);
    par.store("GK", GK);
    par.store("GV", GV);
    par.store("GpAx", GpAx);
    par.store("GpAy", GpAy);
    par.store("GpAz", GpAz);
    par.store("EV", EV);
    par.store("EK", EK);
    par.store("EpAy", EpAy);
    par.store("EpAx", EpAx);
    par.store("EpAz", EpAz);
    par.store("EappliedField", EappliedField);
    par.store("wfc_gpu", wfc_gpu);
    par.store("K_gpu", K_gpu);
    par.store("pAy_gpu", pAy_gpu);
    par.store("pAx_gpu", pAx_gpu);
    par.store("pAz_gpu", pAz_gpu);
    par.store("par_sum", par_sum);

    par.store("result", result);
    par.store("plan_1d", plan_1d);
    par.store("plan_3d", plan_3d);
    par.store("plan_dim2", plan_dim2);
    par.store("plan_dim3", plan_dim3);

    par.threads = threads;
    par.grid = grid;

    std::cout << "variables stored" << '\n';

    return 0;
}


int main(int argc, char **argv){

    Grid par = parseArgs(argc,argv);
    //Grid par2 = parseArgs(argc,argv);
    Wave wave;
    Op opr;

    int device = par.ival("device");
    int dimnum = par.ival("dimnum");
    hipSetDevice(device);

    std::string buffer;
    time_t start,fin;
    time(&start);
    printf("Start: %s\n", ctime(&start));

    //************************************************************//
    /*
    * Initialise the Params data structure to track params and variables
    */
    //************************************************************//

    // If we want to read in a wfc, we may also need to imprint a phase. This
    // will be done in the init_2d and init_3d functions
    // We need a number of parameters for now
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    int zDim = par.ival("zDim");
    if(par.bval("read_wfc") == true){

        // Initializing the wfc
        int gSize = xDim * yDim * zDim;
        hipfftDoubleComplex *wfc;
        //wfc = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gSize);

        std::string infile = par.sval("infile");
        std::string infilei = par.sval("infilei");
        printf("Loading wavefunction...");
        wfc=FileIO::readIn(infile,infilei,gSize);
        par.store("wfc",wfc);
        printf("Wavefunction loaded.\n");
        //std::string data_dir = par.sval("data_dir");
        //FileIO::writeOut(buffer, data_dir + "WFC_CHECK",wfc,gSize,0);
    }

    if (dimnum == 2){
        init(opr, par, wave);
    }
    else{
        //init_3d(opr, par2, wave);
        init(opr, par, wave);
    }

/*
    std::cout
    << par.ival("plan_3d") << par2.ival("plan_3d") << '\n'
    << par.ival("plan_1d") << par2.ival("plan_1d") << '\n'
    << par.ival("plan_2d") << par2.ival("plan_2d") << '\n'
    << par.ival("plan_other2d") << par2.ival("plan_other2d") << '\n'
    << par.ival("plan_dim1") << par2.ival("plan_dim1") << '\n'
    << par.ival("plan_dim2") << par2.ival("plan_dim2") << '\n';

    std::cout 
    << (par.cufftDoubleComplexval("GpAx") == par2.cufftDoubleComplexval("GpAx")) << '\n'
    << (par.cufftDoubleComplexval("GpAy") == par2.cufftDoubleComplexval("GpAy")) << '\n'
    << (par.cufftDoubleComplexval("GpAz") == par2.cufftDoubleComplexval("GpAz")) << '\n'
    << (par.dsval("V") == par2.dsval("V")) << '\n'
    << (par.dsval("K") == par2.dsval("K")) << '\n';
*/
    //std::cout << "initialized" << '\n';


    // Re-establishing variables from parsed Grid class
    // Note that 3d variables are set to nullptr's unless needed
    //      This might need to be fixed later
    std::string data_dir = par.sval("data_dir");
    double dx = par.dval("dx");
    double dy = par.dval("dy");
    double *x = par.dsval("x");
    double *y = par.dsval("y");
    double *V_opt = par.dsval("V_opt");
    double *pAy = par.dsval("pAy");
    double *pAx = par.dsval("pAx");
    double2 *pAy_gpu;
    double2 *pAx_gpu;
    double2 *pAz_gpu;
    double2 *V_gpu;
    double2 *K_gpu;
    int gsteps = par.ival("gsteps");
    int esteps = par.ival("esteps");
    hipfftDoubleComplex *wfc = par.cufftDoubleComplexval("wfc");
    hipfftDoubleComplex *GK = par.cufftDoubleComplexval("GK");
    hipfftDoubleComplex *GV = par.cufftDoubleComplexval("GV");
    hipfftDoubleComplex *GpAx = par.cufftDoubleComplexval("GpAx");
    hipfftDoubleComplex *GpAy = par.cufftDoubleComplexval("GpAy");
    hipfftDoubleComplex *GpAz = nullptr;
    hipfftDoubleComplex *EV = par.cufftDoubleComplexval("EV");
    hipfftDoubleComplex *EK = par.cufftDoubleComplexval("EK");
    hipfftDoubleComplex *EpAy = par.cufftDoubleComplexval("EpAy");
    hipfftDoubleComplex *EpAx = par.cufftDoubleComplexval("EpAx");
    hipfftDoubleComplex *EpAz = nullptr;
    hipfftDoubleComplex *wfc_gpu = par.cufftDoubleComplexval("wfc_gpu");
    hipfftDoubleComplex *par_sum = par.cufftDoubleComplexval("par_sum");
    hipError_t err;
    int gsize = xDim * yDim;

    // Special variables for the 3d case
    if (dimnum == 3){
        double dz = par.dval("dz");
        double *z = par.dsval("z");
        double *pAz = par.dsval("pAz");
        hipfftDoubleComplex *GpAz = par.cufftDoubleComplexval("GpAz");
        hipfftDoubleComplex *EpAz = par.cufftDoubleComplexval("EpAz");
        gsize = xDim*yDim*zDim;
    }
    hipMalloc((void**) &V_gpu, sizeof(double2)*gsize);
    hipMalloc((void**) &K_gpu, sizeof(double2)*gsize);
    hipMalloc((void**) &pAx_gpu, sizeof(double2)*gsize);
    hipMalloc((void**) &pAy_gpu, sizeof(double2)*gsize);
    hipMalloc((void**) &pAz_gpu, sizeof(double2)*gsize);

    std::cout << "variables re-established" << '\n';
    //std::cout << read_wfc << '\n';

    //************************************************************//
    /*
    * Groundstate finder section
    */
    //************************************************************//
    if (par.bval("write_file")){
        FileIO::writeOutParam(buffer, par, data_dir + "Params.dat");
    }

    if(gsteps > 0){
        err=hipMemcpy(K_gpu, GK, sizeof(hipfftDoubleComplex)*gsize,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy K_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(V_gpu, GV, sizeof(hipfftDoubleComplex)*gsize,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy V_gpu to device" << '\n';
            exit(1);
        }
        if (par.bval("write_file")){
            FileIO::writeOut(buffer, data_dir + "GK1",GK,gsize,0);
            FileIO::writeOut(buffer, data_dir + "GV1",GV,gsize,0);
        }
        err=hipMemcpy(pAy_gpu, GpAy, sizeof(hipfftDoubleComplex)*gsize,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy pAy_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(pAx_gpu, GpAx, sizeof(hipfftDoubleComplex)*gsize,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy pAx_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(wfc_gpu, wfc, sizeof(hipfftDoubleComplex)*gsize,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy wfc_gpu to device" << '\n';
            exit(1);
        }
        par.store("pAx", pAx);
        par.store("pAy", pAy);
        par.store("GK", GK);
        par.store("GV", GV);
        par.store("wfc", wfc);
        par.store("K_gpu", K_gpu);
        par.store("V_gpu", V_gpu);
        par.store("wfc_gpu", wfc_gpu);
        par.store("pAy_gpu", pAy_gpu);
        par.store("pAx_gpu", pAx_gpu);

        // Special cases for 3d
        if (dimnum == 3){
            GpAz = par.cufftDoubleComplexval("GpAz");
            err=hipMemcpy(pAz_gpu, GpAz, sizeof(hipfftDoubleComplex)*gsize,
                           hipMemcpyHostToDevice);

            if(err!=hipSuccess){
                std::cout << "ERROR: Could not copy pAz_gpu to device" << '\n';
                exit(1);
            }
            par.store("pAz_gpu", pAz_gpu);

            evolve_3d(wave, opr, par_sum,
                      gsteps, 0, par, buffer);
        }
        if (dimnum == 2){
            evolve_2d(wave, opr, par_sum,
                      gsteps,  0, par, buffer);
        }
        wfc = par.cufftDoubleComplexval("wfc");
        wfc_gpu = par.cufftDoubleComplexval("wfc_gpu");
        hipMemcpy(wfc, wfc_gpu, sizeof(hipfftDoubleComplex)*gsize,
                   hipMemcpyDeviceToHost);
    }

    std::cout << GV[0].x << '\t' << GK[0].x << '\t'
              << pAy[0] << '\t' << pAx[0] << '\n';

    std::cout << "evolution started..." << '\n';
    std::cout << "esteps: " << esteps << '\n';

    //************************************************************//
    /*
    * Evolution
    */
    //************************************************************//
    if(esteps > 0){
        err=hipMemcpy(pAy_gpu, EpAy, sizeof(hipfftDoubleComplex)*gsize,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy pAy_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(pAx_gpu, EpAx, sizeof(hipfftDoubleComplex)*gsize,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy pAx_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(K_gpu, EK, sizeof(hipfftDoubleComplex)*gsize,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy K_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(V_gpu, EV, sizeof(hipfftDoubleComplex)*gsize,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy V_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(wfc_gpu, wfc, sizeof(hipfftDoubleComplex)*gsize,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy wfc_gpu to device" << '\n';
            exit(1);
        }

        par.store("pAx", pAx);
        par.store("pAy", pAy);
        par.store("EK", EK);
        par.store("EV", EV);
        par.store("wfc", wfc);
        par.store("K_gpu", K_gpu);
        par.store("V_gpu", V_gpu);
        par.store("wfc_gpu", wfc_gpu);
        par.store("pAy_gpu", pAy_gpu);
        par.store("pAx_gpu", pAx_gpu);
        FileIO::writeOutDouble(buffer, data_dir + "V_opt",V_opt,gsize,0);
        // Special variables / instructions for 3d case
        if (dimnum == 3){
            pAz_gpu = par.cufftDoubleComplexval("pAz_gpu");
            EpAz = par.cufftDoubleComplexval("EpAz");
            err=hipMemcpy(pAz_gpu, EpAz, sizeof(hipfftDoubleComplex)*gsize,
                           hipMemcpyHostToDevice);
            if(err!=hipSuccess){
                std::cout << "ERROR: Could not copy pAz_gpu to device" << '\n';
                exit(1);
            }
            par.store("pAz_gpu", pAz_gpu);
            evolve_3d(wave, opr, par_sum,
                      esteps, 1, par, buffer);
        }
        if (dimnum == 2){
            evolve_2d(wave, opr, par_sum,
                      esteps, 1, par, buffer);
        }
        wfc = par.cufftDoubleComplexval("wfc");
        wfc_gpu = par.cufftDoubleComplexval("wfc_gpu");
    }
    std::cout << "done evolving" << '\n';
    free(EV); free(EK); free(EpAy); free(EpAx);
    free(x);free(y);
    hipFree(wfc_gpu); hipFree(K_gpu); hipFree(V_gpu); hipFree(pAx_gpu);
    hipFree(pAy_gpu); hipFree(par_sum);
    time(&fin);
    printf("Finish: %s\n", ctime(&fin));
    printf("Total time: %ld seconds\n ",(long)fin-start);
    std::cout << '\n';
    return 0;
}

