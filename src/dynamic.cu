#include "hip/hip_runtime.h"
#include <algorithm>
#include <limits>
#include <stack>

#include "../include/dynamic.h"

// Simple functions to subtract, add, multiply and divide
double subtract(double a, double b){
    return a-b;
}

double add(double a, double b){
    return a+b;
}

double multiply(double a, double b){
    return a*b;
}

double divide(double a, double b){
    return a/b;
}

double cos(double a, double b){
    return cos(a);
}

// We assume that we have already removed unnecessary spaces and such from 
// our eqn_string
EqnNode parse_eqn(Grid &par, std::string eqn_string){

    std::cout << eqn_string << '\n';

    // boolean value iff first minus
    bool minus = false;

    //std::cout << equation << '\n';

    // Because this will be called recursively, we need to return if the string
    // length is 0
    if (eqn_string.length() == 0){
        std::cout << "There's nothing here!" << '\n';
        exit(1);
        //return;
    }

    // vector of all possibe mathematical operators (not including functions)
    std::vector<std::string> moperators(4);
    moperators = {
        "-", "+", "/", "*"
    };

    // we need a map for these operators
    typedef double (*functionPtr_mop)(double, double);
    std::unordered_map<char, functionPtr_mop> moperator_map;
    moperator_map['-'] = subtract;
    moperator_map['+'] = add;
    moperator_map['*'] = multiply;
    moperator_map['/'] = divide;

    // And another vector for brackets of various types which indicate recursive
    // parsing of the equation
    std::vector<char> mbrackets;
    mbrackets = {
        '(', '[', ']', ')'
    };

    // vector of all possible mathematical functions... more to come
    std::vector<std::string> mfunctions(5);
    mfunctions = {
        "sin", "cos", "exp", "tan", "erf", "sqrt"
    };

    // We also need a specific map for the functions above
    typedef double (*functionPtr)(double, double);
    std::unordered_map<std::string, functionPtr> mfunctions_map;
    mfunctions_map["cos"] = cos;
/*
    mfunctions_map["sin"] = sin;
    mfunctions_map["cos"] = cos;
    mfunctions_map["tan"] = tan;
    mfunctions_map["exp"] = exp;
    mfunctions_map["erf"] = erf;
    mfunctions_map["sqrt"] = sqrt;
*/

    // first, we need to parse the equation string and remove parentheses
    // Then we'll sort according to the math operators (mops)
    int half = mbrackets.size() / 2;
    std::stack<int> open_bra;
    std::vector<int> ignored_positions;
    for (int i = 0; i < eqn_string.size(); ++i){
        for (int j = 0; j < mbrackets.size() / 2; ++j){
            if (eqn_string[i] == mbrackets[j]){
                open_bra.push(i);
            }
        }

        // Now we need to look for the closing bracket
        for (int j = mbrackets.size()/2; j < mbrackets.size(); ++j){
            if (eqn_string[i] == mbrackets[j]){
                ignored_positions.push_back(open_bra.top());
                ignored_positions.push_back(i);
                open_bra.pop();
            }
        }
    }


    // If parentheses cover the entire expression, we 
    //    1. Remove the parentheses
    //    2. subtract 1 from bra_positions
    std::string temp_string = eqn_string;
    if (ignored_positions.size() > 0){
        if (ignored_positions[ignored_positions.size()-1] 
                == temp_string.size() - 1 &&
            ignored_positions[ignored_positions.size()-2] == 0){
            ignored_positions.erase(ignored_positions.end()-1);
            ignored_positions.erase(ignored_positions.end()-1);

            eqn_string = eqn_string.substr(1, eqn_string.size() - 2);
    
            for (int i = 0; i < ignored_positions.size(); ++i){
                --ignored_positions[i];
                std::cout << ignored_positions[i] << '\n';
            }
        }
        temp_string = eqn_string;
    
        // Now we remove the parentheses from the eqn_string
        int offset = 0;
        std::vector<int> temp_positions = ignored_positions;
        for (int i = 0; i < temp_positions.size(); i += 2){
            temp_string.erase(temp_positions[i],
                              temp_positions[i+1] - temp_positions[i]+1);
            for (int j = i+2; j < temp_positions.size(); ++j){
                if (temp_positions[j] > temp_positions[i]){
                    temp_positions[j] 
                        -= temp_positions[i+1] - temp_positions[i] + 1;
                }
            }
            std::cout << temp_string << '\n';
        }
    }

    std::cout << "Done parsing equation" << '\n';

    // Creating the EqnNode
    EqnNode eqn_tree;

    bool only_nums = 
        (temp_string.find_first_not_of("0123456789") 
            == std::string::npos);
    if (only_nums){
        eqn_tree.val = atof(temp_string.c_str());
        return eqn_tree;
    }
    else if(temp_string.size() == 1){
        if(temp_string[0] == 'x'){
            eqn_tree.is_dynamic = true;
            eqn_tree.var = 'x';
            return eqn_tree;
        }
        else if(temp_string[0] == 'y'){
            eqn_tree.is_dynamic = true;
            eqn_tree.var = 'y';
            return eqn_tree;
        }
        else if(temp_string[0] == 'z'){
            eqn_tree.is_dynamic = true;
            eqn_tree.var = 'z';
            return eqn_tree;
        }
        else if(temp_string[0] == 't'){
            eqn_tree.is_dynamic = true;
            eqn_tree.var = 't';
            return eqn_tree;
        }
    }


    // We'll need to parse the equation string in reverse PEMDAS
    // So we go through the moperators, then mbrackets / mfunctions
    bool mop_found = false;
    int mop_point = 0;
    while (!mop_found){
        for (auto &mop : moperators){
            if (temp_string.find(mop) < temp_string.size()){
                mop_point = temp_string.find(mop);
                mop_found = true;
            } 
        }
        if (!mop_found){
            if(auto it = mfunctions_map.find(temp_string)
                    != mfunctions_map.end()){

                mop_point = temp_string.size()-1;
                eqn_tree.op = mfunctions_map[temp_string];
                eqn_tree.left = (EqnNode *)malloc(sizeof(EqnNode));
                std::cout << eqn_string.substr(mop_point+1, 
                                    eqn_string.size() - mop_point-1) << '\n';
                eqn_tree.left[0] = parse_eqn(par, 
                                    eqn_string.substr(mop_point+1, 
                                    eqn_string.size() - mop_point-1));

            }
            else{
                eqn_tree.val = par.dval(temp_string);
            }
            return eqn_tree;
                
        }
    }

    std::cout << mop_point << '\n';

    // Now we need to find the mop_point position in the eqn_string
    // We know the temp_string and how many positions we removed and where.
    if (ignored_positions.size() > 0){
        int count = 0;
        for (int i = 0; i <= mop_point; ++i){
            for (int j = 0; j < ignored_positions.size(); j += 2){
                if (ignored_positions[j] == i){
                    count += ignored_positions[j+1] - ignored_positions[j];
                }
            }
            count++;
        }

        mop_point = count;
    }

    std::cout << mop_point << '\n';
    std::cout << eqn_string[mop_point] << '\n';

    // Now we need to store the operator into the eqn_tree
    eqn_tree.op = moperator_map[eqn_string[mop_point]];

    // Now we need to parse the left and right banches...
    eqn_tree.left = (EqnNode *)malloc(sizeof(EqnNode));
    eqn_tree.left[0] = parse_eqn(par, eqn_string.substr(0, mop_point));

    eqn_tree.right = (EqnNode *)malloc(sizeof(EqnNode));
    eqn_tree.right[0] = parse_eqn(par, eqn_string.substr(mop_point+1, 
                                       eqn_string.size() - mop_point-1));

    return eqn_tree;
}

double evaluate_eqn(EqnNode eqn, double x, double y, double z, 
                    double time){

    if (eqn.op == NULL){
        if (eqn.is_dynamic){
            if(eqn.var == 'x'){
                return x;
            }
            if(eqn.var == 'y'){
                return y;
            }
            if(eqn.var == 'z'){
                return z;
            }
            if(eqn.var == 't'){
                return time;
            }
        }
        else{
            return eqn.val;
        }
    }

    double val1 = evaluate_eqn(eqn.left[0], x, y, z, time);
    double val2 = evaluate_eqn(eqn.right[0], x, y, z, time);
    return eqn.op(val1, val2);

}

void allocate_eqn(EqnNode *eqn, EqnNode *eqn_gpu){
}
