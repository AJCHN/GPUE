#include "hip/hip_runtime.h"
/*** kernels.cu - GPUE: Split Operator based GPU solver for Nonlinear 
Schrodinger Equation, Copyright (C) 2011-2015, Lee J. O'Riordan 
<loriordan@gmail.com>, Tadhg Morgan, Neil Crowley. 
All rights reserved.

Redistribution and use in source and binary forms, with or without 
modification, are permitted provided that the following conditions are 
met:

1. Redistributions of source code must retain the above copyright 
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright 
notice, this list of conditions and the following disclaimer in the 
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its 
contributors may be used to endorse or promote products derived from 
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT 
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR 
PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF 
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING 
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS 
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "../include/constants.h"
#include <stdio.h>


__constant__ double gDenConst = 6.6741e-40;//Evaluted in MATLAB: N*4*HBAR*HBAR*PI*(4.67e-9/mass)*sqrt(mass*(omegaZ)/(2*PI*HBAR))
//inline __device__ unsigned int getGid3d3d(){

__device__ unsigned int getGid3d3d(){
	return blockDim.x * ( ( blockDim.y * ( ( blockIdx.z * blockDim.z + threadIdx.z ) + blockIdx.y ) + threadIdx.y ) + blockIdx.x ) + threadIdx.x;
}

//inline __device__ unsigned int getBid3d3d(){
__device__ unsigned int getBid3d3d(){
	return blockIdx.x + gridDim.x*(blockIdx.y + gridDim.y * blockIdx.z);
}


//inline __device__ unsigned int getTid3d3d(){
__device__ unsigned int getTid3d3d(){
	return blockDim.x * ( blockDim.y * ( blockDim.z + ( threadIdx.z * blockDim.y ) )  + threadIdx.y )  + threadIdx.x;
}

__device__ double2 conjugate(double2 in){
	double2 result = in;
	result.y = -result.y;
	return result;
}

__device__ double2 realCompMult(double scalar, double2 comp){
	double2 result;
	result.x = scalar * comp.x;
	result.y = scalar * comp.y;
	return result;
}

//inline __device__ double complexMagnitude(double2 in){
__device__ double complexMagnitude(double2 in){
	return sqrt(in.x*in.x + in.y*in.y);
}

__host__ __device__ double complexMagnitudeSquared(double2 in){
	return in.x*in.x + in.y*in.y;
}

__host__ __device__ double2 complexMultiply(double2 in1, double2 in2){
	double2 result;
	result.x = (in1.x*in2.x - in1.y*in2.y);
	result.y = (in1.x*in2.y + in1.y*in2.x);
	return result;
}

/*
* Used to perform conj(in1)*in2; == < in1 | in2 >
*/
inline __device__ double2 braKetMult(double2 in1, double2 in2){
	return complexMultiply(conjugate(in1),in2);
}

/**
 * Performs complex multiplication of in1 and in2, giving result as out. 
 */
__global__ void cMult(double2* in1, double2* in2, double2* out){
	double2 result;
	unsigned int gid = getGid3d3d();
	result.x = (in1[gid].x*in2[gid].x - in1[gid].y*in2[gid].y);
	result.y = (in1[gid].x*in2[gid].y + in1[gid].y*in2[gid].x);
	out[gid] = result;
}

__global__ void cMultDensity(double2* in1, double2* in2, double2* out, double dt, double mass,double omegaZ, int gstate, int N){
	double2 result;
	double gDensity;
	int tid = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
	gDensity = gDenConst*complexMagnitudeSquared(in2[tid]); // scaling of interaction strength doesn't work now

	if(gstate == 0){
		double tmp = in1[tid].x*exp(-gDensity*(dt/HBAR) );
		result.x = (tmp)*in2[tid].x - (in1[tid].y)*in2[tid].y;
		result.y = (tmp)*in2[tid].y + (in1[tid].y)*in2[tid].x;
	}
	else{
		double2 tmp;
		tmp.x = in1[tid].x*cos(-gDensity*(dt/HBAR)) - in1[tid].y*sin(-gDensity*(dt/HBAR));
		tmp.y = in1[tid].y*cos(-gDensity*(dt/HBAR)) + in1[tid].x*sin(-gDensity*(dt/HBAR));
		
		result.x = (tmp.x)*in2[tid].x - (tmp.y)*in2[tid].y;
		result.y = (tmp.x)*in2[tid].y + (tmp.y)*in2[tid].x;
	}
	out[tid] = result;
}

/**
 * Divides both components of vector type "in", by the value "factor".
 * Results given with "out"
 */
__global__ void scalarDiv(double2* in, double factor, double2* out){
	double2 result;
	//extern __shared__ double2 tmp_in[];
	unsigned int gid = getGid3d3d();
	result.x = (in[gid].x*factor);
	result.y = (in[gid].y*factor);
	out[gid] = result;
}

/**
 * As above, but normalises for wfc
 */
__global__ void scalarDiv_wfcNorm(double2* in, double dr, double2* pSum, double2* out){
	unsigned int gid = getGid3d3d();
	double2 result;
	double norm = sqrt((pSum[0].x + pSum[0].y)*dr);
	result.x = (in[gid].x/norm);
	result.y = (in[gid].y/norm);
	out[gid] = result;
}

/**
 */
__global__ void angularOp(double omega, double dt, double2* wfc, double* xpyypx, double2* out){
	unsigned int gid = getGid3d3d();
	double2 result;
	double op;
	op = exp( -omega*xpyypx[gid]*dt);
	result.x=wfc[gid].x*op;
	result.y=wfc[gid].y*op;
	out[gid]=result;
}

/**
 * Routine for parallel summation. Can be looped over from host.
 */
__global__ void multipass(double2* input, double2* output, int pass){
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x;// printf("bid0=%d\n",bid);
	unsigned int gid = getGid3d3d();
	extern __shared__ double2 sdata[];
	sdata[tid] = input[gid];
	if(pass == 0){
		sdata[tid].x *= sdata[tid].x;
		sdata[tid].y *= sdata[tid].y;	
	}
	__syncthreads();
	for(int i = blockDim.x>>1; i > 0; i>>=1){
		if(tid < blockDim.x>>1){
			sdata[tid].x += sdata[tid + i].x;
			sdata[tid].y += sdata[tid + i].y;
		}
		__syncthreads();
	}
	if(tid==0){
		output[bid] = sdata[0];
	}
}


/*
* Calculates all of the energy of the current state. sqrt_omegaz_mass = sqrt(omegaZ/mass), part of the nonlin interaction term
*/
__global__ void energyCalc(double2 *wfc, double2 *op, double dt, double2 *energy, int gnd_state, int op_space, double sqrt_omegaz_mass){
	unsigned int gid = getGid3d3d();
	double hbar_dt = HBAR/dt;
	double g_local = 0.0;
	double2 result;
	double opLocal;
	if(op_space)
		g_local = gDenConst*sqrt_omegaz_mass*complexMagnitudeSquared(wfc[gid]);
	if(!gnd_state){
		opLocal = -log(op[gid].x + g_local)*hbar_dt;
	}
	else{
		opLocal = cos(op[gid].x + g_local)*hbar_dt;
	}
	result = braKetMult(wfc[gid], realCompMult(opLocal,wfc[gid]));
	//printf("oplocal=%e	Resx=%e	Resy=%e\n",opLocal,result.x,result.y);
	energy[gid].x += result.x;
	energy[gid].y += result.y;
}


//#############################################################################################
//#############################################################################################

/**
 * Routine for parallel summation. Can be looped over from host.
 */
template<typename T> __global__ void pSumT(T* in1, T* output, int pass){
        unsigned int tid = threadIdx.x;
        unsigned int bid = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x;// printf("bid0=%d\n",bid);
        unsigned int gid = getGid3d3d();
        extern __shared__ T sdata[];
        for(int i = blockDim.x>>1; i > 0; i>>=1){
                if(tid < blockDim.x>>1){
                        sdata[tid] += sdata[tid + i];
                }
                __syncthreads();
        }
        if(tid==0){
                output[bid] = sdata[0];
        }
}

/**
 * Routine for parallel summation. Can be looped over from host.
 */
__global__ void pSum(double* in1, double* output, int pass){
        unsigned int tid = threadIdx.x;
        unsigned int bid = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x;// printf("bid0=%d\n",bid);
        unsigned int gid = getGid3d3d();
        extern __shared__ double sdata2[];
        for(int i = blockDim.x>>1; i > 0; i>>=1){
                if(tid < blockDim.x>>1){
                        sdata2[tid] += sdata2[tid + i];
                }
                __syncthreads();
        }
        if(tid==0){
                output[bid] = sdata2[0];
        }
}



//#############################################################################################
//#############################################################################################
