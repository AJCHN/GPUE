/*
* unit_test.cc - GPUE: Split Operator based GPU solver for Nonlinear 
Schrodinger Equation, Copyright (C) 2011-2015, Lee J. O'Riordan 
<loriordan@gmail.com>, Tadhg Morgan, Neil Crowley. All rights reserved.

Redistribution and use in source and binary forms, with or without 
modification, are permitted provided that the following conditions are 
met:

1. Redistributions of source code must retain the above copyright 
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright 
notice, this list of conditions and the following disclaimer in the 
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its 
contributors may be used to endorse or promote products derived from 
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT 
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR 
PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF 
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING 
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS 
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "../include/ds.h"
#include "../include/unit_test.h"
#include "../include/parser.h"
#include <string>
#include <assert.h>
#include <hipfft/hipfft.h>
#include <vector>

// Test for the Grid structure with paramters in it 
void parameter_test();

// Test for the parsing function
void parser_test();

// Kernel testing will be added later

/*----------------------------------------------------------------------------//
* MAIN
*-----------------------------------------------------------------------------*/

void test_all(){
    std::cout << "Starting unit tests..." << '\n';
    parameter_test();
    parser_test();
    std::cout << "All tests completed. GPUE passed." << '\n';
}

// Test for the Grid structure with paramters in it
// Initialize all necessary variables and read them back out
void parameter_test(){
    // For this test, we simply need to read in and out stuff from each 
    // class and structure in ds.h / ds.cc
    
    // Certain variables will be used multiple times. 
    double *dstar_var;
    dstar_var = (double *)malloc(sizeof(double) * 5);
    hipfftDoubleComplex *cdc_var;
    cdc_var = (hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex) * 5);
    for (int i = 0; i < 5; ++i){
        dstar_var[i] = (double)i * 0.5;
        cdc_var[i].x = (double)i * 0.5;
        cdc_var[i].y = (double)i * 0.5;
    }

    double dvar = 1.05;
    int ivar = 5;

    // Now testing the Grid class
    Grid grid_test;
    grid_test.store("dstar_var",dstar_var);
    grid_test.store("dvar", dvar);
    grid_test.store("ivar", ivar);

    assert(dstar_var == grid_test.dsval("dstar_var"));
    assert(dvar == grid_test.dval("dvar"));
    assert(ivar == grid_test.ival("ivar"));

    std::cout << "Grid class checked, now checking the Cuda class..." << '\n';

    // Now checking the Cuda class
    // This one will require creating a list of variables...
    hipError_t err = hipSuccess;
    hipfftHandle plan_1d = 4, plan_2d = 6;
    hipStream_t streamA = 0, streamB = 0, streamC = 0, streamD = 0;
    hipfftResult result = HIPFFT_SUCCESS;
    dim3 grid;

    grid.x = 1; grid.y = 2; grid.z = 3;

    // Creating Cuda class to test with
    Cuda cuda_test;

    // Testing the store and value functions
    cuda_test.store("err", err);
    cuda_test.store("result", result);
    cuda_test.store("plan_1d", plan_1d);
    cuda_test.store("plan_2d", plan_2d);
    cuda_test.store("streamA", streamA);
    cuda_test.store("streamB", streamB);
    cuda_test.store("streamC", streamC);
    cuda_test.store("streamD", streamD);
    cuda_test.store("grid", grid);

    assert(err == cuda_test.cudaError_tval("err"));
    assert(result == cuda_test.cufftResultval("result"));
    assert(plan_1d == cuda_test.cufftHandleval("plan_1d"));
    assert(plan_2d == cuda_test.cufftHandleval("plan_2d"));
    assert(streamA == cuda_test.cudaStream_tval("streamA"));
    assert(streamB == cuda_test.cudaStream_tval("streamB"));
    assert(streamC == cuda_test.cudaStream_tval("streamC"));
    assert(streamD == cuda_test.cudaStream_tval("streamD"));
    assert(grid.x == cuda_test.dim3val("grid").x);
    assert(grid.y== cuda_test.dim3val("grid").y);
    assert(grid.y == cuda_test.dim3val("grid").y);

    std::cout << "Cuda class checked, now checking Op class..." << '\n';

    // Now checking the Op class
    // Creating Op class to test with
    Op op_test;
    op_test.store("dstar_var",dstar_var);
    op_test.store("cdc_var",cdc_var);

    assert(dstar_var == op_test.dsval("dstar_var"));
    assert(cdc_var == op_test.cufftDoubleComplexval("cdc_var"));

    std::cout << "Op class checked, now checking Wave class..." << '\n';

    // Now checking the Op class
    // Creating Op class to test with
    Wave wave_test; 
    wave_test.store("dstar_var",dstar_var);
    wave_test.store("cdc_var",cdc_var);

    assert(dstar_var == wave_test.dsval("dstar_var"));
    assert(cdc_var == wave_test.cufftDoubleComplexval("cdc_var"));

    std::cout << "All data structures checked" << '\n';

}

// Test for the parsing function
void parser_test(){

    // Testing the command-line parser with defaults and with modifications
    std::cout << "Testing command-line parser with no arguments..." << '\n';

    // First testing default values in and out of the parser function
    char **fake_noargv;
    fake_noargv = (char **)malloc(sizeof(char) * 1);
    Grid noarg_grid;
    noarg_grid = parseArgs(0,fake_noargv);

    // Checking contents of noarg_grid:
    assert(noarg_grid.ival("xDim") == 256);
    assert(noarg_grid.ival("yDim") == 256);
    assert(noarg_grid.ival("zDim") == 256);
    assert(noarg_grid.dval("omega") == 0);
    assert(noarg_grid.dval("gammaY") == 1.0);
    assert(noarg_grid.dval("gsteps") == 1e4);
    assert(noarg_grid.dval("esteps") == 1000);
    assert(noarg_grid.dval("gdt") == 1e-4);
    assert(noarg_grid.dval("dt") == 1e-4);
    assert(noarg_grid.ival("device") == 0);
    assert(noarg_grid.ival("atoms") == 1);
    assert(noarg_grid.ival("read_wfc") == 0);
    assert(noarg_grid.ival("printSteps") == 100);
    assert(noarg_grid.dval("winding") == 0);
    assert(noarg_grid.ival("corotating") == 0);
    assert(noarg_grid.ival("gpe") == 0);
    assert(noarg_grid.dval("omegaZ") == 0);
    assert(noarg_grid.dval("int_scaling") == 0);
    assert(noarg_grid.dval("laser_power") == 0);
    assert(noarg_grid.dval("angle_sweep") == 0);
    assert(noarg_grid.ival("kick_it") == 0);
    assert(noarg_grid.ival("write_it") == 1);
    assert(noarg_grid.dval("x0_shift") == 0);
    assert(noarg_grid.dval("y0_shift") == 0);
    assert(noarg_grid.dval("sepMinEpsilon") == 0);
    assert(noarg_grid.ival("graph") == 0);

    // Now testing all values specified by command-line arguments
    std::cout << "Testing command-line parser with all arguments..." << '\n';

    std::string cmdline;
    cmdline = "./gpue -a 0 -d 0 -e 1000 -G 1 -g 1e4 -i 0 -k 0 -L 0 -l 0 -n 1 -O 0 -o 0 -P 0 -p 100 -r 0 -S 0 -s 0 -T 1e-4 -t 1e-4 -U 0 -V 0 -W 1 -w 0 -X 6.283 -x 256 -Y 6.283 -y 256";

    // Fake argc is number of arguments above
    int fake_argc = 55;

    // Vector for the arguments for easier parsing
    std::vector<std::string> arguments(fake_argc);

    // Parsing the cmdline argument into vector
    int count = 0;
    for (size_t i = 0; i < cmdline.size(); ++i){
        if (cmdline.at(i) != ' '){
            arguments[count] += cmdline.at(i);
        }
        else{
            count++;
        }
    }

    char **fake_fullargv;
    fake_fullargv = (char **)malloc(fake_argc * sizeof(char *));
    for (int i = 0; i < fake_argc; ++i){
        fake_fullargv[i] = (char *)malloc(arguments[i].size() * sizeof(char));
        for (size_t j = 0; j < arguments[i].size(); ++j){
            fake_fullargv[i][j] = arguments[i].at(j);
        }
    }

    // Now to read into gpue and see what happens
    Grid fullarg_grid;
    fullarg_grid = parseArgs(fake_argc, fake_fullargv);

    // Checking contents of fullarg_grid:
    assert(fullarg_grid.ival("xDim") == 256);
    assert(fullarg_grid.ival("yDim") == 256);
    assert(fullarg_grid.ival("zDim") == 256);
    assert(fullarg_grid.dval("omega") == 0);
    assert(fullarg_grid.dval("gammaY") == 1.0);
    assert(fullarg_grid.dval("gsteps") == 1e4);
    assert(fullarg_grid.dval("esteps") == 1000);
    assert(fullarg_grid.dval("gdt") == 1e-4);
    assert(fullarg_grid.dval("dt") == 1e-4);
    assert(fullarg_grid.ival("device") == 0);
    assert(fullarg_grid.ival("atoms") == 1);
    assert(fullarg_grid.ival("read_wfc") == 0);
    assert(fullarg_grid.ival("printSteps") == 100);
    assert(fullarg_grid.dval("winding") == 0);
    assert(fullarg_grid.ival("corotating") == 0);
    assert(fullarg_grid.ival("gpe") == 0);
    assert(fullarg_grid.dval("omegaZ") == 0);
    assert(fullarg_grid.dval("int_scaling") == 0);
    assert(fullarg_grid.dval("laser_power") == 0);
    assert(fullarg_grid.dval("angle_sweep") == 0);
    assert(fullarg_grid.ival("kick_it") == 0);
    assert(fullarg_grid.ival("write_it") == 1);
    assert(fullarg_grid.dval("x0_shift") == 0);
    assert(fullarg_grid.dval("y0_shift") == 0);
    assert(fullarg_grid.dval("sepMinEpsilon") == 0);
    assert(fullarg_grid.ival("graph") == 0);
    assert(fullarg_grid.dval("omegaY") == 6.283);
    assert(fullarg_grid.dval("omegaX") == 6.283);

    std::cout << "All arguments parsed" << '\n';

}
