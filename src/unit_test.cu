#include "hip/hip_runtime.h"
#include "../include/ds.h"
#include "../include/unit_test.h"
#include "../include/parser.h"
#include "../include/evolution.h"
#include "../include/init.h"
#include "../include/dynamic.h"
#include "../include/vortex_3d.h"
#include <string.h>
#include <assert.h>
#include <hipfft/hipfft.h>
#include <vector>
#include <fstream>

// Adding tests for mathematical operator kernels
void math_operator_test();
__global__ void add_test(double2 *a, double2 *b, double2 *c);
__global__ void subtract_test(double2 *a, double2 *b, double2 *c);
__global__ void mult_test(double2 *a, double2 *b, double2 *c);
__global__ void mult_test(double2 *a, double b, double2 *c);
__global__ void pow_test(double2 *a, int b, double2 *c);

// Tests for hipfftDoubleComplex functions
void cufftDoubleComplex_functions_test();
__global__ void complexMag_test(double2 *in, double *out);
__global__ void complexMag2_test(double2 *in, double *out);
void realCompMult_test();
void cMult_test();

// Tests for quantum operations
__global__ void make_complex_kernel(double *in, int *evolution_type, 
                                    double2 *out);
void make_complex_test();
void cMultPhi_test();
void cMultDens_test();

// Tests for complex mathematical operations
void vecMult_test();
void scalarDiv_test();
void vecConj_test();

// AST tests
void ast_mult_test();
void ast_cmult_test();
void ast_op_mult_test();
void real_ast_test();
void im_ast_test();

// Other
void energyCalc_test();
void braKetMult_test();

// Test for the Grid structure with parameters in it 
void parameter_test();

// Test for the parsing function
void parser_test();

// Testing the evolve_2d function in evolution.cu
void evolve_2d_test();

// Testing the parSum function
void parSum_test();

// Simple test of grid / cuda stuff
void grid_test2d();
void grid_test3d();

// Test of 1D fft's along all 3d grids
void fft_test();

// Test to check the equation parser for dynamic fields
void dynamic_test();

// Test to make sure the kernel for the polynomial approx. of Bessel fxns works
void bessel_test();

// Test for the vortex tracking functions in vortex_3d
void vortex3d_test();

// Kernel testing will be added later
__device__ bool close(double a, double b, double threshold){
    return (abs(a-b) < threshold);
}


/*----------------------------------------------------------------------------//
* MAIN
*-----------------------------------------------------------------------------*/

void test_all(){
    std::cout << "Starting unit tests..." << '\n';
    parameter_test();

    std::cout 
        << "Beginning testing of standard mathematical operation kernels...\n";
    math_operator_test();

    std::cout << "Beginning testing of hipfftDoubleComplex kernels...\n";
    cufftDoubleComplex_functions_test();

    // Do not uncomment these 2
    //parser_test();
    //evolve_2d_test();

    grid_test2d();
    grid_test3d();
    parSum_test();
    fft_test();
    dynamic_test();
    bessel_test();
    //vortex3d_test();
    make_complex_test();
    cMultPhi_test();
    cMultDens_test();

    std::cout << "All tests completed. GPUE passed." << '\n';
}

void math_operator_test(){

    // First, we need to create a set of grids and threads to read into the 
    // kernels for testing
    dim3 grid = {1,1,1};
    dim3 threads = {1,1,1};

    double2 *ha, *hb, *hc;
    double2 *da, *db, *dc;
    
    // Allocating single-element arrays to test kernels with. 
    ha = (double2*)malloc(sizeof(double2));
    hb = (double2*)malloc(sizeof(double2));
    hc = (double2*)malloc(sizeof(double2));

    ha[0].x = 0.01;
    ha[0].y = 0.1;
    hb[0].x = 0.02;
    hb[0].y = 0.2;

    hipMalloc((void**) &da, sizeof(double2));
    hipMalloc((void**) &db, sizeof(double2));
    hipMalloc((void**) &dc, sizeof(double2));

    hipMemcpy(da, ha, sizeof(double2), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, sizeof(double2), hipMemcpyHostToDevice);

    add_test<<<grid, threads>>>(da, db, dc);
    hipMemcpy(hc, dc, sizeof(double2), hipMemcpyDeviceToHost);

    if (abs(hc[0].x - 0.03) > 1e-16 || abs(hc[0].y - 0.3) > 1e-16){
        std::cout << "Complex addition test failed!\n";
        exit(1);
    }

    subtract_test<<<grid, threads>>>(da, db, dc);
    hipMemcpy(hc, dc, sizeof(double2), hipMemcpyDeviceToHost);

    if (hc[0].x != -0.01 || hc[0].y != -0.1){
        std::cout << "Complex subtraction test failed!\n";
        exit(1);
    }

    pow_test<<<grid, threads>>>(da, 3, dc);
    hipMemcpy(hc, dc, sizeof(double2), hipMemcpyDeviceToHost);

    if (abs(hc[0].x + 0.000299) > 1e-16 || abs(hc[0].y + 0.00097) > 1e-16){
        std::cout << "Complex power test failed!\n";
        exit(1);
    }

    mult_test<<<grid, threads>>>(da, db, dc);
    hipMemcpy(hc, dc, sizeof(double2), hipMemcpyDeviceToHost);

    if (abs(hc[0].x + 0.0198) > 1e-16 || abs(hc[0].y - 0.004) > 1e-16){
        std::cout << "Complex multiplication test failed!\n";
        exit(1);
    }

    mult_test<<<grid, threads>>>(da, 3.0, dc);
    hipMemcpy(hc, dc, sizeof(double2), hipMemcpyDeviceToHost);

    if (abs(hc[0].x - 0.03) > 1e-16 || abs(hc[0].y - 0.3) > 1e-16){
        std::cout << "Complex multiplication test with real number failed!\n";
        exit(1);
    }

    std::cout << "Complex addition, subtraction, multiplication, and powers have been tested\n";
}

__global__ void add_test(double2 *a, double2 *b, double2 *c){
    c[0] = add(a[0],b[0]);
}

__global__ void subtract_test(double2 *a, double2 *b, double2 *c){
    c[0] = subtract(a[0],b[0]);
}

__global__ void pow_test(double2 *a, int b, double2 *c){
    c[0] = pow(a[0],b);
}

__global__ void mult_test(double2 *a, double2 *b, double2 *c){
    c[0] = mult(a[0],b[0]);
}

__global__ void mult_test(double2 *a, double b, double2 *c){
    c[0] = mult(a[0],b);
}

void cufftDoubleComplex_functions_test(){

    // first creating the grid and threads
    dim3 grid = {1,1,1};
    dim3 threads = {1,1,1};

    double *hval_double, *dval_double, *dout, *hout;
    double2 *hval_double2, *dval_double2;

    double2 *hin, *din;

    hval_double = (double*)malloc(sizeof(double));
    hval_double2 = (double2*)malloc(sizeof(double2));
    hout = (double*)malloc(sizeof(double));
    hin = (double2*)malloc(sizeof(double2));

    hval_double[0] = 3.0;
    hval_double2[0].x = 0.3;
    hval_double2[0].y = 0.4;

    hin[0].x = 3.0;
    hin[0].y = 4.0;

    hipMalloc((void**)&dval_double, sizeof(double));
    hipMalloc((void**)&dval_double2, sizeof(double2));
    hipMalloc((void**)&dout, sizeof(double));
    hipMalloc((void**)&din, sizeof(double2));


    // Testing make_cufftDoubleComplex function
    hipMemcpy(dval_double, hval_double, sizeof(double), 
               hipMemcpyHostToDevice);

    make_cufftDoubleComplex<<<grid, threads>>>(dval_double, dval_double2);

    hipMemcpy(hval_double2, dval_double2, sizeof(double2),
               hipMemcpyDeviceToHost);

    if (hval_double2[0].x != 3.0 || hval_double2[0].y != 0){
        std::cout << "Test of make_cufftDoubleComplex failed!\n";
        exit(1);
    }

    // testing device complexMagnitude function
    hipMemcpy(din, hin, sizeof(double2), hipMemcpyHostToDevice);
    complexMag_test<<<grid, threads>>>(din, dout);

    hipMemcpy(hout, dout, sizeof(double), hipMemcpyDeviceToHost);

    if (hout[0] != 5.0){
        std::cout << hout[0] << '\n';
        std::cout << "Test of device complexMagnitude failed!\n";
        exit(1);
    }

    // Testing global complexMagnitude function
    complexMagnitude<<<grid, threads>>>(din, dout);
    hipMemcpy(hout, dout, sizeof(double), hipMemcpyDeviceToHost);

    if (hout[0] != 5.0){
        std::cout << hout[0] << '\n';
        std::cout << "Test of global complexMagnitude failed!\n";
        exit(1);
    }

    complexMag2_test<<<grid, threads>>>(din, dout);

    hipMemcpy(hout, dout, sizeof(double), hipMemcpyDeviceToHost);

    if (hout[0] != 25.0){
        std::cout << hout[0] << '\n';
        std::cout << "Test of device complexMagnitudeSquared failed!\n";
        exit(1);
    }

    // Testing global complexMagnitude function
    complexMagnitudeSquared<<<grid, threads>>>(din, dout);
    hipMemcpy(hout, dout, sizeof(double), hipMemcpyDeviceToHost);

    if (hout[0] != 25.0){
        std::cout << hout[0] << '\n';
        std::cout << "Test of global complexMagnitudeSquared failed!\n";
        exit(1);
    }


    std::cout << "make_cufftDoubleComplex, and complexMagnitude[Squared] have been tested\n";

}

__global__ void complexMag_test(double2 *in, double *out){
    out[0] = complexMagnitude(in[0]);
}

__global__ void complexMag2_test(double2 *in, double *out){
    out[0] = complexMagnitudeSquared(in[0]);
}

// Test to check the equation parser for dynamic fields
// For this test, we will need a general set of parameters to read in and a
// standard equation string to look at. 
void dynamic_test(){

    std::cout << "Beginning test of dynamic functions..." <<'\n';
    std::string eqn_string = "(((3*x)+7)+(5-7)+cos(0))+pow(120,2)";

    Grid par;
    par.store("x",5);
    std::string val_string = "check_var";

    EqnNode eqn_tree = parse_eqn(par, eqn_string, val_string);

    std::cout << "finding the number of elements in abstract syntax tree...\n";

    int num = 0;
    find_element_num(eqn_tree, num);
    int element_num = num;

    std::cout << "Total number of elements is: " << num << '\n';

    std::cout << "Now to copy the tree to the GPU..." << '\n';

    EqnNode_gpu *eqn_gpu, *eqn_cpu;
    eqn_cpu = (EqnNode_gpu *)malloc(sizeof(EqnNode_gpu)*element_num);
    num = 0;
    tree_to_array(eqn_tree, eqn_cpu, num);

/*
    for (int i = 0; i < num; ++i){
        std::cout << eqn_cpu[i].val << '\n';
        std::cout << eqn_cpu[i].left << '\n';
        std::cout << eqn_cpu[i].right << '\n' << '\n';
    }
*/

    hipMalloc((void**)&eqn_gpu, sizeof(EqnNode_gpu)*element_num);
    hipMemcpy(eqn_gpu, eqn_cpu, sizeof(EqnNode_gpu)*element_num,
               hipMemcpyHostToDevice);

    // Now to check some simple evaluation
    std::cout << "Now to check simple GPU evaluation..." << '\n';
    int n = 64;
    double *array, *array_gpu;
    array = (double *)malloc(sizeof(double)*n);
    hipMalloc(&array_gpu, sizeof(double)*n);

    int threads = 64;
    int grid = (int)ceil((float)n/threads);

    //zeros<<<grid, threads>>>(array_gpu, n);
    find_field<<<grid, threads>>>(array_gpu, 1, 0.0, 0.0, 0.0, eqn_gpu);

    hipMemcpy(array, array_gpu, sizeof(double)*n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i){
        std::cout << array[i] << '\n';
    }

    // Now testing simple parsing of example "example.cfg"
    std::cout << "Testing simple parameter parsing." << '\n';
    par.store("param_file", (std::string)"src/example.cfg");
    parse_param_file(par);

    std::cout << "Dynamic tests passed" <<'\n';
}

__global__ void bessel_test_kernel(double *j, double *j_poly, bool *val){
    int xid = blockDim.x*blockIdx.x + threadIdx.x;
    j[xid] = j0(xid * 2.0 / 128);
    j_poly[xid] = poly_j(0,xid * 2.0 / 128, 40);

    if (!close(j[xid],j_poly[xid], 0.0001)){
        val[0] = false;
        printf("Error at element %u in Bessel test!\tValues: %f, %f\n", 
               xid, j[xid], j_poly[xid]);
    }
}

// Test for bessel functions
void bessel_test(){

    std::cout << "Testing Bessel Functions..." << '\n';

    double *j_gpu, *j_poly_gpu;
    bool *val, *val_gpu;
    int n = 128;
    hipMalloc((void **)&j_gpu, sizeof(double)*n);
    hipMalloc((void **)&j_poly_gpu, sizeof(double)*n);

    hipMalloc((void **)&val_gpu, sizeof(bool));
    val = (bool *)malloc(sizeof(bool));
    val[0] = true;
    hipMemcpy(val_gpu, val, sizeof(bool), hipMemcpyHostToDevice);

    bessel_test_kernel<<<64,2>>>(j_gpu, j_poly_gpu, val_gpu);
    hipMemcpy(val, val_gpu, sizeof(bool), hipMemcpyDeviceToHost);

    if(val[0]){
        std::cout << "Bessel Test Passed!" << '\n';
    }
    else{
        std::cout << "Bessel Test Failed!" << '\n';
        exit(1);
    }

}

// Test of 1D fft's along all 3d grids
// In particular, we need to test the generate_plan_other3d function
// These will be checked against 1d 
void fft_test(){

    // For these tests, we are assuming that the x, y and z dimensions are 
    // All the same (2x2x2)
    // Note that yDim needs to be singled out differently, but z/x need no loops

    // now we need to create the necessary parameters and store everything
    int xDim = 2;
    int yDim = 2;
    int zDim = 2;
    int gsize = xDim * yDim * zDim;

    Grid par;
    par.store("xDim", xDim);
    par.store("yDim", yDim);
    par.store("zDim", zDim);

    hipfftHandle plan_x, plan_y, plan_z;
    // Now creating the plans
    generate_plan_other3d(&plan_x, par, 0);
    generate_plan_other3d(&plan_y, par, 1);
    generate_plan_other3d(&plan_z, par, 2);

    // And the result / error
    hipError_t err;
    hipfftResult result;

    // Creating the initial array for the x dimension fft
    double2 *array, *gpu_array;
    array = (double2 *) malloc(sizeof(double2)*gsize);
    hipMalloc((void**) &gpu_array, sizeof(double2)*gsize);
    for (int i = 0; i < gsize; i++){
        array[i].x = 1;
        array[i].y = 0;
    }

    // transferring to gpu
    err = hipMemcpy(gpu_array, array, sizeof(double2)*gsize,
                     hipMemcpyHostToDevice);
    if (err != hipSuccess){
        std::cout << "Could not coppy array to device!" << '\n';
        std::cout << "error code: " << err << '\n';
        exit(1);
    }

    // Performing the x transformation
    for (int i = 0; i < yDim; i++){
        result = hipfftExecZ2Z(plan_y, &gpu_array[i*xDim*yDim], 
                                      &gpu_array[i*xDim*yDim], HIPFFT_FORWARD);
    }
    //result = hipfftExecZ2Z(plan_z, gpu_array, gpu_array, HIPFFT_FORWARD);

    // transferring back to host to check output
    err = hipMemcpy(array, gpu_array, sizeof(double2)*gsize, 
                     hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cout << "Could not coppy gpu_array to host!" << '\n';
        std::cout << "error code: " << err << '\n';
        exit(1);
    }

    for (int i = 0; i < gsize; i++){
        std::cout << array[i].x << '\t' << array[i].y << '\n';
    }

    // Now to try the inverse direction

    for (int i = 0; i < yDim; i++){
        result = hipfftExecZ2Z(plan_y, &gpu_array[i*xDim*yDim], 
                                      &gpu_array[i*xDim*yDim], HIPFFT_BACKWARD);
    }
    //result = hipfftExecZ2Z(plan_z, gpu_array, gpu_array, HIPFFT_BACKWARD);

    // copying back
    err = hipMemcpy(array, gpu_array, sizeof(double2)*gsize, 
                     hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        std::cout << "Could not coppy gpu_array to host!" << '\n';
        std::cout << "error code: " << err << '\n';
        exit(1);
    }

    for (int i = 0; i < gsize; i++){
        std::cout << array[i].x << '\t' << array[i].y << '\n';
    }



}

// Simple test of CUDA grid stuff
void grid_test2d(){

    std::cout << "testing grid / threads and stuff" << '\n';

    int max_threads = 128;

    int xDim = 1024;
    int yDim = 1024;
    int zDim = 1;

    int xD = 1, yD = 1, zD = 1;

    int gsize = xDim * yDim;

    // Now to set up the CUDA grid / threads
    dim3 block;
    dim3 grid;

    if (xDim <= max_threads){
        block.x = xDim;
        block.y = 1;
        block.z = 1;

        xD = 1;
        yD = yDim;
        zD = 1;
    } 
    else{
        int count = 0;
        int dim_tmp = xDim;
        while (dim_tmp > max_threads){
            count++;
            dim_tmp /= 2;
        }

        std::cout << "count is: " << count << '\n';

        block.x = dim_tmp;
        block.y = 1;
        block.z = 1;
        xD = pow(2,count);
        yD = yDim;
        zD = 1;
    }

    std::cout << "threads in x are: " << block.x << '\n';
    std::cout << "dimensions are: " << xD << '\t' << yD << '\t' << zD << '\n';

    grid.x=xD; 
    grid.y=yD; 
    grid.z=zD; 

    int total_threads = block.x * block.y * block.z;

    // Now we need to initialize our double * and send it to the gpu
    double *host_array, *device_array;
    host_array = (double *) malloc(sizeof(double)*gsize);
    hipMalloc((void**) &device_array, sizeof(double)*gsize);

    // initializing 2d array
    for (int i = 0; i < gsize; i++){
        host_array[i] = -1;
    }

    // Now to copy to device
    hipMemcpy(device_array, host_array,
               sizeof(double)*gsize,
               hipMemcpyHostToDevice);

    // Test
    thread_test<<<grid,block>>>(device_array,device_array);

    // Now to copy back and print
    hipMemcpy(host_array, device_array,
               sizeof(double)*gsize,
               hipMemcpyDeviceToHost);
    
    
/*
    for (int i = 0; i < gsize; i++){
        std::cout << i << '\t' <<  host_array[i] << '\n';
    }
*/
    std::cout << "1024 x 1024 is: " << host_array[gsize-1] << '\n';
    assert(host_array[gsize-1] == 1024*1024-1);

    std::cout << "2d grid tests completed. now for 3d cases" << '\n';

}

// Simple test of CUDA grid stuff
void grid_test3d(){

    std::cout << "testing grid / threads and stuff for 3d" << '\n';

    int max_threads = 128;

    int xDim = 256;
    int yDim = 256;
    int zDim = 256;

    int xD = 1, yD = 1, zD = 1;

    int gsize = xDim * yDim * zDim;

    // Now to set up the CUDA grid / threads
    dim3 block;
    dim3 grid;

    if (xDim <= max_threads){
        block.x = xDim;
        block.y = 1;
        block.z = 1;

        xD = 1;
        yD = yDim;
        zD = zDim;
    } 
    else{
        int count = 0;
        int dim_tmp = xDim;
        while (dim_tmp > max_threads){
            count++;
            dim_tmp /= 2;
        }

        std::cout << "count is: " << count << '\n';

        block.x = dim_tmp;
        block.y = 1;
        block.z = 1;
        xD = pow(2,count);
        yD = yDim;
        zD = zDim;
    }

    std::cout << "threads in x are: " << block.x << '\n';
    std::cout << "dimensions are: " << xD << '\t' << yD << '\t' << zD << '\n';

    grid.x=xD; 
    grid.y=yD; 
    grid.z=zD; 

    int total_threads = block.x * block.y * block.z;

    // Now we need to initialize our double * and send it to the gpu
    double *host_array, *device_array;
    host_array = (double *) malloc(sizeof(double)*gsize);
    hipMalloc((void**) &device_array, sizeof(double)*gsize);

    // initializing 2d array
    for (int i = 0; i < gsize; i++){
        host_array[i] = -1;
    }

    // Now to copy to device
    hipMemcpy(device_array, host_array,
               sizeof(double)*gsize,
               hipMemcpyHostToDevice);

    // Test
    thread_test<<<grid,block>>>(device_array,device_array);

    // Now to copy back and print
    hipMemcpy(host_array, device_array,
               sizeof(double)*gsize,
               hipMemcpyDeviceToHost);
    
    
/*
    for (int i = 0; i < gsize; i++){
        std::cout << i << '\t' <<  host_array[i] << '\n';
    }
*/
    std::cout << "256x256x256 is: " << host_array[gsize-1] << '\n';
    assert(host_array[gsize-1] == 256*256*256-1);

    std::cout << "3d grid tests completed. now for 3d cases" << '\n';

}

// Test of the parSum function in 3d
void parSum_test(){

    // Setting error
    hipError_t err;

    // first, we need to initialize the Grid and Cuda classes
    Grid par;

    // 2D test first

    // For now, we will assume an 8x8 array for summing
    dim3 threads(16, 1, 1);
    int total_threads = threads.x*threads.y*threads.z;

    par.store("dimnum", 2);
    par.store("xDim", 64);
    par.store("yDim", 64);
    par.store("zDim", 1);
    par.store("dx",1.0);
    par.store("dy",1.0);
    par.store("dz",1.0);
    par.threads = threads;

    // Now we need to initialize the grid for the getGid3d3d kernel
    int gsize = 64*64;
    dim3 grid;
    grid.x = 4;
    grid.y = 64;

    par.grid = grid;

    // now we need to initialize the wfc to all 1's;
    double2 *wfc, *host_sum;
    wfc = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * gsize);
    host_sum = (hipfftDoubleComplex *) 
               malloc(sizeof(hipfftDoubleComplex) * gsize / total_threads);

    // init wfc
    for (int i = 0; i < gsize; i++){
        wfc[i].x = 1;
        wfc[i].y = 0;
    }

    double2 *gpu_wfc;
    hipMalloc((void**) &gpu_wfc, sizeof(hipfftDoubleComplex)*gsize);

    // copying wfc to device
    err = hipMemcpy(gpu_wfc, wfc, sizeof(hipfftDoubleComplex)*gsize,
                     hipMemcpyHostToDevice);

    if (err!=hipSuccess){
        std::cout << "ERROR: Could not copy wfc to device!" << '\n';
    }

    // Creating parsum on device
    double2 *par_sum;
    hipMalloc((void**) &par_sum, 
                   sizeof(hipfftDoubleComplex)*gsize/total_threads);

    parSum(gpu_wfc, par_sum, par);

    // copying parsum back
    err = hipMemcpy(host_sum, par_sum, 
                     sizeof(hipfftDoubleComplex)*gsize / total_threads, 
                     hipMemcpyDeviceToHost);
    if (err!=hipSuccess){
        std::cout << err << '\n';
        std::cout << "ERROR: Could not copy par_sum to the host!" << '\n';
        exit(1);
    }

    // The output value should be 4096
    std::cout << "2d parSum is:" << '\n';
    std::cout << host_sum[0].x << " + " << host_sum[0].y << " i" << '\n';

    if (host_sum[0].x != 4096){
        std::cout << "parSum 2d test has failed! Sum is: "
                  << host_sum[0].x << '\n';
        assert((int)host_sum[0].x == 4096);
    }

    // Now for the 3d case
    // For now, we will assume a 16x16x16 array for summing
    par.store("dimnum", 3);
    par.store("xDim", 16);
    par.store("yDim", 16);
    par.store("zDim", 16);
    par.store("dx",1.0);
    par.store("dy",1.0);
    par.store("dz",1.0);

    // Now we need to initialize the grid for the getGid3d3d kernel
    grid.x = 1;
    grid.y = 16;
    grid.z = 16;

    par.grid = grid;

    // copying host wfc back to device
    err = hipMemcpy(gpu_wfc, wfc, sizeof(hipfftDoubleComplex)*gsize,
                     hipMemcpyHostToDevice);

    parSum(gpu_wfc, par_sum, par);

    // copying parsum back
    err = hipMemcpy(host_sum, par_sum, 
                     sizeof(hipfftDoubleComplex)*gsize / total_threads, 
                     hipMemcpyDeviceToHost);
    if (err!=hipSuccess){
        std::cout << "ERROR: Could not copy par_sum to the host!" << '\n';
        exit(1);
    }

    std::cout << "3d parSum is:" << '\n';
    std::cout << host_sum[0].x << " + " << host_sum[0].y << " i" << '\n';

    if (host_sum[0].x != 4096){
        std::cout << "parSum 3d test has failed!" << '\n';
        assert((int)host_sum[0].x == 4096);
    }

}

// Test for the Grid structure with paramters in it
// Initialize all necessary variables and read them back out
void parameter_test(){
    // For this test, we simply need to read in and out stuff from each 
    // class and structure in ds.h / ds.cc
    
    // Certain variables will be used multiple times. 
    double *dstar_var;
    dstar_var = (double *)malloc(sizeof(double) * 5);
    hipfftDoubleComplex *cdc_var;
    cdc_var = (hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex) * 5);
    for (int i = 0; i < 5; ++i){
        dstar_var[i] = (double)i * 0.5;
        cdc_var[i].x = (double)i * 0.5;
        cdc_var[i].y = (double)i * 0.5;
    }

    double dvar = 1.05;
    int ivar = 5;
    bool bvar = true;

    // Now testing the Grid class
    Grid grid_test;
    grid_test.store("dstar_var",dstar_var);
    grid_test.store("dvar", dvar);
    grid_test.store("ivar", ivar);
    grid_test.store("bvar", bvar);

    assert(dstar_var == grid_test.dsval("dstar_var"));
    assert(dvar == grid_test.dval("dvar"));
    assert(ivar == grid_test.ival("ivar"));
    assert(bvar == grid_test.bval("bvar"));

    std::cout << "Grid class checked, now checking the Cuda class..." << '\n';

    std::cout << "All data structures checked" << '\n';

}

// Test for the parsing function
void parser_test(){

    // Testing the command-line parser with defaults and with modifications
    std::cout << "Testing command-line parser with no arguments..." << '\n';

    // First testing default values in and out of the parser function
    char *fake_noargv[] = {NULL};
    Grid noarg_grid;
    noarg_grid = parseArgs(0,fake_noargv);

    // Checking contents of noarg_grid:
    assert(noarg_grid.ival("xDim") == 256);
    assert(noarg_grid.ival("yDim") == 256);
    assert(noarg_grid.ival("zDim") == 256);
    assert(noarg_grid.dval("omega") == 0);
    assert(noarg_grid.dval("gammaY") == 1.0);
    assert(noarg_grid.ival("gsteps") == 1);
    assert(noarg_grid.ival("esteps") == 1);
    assert(noarg_grid.dval("gdt") == 1e-4);
    assert(noarg_grid.dval("dt") == 1e-4);
    assert(noarg_grid.ival("device") == 0);
    assert(noarg_grid.ival("atoms") == 1);
    assert(noarg_grid.bval("read_wfc") == false);
    assert(noarg_grid.ival("printSteps") == 100);
    assert(noarg_grid.dval("winding") == 0);
    assert(noarg_grid.bval("corotating") == false);
    assert(noarg_grid.bval("gpe") == false);
    assert(noarg_grid.dval("omegaZ") == 6.283);
    assert(noarg_grid.dval("interaction") == 1);
    assert(noarg_grid.dval("laser_power") == 0);
    assert(noarg_grid.dval("angle_sweep") == 0);
    assert(noarg_grid.ival("kick_it") == 0);
    assert(noarg_grid.bval("write_it") == false);
    assert(noarg_grid.dval("x0_shift") == 0);
    assert(noarg_grid.dval("y0_shift") == 0);
    assert(noarg_grid.dval("z0_shift") == 0);
    assert(noarg_grid.dval("sepMinEpsilon") == 0);
    assert(noarg_grid.bval("graph") == false);
    assert(noarg_grid.bval("unit_test") == false);
    assert(noarg_grid.dval("omegaX") == 6.283);
    assert(noarg_grid.dval("omegaY") == 6.283);
    assert(noarg_grid.sval("data_dir") == "data/");
    assert(noarg_grid.bval("ramp") == false);
    assert(noarg_grid.ival("ramp_type") == 1);
    assert(noarg_grid.ival("dimnum") == 2);
    assert(noarg_grid.bval("write_file") == true);
    assert(noarg_grid.dval("fudge") == 1.0);
    assert(noarg_grid.ival("kill_idx") == -1);
    assert(noarg_grid.dval("DX") == 0.0);
    assert(noarg_grid.dval("mask_2d") == 1.5e-4);
    assert(noarg_grid.dval("box_size") == 2.5e-5);
    assert(noarg_grid.bval("found_sobel") == false);
    assert(noarg_grid.Afn == "rotation");
    assert(noarg_grid.Kfn == "rotation_K");
    assert(noarg_grid.Vfn == "2d");
    assert(noarg_grid.Wfcfn == "2d");
    assert(noarg_grid.sval("conv_type") == "FFT");
    assert(noarg_grid.ival("charge") == 0);
    assert(noarg_grid.bval("flip") == false);

    // Now testing all values specified by command-line arguments
    std::cout << "Testing command-line parser with all arguments..." << '\n';
    std::vector<std::string> argarray(10);

    // I apologize for the mess... If you have a better way of creating the 
    // char ** for this without running into memory issues, let me know!
    char *fake_fullargv[] = {strdup("./gpue"), 
                             strdup("-A"), strdup("rotation"), 
                             strdup("-a"),
                             strdup("-b"), strdup("2.5e-5"), 
                             strdup("-C"), strdup("0"), 
                             strdup("-c"), strdup("3"), 
                             strdup("-D"), strdup("data"), 
                             strdup("-E"), 
                             strdup("-e"), strdup("1"), 
                             strdup("-f"), 
                             strdup("-G"), strdup("1"),
                             strdup("-g"), strdup("1"), 
                             strdup("-i"), strdup("1"), 
                             strdup("-K"), strdup("0"), 
                             strdup("-k"), strdup("0"),
                             strdup("-L"), strdup("0"), 
                             strdup("-l"), 
                             strdup("-n"), strdup("1"), 
                             strdup("-O"), strdup("0"),
                             strdup("-P"), strdup("0"), 
                             strdup("-p"), strdup("100"),
                             strdup("-Q"), strdup("0"), 
                             strdup("-q"), strdup("0"), 
                             strdup("-R"), strdup("1"), 
                             //strdup("-r"),
                             strdup("-S"), strdup("0"), 
                             strdup("-s"),
                             strdup("-T"), strdup("1e-4"), 
                             strdup("-t"), strdup("1e-4"), 
                             strdup("-U"), strdup("0"), 
                             strdup("-V"), strdup("0"), 
                             strdup("-W"), 
                             strdup("-w"), strdup("0"), 
                             strdup("-X"), strdup("1.0"),
                             strdup("-x"), strdup("256"), 
                             strdup("-Y"), strdup("1.0"), 
                             strdup("-y"), strdup("256"),
                             strdup("-Z"), strdup("6.283"), 
                             strdup("-z"), strdup("256"), 
                             NULL};
    int fake_argc = sizeof(fake_fullargv) / sizeof(char *) - 1;

    // Now to read into gpue and see what happens
    Grid fullarg_grid;
    fullarg_grid = parseArgs(fake_argc, fake_fullargv);

    // Checking contents of fullarg_grid:
    assert(fullarg_grid.ival("xDim") == 256);
    assert(fullarg_grid.ival("yDim") == 256);
    assert(fullarg_grid.ival("zDim") == 256);
    assert(fullarg_grid.dval("omega") == 0);
    assert(fullarg_grid.dval("gammaY") == 1.0);
    assert(fullarg_grid.ival("gsteps") == 1);
    assert(fullarg_grid.ival("esteps") == 1);
    assert(fullarg_grid.dval("gdt") == 1e-4);
    assert(fullarg_grid.dval("dt") == 1e-4);
    assert(fullarg_grid.ival("device") == 0);
    assert(fullarg_grid.ival("atoms") == 1);
    assert(fullarg_grid.bval("read_wfc") == false);
    assert(fullarg_grid.ival("printSteps") == 100);
    assert(fullarg_grid.dval("winding") == 0);
    assert(fullarg_grid.bval("corotating") == true);
    assert(fullarg_grid.bval("gpe") == true);
    assert(fullarg_grid.dval("omegaZ") == 6.283);
    assert(fullarg_grid.dval("interaction") == 1);
    assert(fullarg_grid.dval("laser_power") == 0);
    assert(fullarg_grid.dval("angle_sweep") == 0);
    assert(fullarg_grid.ival("kick_it") == 0);
    assert(fullarg_grid.bval("write_it") == true);
    assert(fullarg_grid.dval("x0_shift") == 0);
    assert(fullarg_grid.dval("y0_shift") == 0);
    assert(fullarg_grid.dval("z0_shift") == 0);
    assert(fullarg_grid.dval("sepMinEpsilon") == 0);
    assert(fullarg_grid.bval("graph") == true);
    assert(fullarg_grid.bval("unit_test") == false);
    assert(fullarg_grid.dval("omegaX") == 1.0);
    assert(fullarg_grid.dval("omegaY") == 1.0);
    assert(fullarg_grid.sval("data_dir") == "data/");
    assert(fullarg_grid.bval("ramp") == true);
    assert(fullarg_grid.ival("ramp_type") == 1);
    assert(fullarg_grid.ival("dimnum") == 3);
    assert(fullarg_grid.bval("write_file") == false);
    assert(fullarg_grid.dval("fudge") == 1.0);
    assert(fullarg_grid.ival("kill_idx") == 0);
    assert(fullarg_grid.dval("DX") == 0.0);
    assert(fullarg_grid.dval("mask_2d") == 1.5e-4);
    assert(fullarg_grid.dval("box_size") == 2.5e-5);
    assert(fullarg_grid.bval("found_sobel") == false);
    assert(fullarg_grid.Afn == "rotation");
    assert(fullarg_grid.Kfn == "rotation_K3d");
    assert(fullarg_grid.Vfn == "3d");
    assert(fullarg_grid.Wfcfn == "3d");
    assert(fullarg_grid.sval("conv_type") == "FFT");
    assert(fullarg_grid.ival("charge") == 0);
    assert(fullarg_grid.bval("flip") == true);

}

// Testing the evolve_2d function in evolution.cu
void evolve_2d_test(){
    // First, we need to create all the necessary data structures for the
    // The evolve_2d function, FOLLOWING INIT.CU

    std::cout << "Testing the evolve_2d function" << '\n';

    // Note: the omega_z value (-o flag) is arbitrary
    char * fake_argv[] = {strdup("./gpue"), 
                          strdup("-C"), strdup("0"), 
                          strdup("-e"), strdup("2.01e4"), 
                          strdup("-G"), strdup("1.0"), 
                          strdup("-g"), strdup("0"), 
                          strdup("-i"), strdup("1.0"), 
                          strdup("-k"), strdup("0"), 
                          strdup("-L"), strdup("0"), 
                          strdup("-n"), strdup("1e6"), 
                          strdup("-O"), strdup("0.0"), 
                          strdup("-Z"), strdup("10.0"), 
                          strdup("-P"), strdup("0.0"), 
                          strdup("-p"), strdup("1000"), 
                          strdup("-S"), strdup("0.0"), 
                          strdup("-T"), strdup("1e-4"), 
                          strdup("-t"), strdup("1e-4"), 
                          strdup("-U"), strdup("0"), 
                          strdup("-V"), strdup("0"), 
                          strdup("-w"), strdup("0.0"), 
                          strdup("-X"), strdup("1.0"), 
                          strdup("-x"), strdup("256"), 
                          strdup("-Y"), strdup("1.0"), 
                          strdup("-y"), strdup("256"), 
                          strdup("-W"), 
                          strdup("-D"), strdup("data"), NULL};
    int fake_argc = sizeof(fake_argv) / sizeof(char *) - 1;

    // Now to read into gpue and see what happens
    Grid par;
    par = parseArgs(fake_argc, fake_argv);

    std::cout << "omegaX is: " << par.dval("omegaX") << '\n';
    std::cout << "x / yDim are: " << par.ival("xDim") << '\t' 
              << par.ival("yDim") << '\n';
    int device = par.ival("device");
    hipSetDevice(device);

    std::string buffer;

    //************************************************************//
    /*
    * Initialise the Params data structure to track params and variables
    */
    //************************************************************//

    init(par);

    // Re-establishing variables from parsed Grid class
    double dx = par.dval("dx");
    double dy = par.dval("dy");
    double *x = par.dsval("x");
    double *y = par.dsval("y");
    double *V_opt = par.dsval("V_opt");
    double *pAy = par.dsval("pAy");
    double *pAx = par.dsval("pAx");
    double *pAy_gpu = par.dsval("pAy_gpu");
    double *pAx_gpu = par.dsval("pAx_gpu");
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    bool read_wfc = par.bval("read_wfc");
    int gsteps = par.ival("gsteps");
    int esteps = par.ival("esteps");
    hipfftDoubleComplex *wfc = par.cufftDoubleComplexval("wfc");
    hipfftDoubleComplex *V_gpu = par.cufftDoubleComplexval("V_gpu");
    hipfftDoubleComplex *GK = par.cufftDoubleComplexval("GK");
    hipfftDoubleComplex *GV = par.cufftDoubleComplexval("GV");
    hipfftDoubleComplex *EV = par.cufftDoubleComplexval("EV");
    hipfftDoubleComplex *EK = par.cufftDoubleComplexval("EK");
    hipfftDoubleComplex *EpAy = par.cufftDoubleComplexval("EpAy");
    hipfftDoubleComplex *EpAx = par.cufftDoubleComplexval("EpAx");
    hipfftDoubleComplex *GpAx = par.cufftDoubleComplexval("GpAx");
    hipfftDoubleComplex *GpAy = par.cufftDoubleComplexval("GpAy");
    hipfftDoubleComplex *wfc_gpu = par.cufftDoubleComplexval("wfc_gpu");
    hipfftDoubleComplex *K_gpu = par.cufftDoubleComplexval("K_gpu");
    hipfftDoubleComplex *par_sum = par.cufftDoubleComplexval("par_sum");
    hipError_t err;

    std::cout << "variables re-established" << '\n';
    std::cout << read_wfc << '\n';

    std::cout << "omegaY is: " << par.ival("omegaY") << '\t'
              << "omegaX is: " << par.dval("omegaX") << '\n';

/*
    for (int i = 0; i < xDim * yDim; ++i){
        std::cout << i << '\t' << wfc[i].x << '\t' << wfc[i].y << '\n';
    }
*/

    std::cout << "gsteps: " << gsteps << '\n';
   
    if(gsteps > 0){
        err=hipMemcpy(K_gpu, GK, sizeof(hipfftDoubleComplex)*xDim*yDim,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy K_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(V_gpu, GV, sizeof(hipfftDoubleComplex)*xDim*yDim,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy V_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(pAy_gpu, GpAy, sizeof(hipfftDoubleComplex)*xDim*yDim,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy pAy_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(pAx_gpu, GpAx, sizeof(hipfftDoubleComplex)*xDim*yDim,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy pAx_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(wfc_gpu, wfc, sizeof(hipfftDoubleComplex)*xDim*yDim,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy wfc_gpu to device" << '\n';
            exit(1);
        }
    
        evolve_2d(par, par_sum, gsteps, 0, buffer);
        wfc = par.cufftDoubleComplexval("wfc");
        wfc_gpu = par.cufftDoubleComplexval("wfc_gpu");
        hipMemcpy(wfc, wfc_gpu, sizeof(hipfftDoubleComplex)*xDim*yDim,
                   hipMemcpyDeviceToHost);
    }

    std::cout << GV[0].x << '\t' << GK[0].x << '\t'
              << pAy[0] << '\t' << pAx[0] << '\n';

    //free(GV); free(GK); free(pAy); free(pAx);

    // Re-initializing wfc after evolution
    wfc = par.cufftDoubleComplexval("wfc");
    wfc_gpu = par.cufftDoubleComplexval("wfc_gpu");

    std::cout << "evolution started..." << '\n';
    std::cout << "esteps: " << esteps << '\n';

    //************************************************************//
    /*
    * Evolution
    */
    //************************************************************//
    if(esteps > 0){
        err=hipMemcpy(pAy_gpu, EpAy, sizeof(hipfftDoubleComplex)*xDim*yDim,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy pAy_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(pAx_gpu, EpAx, sizeof(hipfftDoubleComplex)*xDim*yDim,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy pAx_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(K_gpu, EK, sizeof(hipfftDoubleComplex)*xDim*yDim,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy K_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(V_gpu, EV, sizeof(hipfftDoubleComplex)*xDim*yDim,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy V_gpu to device" << '\n';
            exit(1);
        }
        err=hipMemcpy(wfc_gpu, wfc, sizeof(hipfftDoubleComplex)*xDim*yDim,
                       hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            std::cout << "ERROR: Could not copy wfc_gpu to device" << '\n';
            exit(1);
        }

        evolve_2d(par, par_sum,
               esteps, 1, buffer);

    }

    std::cout << "done evolving, checking result" << '\n';

    // At this point, we have a wavefunction that is testable, which we will be
    // doing in much the same way as in the linear/perf branch of GPUE.
    // For this, we must recreate the en.py file in a testable format in cpp
    // Note that we could be using the GPUs for this, but because it is a unit
    // test and we do not care that much about perfomance, we will be using the 
    // CPU instead. We may later add in the appropriate GPU kernels.

    // We first need to grab the wavefunctions from the evolve_2d function
    // After evolution
    wfc = par.cufftDoubleComplexval("wfc");
    wfc_gpu = par.cufftDoubleComplexval("wfc_gpu");
    unsigned int gSize = xDim * yDim;

    // Now to grab K and V, note that these are different than the values used 
    // for K / V_gpu or for E / G K / V in the evolve_2d function
    // The additional 0 in the gpu variable name indicate this (sorry)
    double *K_0_gpu = par.dsval("K");
    double *K = par.dsval("K");
    double *V_0_gpu = par.dsval("V");
    double *V = par.dsval("V");

    // Now we need som CUDA specific variables for the kernels later on...
    int threads = par.ival("threads");
    dim3 grid = par.grid;

    // Momentum-space (p) wavefunction
    double2 *wfc_p = wfc;
    double2 *wfc_p_gpu = wfc_gpu;

    // Conjugate (c) wavefunction
    double2 *wfc_c = wfc;
    double2 *wfc_c_gpu = wfc_gpu;

    // Energies
    double2 *Energy_1, *Energy_2, *Energy_k, *Energy_v;
    Energy_1 = wfc_gpu;
    Energy_2 = wfc_gpu;

    // Plan for 2d FFT
    hipfftHandle plan_2d = par.ival("plan_2d");

    std::cout << "allocating space on device..." << '\n';

    // Allocating space on GPU
    hipMalloc((void **) &wfc_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void **) &K_0_gpu, sizeof(double) * gSize);
    hipMalloc((void **) &V_0_gpu, sizeof(double) * gSize);
    hipMalloc((void **) &wfc_p_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void **) &wfc_c_gpu, sizeof(hipfftDoubleComplex) * gSize);
    hipMalloc((void **) &par_sum, sizeof(hipfftDoubleComplex)*(gSize/threads));

    std::cout << "copying contents... " << '\n';

    // Copy variables over to device
    hipMemcpy(wfc_gpu, wfc, sizeof(hipfftDoubleComplex) * gSize,
               hipMemcpyHostToDevice);
    std::cout << "wfc copied..." << '\n';
    hipMemcpy(K_0_gpu, K, sizeof(hipfftDoubleComplex) * gSize,
               hipMemcpyHostToDevice);
    std::cout << "K copied..." << '\n';
    hipMemcpy(V_0_gpu, GV, sizeof(hipfftDoubleComplex) * gSize,
               hipMemcpyHostToDevice);
    std::cout << "V copied..." << '\n';
    hipMemcpy(wfc_p_gpu, wfc_p, sizeof(hipfftDoubleComplex) * gSize,
               hipMemcpyHostToDevice);
    std::cout << "wfc_p copied..." << '\n';
    hipMemcpy(wfc_c_gpu, wfc_c, sizeof(hipfftDoubleComplex) * gSize,
               hipMemcpyHostToDevice);
    std::cout << "wfc_c copied..." << '\n';

    std::cout << "performing energy calculations..." << '\n';


    // In the example python code, it was necessary to reshape everything, 
    // But let's see what happens if I don't do that here...

    // FFT for the wfc in momentum-space
    hipfftExecZ2Z(plan_2d, wfc_gpu, wfc_p, HIPFFT_FORWARD);

    // Conjugate for the wfc
    vecConjugate<<<grid,threads>>>(wfc_gpu, wfc_c);

    // K * wfc
    vecMult<<<grid,threads>>>(wfc_gpu,K_0_gpu,wfc_p);
    hipfftExecZ2Z(plan_2d, wfc_p, Energy_1, HIPFFT_BACKWARD); 

    vecMult<<<grid,threads>>>(wfc_gpu, V_0_gpu, Energy_2);

/*
    for (int i = 0; i < xDim * yDim; ++i){
        std::cout << Energy_1[i].y << '\t' << Energy_2[i].x << '\n';
    }
*/

    //std::cout << wfc_gpu[0].x << '\t' << wfc_gpu[0].y << '\n';

    free(EV); free(EK); free(EpAy); free(EpAx);
    free(x);free(y);
    hipFree(wfc_gpu); hipFree(K_gpu); hipFree(V_gpu); hipFree(pAx_gpu);
    hipFree(pAy_gpu); hipFree(par_sum);

    std::cout << "Evolution test complete." << '\n';
    std::cout << "EVOLUTION TEST UNFINISHED!" << '\n';
    
}

void vortex3d_test(){

    std::cout << "Testing functions in vortex_3d..." << '\n';

    // setting up array for scan_2d() thresholding test
    // We are creating 
    double *array, *darray;
    bool *barray, *dbarray;
    bool *dcheck, *check, *sum, *dsum;
    int dim = 8;
    double threshold = 0.5;

    array = (double *)malloc(sizeof(double)*dim*dim*dim);
    barray = (bool *)malloc(sizeof(bool)*dim*dim*dim);
    sum = (bool *)malloc(sizeof(bool)*dim*dim*dim);
    check = (bool *)malloc(sizeof(bool)*dim*dim*dim);

    hipMalloc((void **) &darray, sizeof(double)*dim*dim*dim);
    hipMalloc((void **) &dbarray, sizeof(bool)*dim*dim*dim);
    hipMalloc((void **) &dcheck, sizeof(bool)*dim*dim*dim);
    hipMalloc((void **) &dsum, sizeof(bool)*dim*dim*dim);

    for (int i = 0; i < dim; ++i){
        for (int j = 0; j < dim; ++j){
            for (int k = 0; k < dim; ++k){
                int index = k + j * dim + i * dim * dim;
                if (k == dim / 2){
                    array[index] = 1;
                }
                else{
                    array[index] = 0;
                }
                if (k > dim / 2){
                    barray[index] = 1;
                }
                else{
                    barray[index] = 0;
                }
                sum[index] = 0;
            }
        }
    }

    hipMemcpy(darray, array, sizeof(double)*dim*dim*dim, 
               hipMemcpyHostToDevice);
    hipMemcpy(dbarray, barray, sizeof(bool)*dim*dim*dim, 
               hipMemcpyHostToDevice);
    hipMemcpy(dsum, sum, sizeof(bool)*dim*dim*dim, 
               hipMemcpyHostToDevice);

    dim3 grid = {1, dim, dim};
    dim3 threads = {dim, 1, 1};

    std::cout << "All arrays initialized\n";

    // Now to create the grid and threads
    std::cout << "summing along x\n";
    dim3 temp_grid = {1, dim, 1};
    dim3 temp_threads = {dim, 1, 1};
    scan_2d<<<temp_grid, temp_threads>>>(darray, dcheck, threshold, 0, dim); 

    threshold_sum<<<grid, threads>>>(dsum, dcheck, dsum);
    
    std::cout << "summing along y\n";
    scan_2d<<<temp_grid, temp_threads>>>(darray, dcheck, threshold, 1, dim); 

    threshold_sum<<<grid, threads>>>(dsum, dcheck, dsum);

    std::cout << "summing along z\n";
    scan_2d<<<temp_grid, temp_threads>>>(darray, dcheck, threshold, 2, dim); 

    threshold_sum<<<grid, threads>>>(dsum, dcheck, dsum);

    bool *ans, *dans;
    ans = (bool *)malloc(sizeof(bool));
    ans[0] = 0;
    hipMalloc((void **) &dans, sizeof(bool));

    is_eq<<<grid, threads>>>(dsum, dbarray, dans);

    hipMemcpy(ans, dans, sizeof(bool), hipMemcpyDeviceToHost);
/*
    hipMemcpy(sum, dsum, sizeof(bool)*dim*dim*dim, hipMemcpyDeviceToHost);

    for (int i = 0; i < dim*dim*dim; ++i){
        std::cout << sum[i] << '\t' << barray[i] << '\n';
    }
*/

    if (ans[0]){
        std::cout << "scan_2d function for vortex tracking succeeded!" << '\n';
    }
    else{
        std::cout << "scan_2d function for vortex tracking failed!" << '\n';
        exit(1);
    }
    
}

__global__ void make_complex_kernel(double *in, int *evolution_type, 
                                    double2 *out){

    //int id = threadIdx.x + blockIdx.x*blockDim.x;
    //out[id] = make_complex(in[id], evolution_type[id]);
    for (int i = 0; i < 3; ++i){
        out[i] = make_complex(in[i], evolution_type[i]);
    }
}

void make_complex_test(){

    // Creating a simple array to hold the 3 possible make_complex options
    double *input_array, *dinput_array;
    double2 *output_array, *doutput_array;
    int *evolution_type, *devolution_type;

    input_array = (double *)malloc(sizeof(double)*3);
    output_array = (double2 *)malloc(sizeof(double2)*3);
    evolution_type = (int *)malloc(sizeof(int)*3);

    input_array[0] = 10;
    input_array[1] = 10;
    input_array[2] = 10;

    evolution_type[0] = 0;
    evolution_type[1] = 1;
    evolution_type[2] = 2;

    hipMalloc((void **)&dinput_array, sizeof(double)*3);
    hipMalloc((void **)&doutput_array, sizeof(double2)*3);
    hipMalloc((void **)&devolution_type, sizeof(int)*3);

    hipMemcpy(dinput_array, input_array, sizeof(double)*3,
               hipMemcpyHostToDevice);
    hipMemcpy(devolution_type, evolution_type, sizeof(int)*3,
               hipMemcpyHostToDevice);

    dim3 threads = {1,1,1};
    dim3 grid = {1,1,1};

    make_complex_kernel<<<1,1>>>(dinput_array, devolution_type,
                                           doutput_array);
    hipDeviceSynchronize();

    hipMemcpy(output_array, doutput_array, sizeof(double2)*3, 
               hipMemcpyDeviceToHost);

    bool pass = true;
    double thresh = 0.000001;

    if (abs(output_array[0].x - input_array[0]) > thresh || 
        (output_array[0].y) > thresh){
        std::cout << "failed 1\n";
        pass = false;
    }
    if (abs(output_array[1].x - exp(-input_array[1])) > thresh || 
        abs(output_array[1].y) > thresh){
        std::cout << "failed 2\n";
        pass = false;
    }
    if (abs(output_array[2].x - cos(-input_array[2])) > thresh || 
        abs(output_array[2].y - sin(-input_array[2])) > thresh){
        std::cout << "failed 3\n";
        pass = false;
    }

    if(pass){
        std::cout << "make_complex test passed!\n";
    }
    else{
        std::cout << "make_complex test failed!\n";
        exit(1);
    }
    
}

void cMultPhi_test(){
    // first, we are creating a double2 array to work with
    int n = 32;
    double2 *in1, *out;
    double *in2;
    double2 *din1, *dout;
    double *din2;

    in1 = (double2 *)malloc(sizeof(double2)*n);
    in2 = (double *)malloc(sizeof(double)*n);
    out = (double2 *)malloc(sizeof(double2)*n);

    hipMalloc((void **)&din1, sizeof(double2)*n);
    hipMalloc((void **)&din2, sizeof(double)*n);
    hipMalloc((void **)&dout, sizeof(double2)*n);

    for (int i = 0; i < n; ++i){
        in1[i].x = i;
        in1[i].y = n-i;
        in2[i] = n-i;
    }

    hipMemcpy(din1, in1, sizeof(double2)*n, hipMemcpyHostToDevice);
    hipMemcpy(din2, in2, sizeof(double)*n, hipMemcpyHostToDevice);

    cMultPhi<<<1,n>>>(din1, din2, dout);
    hipDeviceSynchronize();

    hipMemcpy(out, dout, sizeof(double2)*n, hipMemcpyDeviceToHost);

    double thresh = 0.000001;
    bool result = true;
    for (int i = 0; i < n; ++i){
        if (abs(out[i].x-cos(in2[i])*in1[i].x-in1[i].y*sin(in2[i])) < thresh ||
            abs(out[i].y-in1[i].x*sin(in2[i])+in1[i].y*cos(in2[i])) < thresh){
            result = false;
        }
    }

    if (result){
        std::cout << "cMultPhi test passed!\n";
    }
    else{
        std::cout << "cMultPhi test failed!\n";
        exit(1);
    }

}

void cMultDens_test(){
    // first, we are creating a double2 array to work with
    double thresh = 0.001;
    int n = 32;
    double2 *in1, *in2, *out;
    double2 *din1, *din2, *dout;

    in1 = (double2 *)malloc(sizeof(double2)*n);
    in2 = (double2 *)malloc(sizeof(double2)*n);
    out = (double2 *)malloc(sizeof(double2)*n);

    hipMalloc((void **)&din1, sizeof(double2)*n);
    hipMalloc((void **)&din2, sizeof(double2)*n);
    hipMalloc((void **)&dout, sizeof(double2)*n);

    for (int i = 0; i < n; ++i){
        in1[i].x = i;
        in1[i].y = n-i;
        in2[i].x = n-i;
        in2[i].y = i;
    }

    hipMemcpy(din1, in1, sizeof(double2)*n, hipMemcpyHostToDevice);
    hipMemcpy(din2, in2, sizeof(double2)*n, hipMemcpyHostToDevice);

    // Testing imaginary-time evolution
    cMultDensity<<<1, n>>>(din1, din2, dout, 1, 1, 0, 1);
    hipDeviceSynchronize();

    hipMemcpy(out, dout, sizeof(double2)*n, hipMemcpyDeviceToHost);

    bool result = true;
    for (int i = 0; i < n; ++i){
        double gDensity = (in2[i].x*in2[i].x + in2[i].y*in2[i].y)/HBAR;
        if (abs(out[i].x-(in1[i].x*exp(-gDensity)*in2[i].x
                                      -in1[i].y*in2[i].y)) > thresh ||
            abs(out[i].y-(in1[i].x*exp(-gDensity)*in2[i].y
                                      +in1[i].y*in2[i].x)) > thresh){
            result = false;
        }
    }

    if (result){
        std::cout << "cMultDens imaginary time test passed!\n";
    }
    else{
        std::cout << "cMultDens imaginary time test failed!\n";
        exit(1);
    }

    // Testing real-time evolution
    cMultDensity<<<1, n>>>(din1, din2, dout, 1, 1, 1, 1);
    hipDeviceSynchronize();

    hipMemcpy(out, dout, sizeof(double2)*n, hipMemcpyDeviceToHost);

    result = true;
    for (int i = 0; i < n; ++i){
        double2 tmp;
        double gDensity = (in2[i].x*in2[i].x + in2[i].y*in2[i].y)/HBAR;
        tmp.x = in1[i].x*cos(-gDensity) - in1[i].y*sin(-gDensity);
        tmp.y = in1[i].y*cos(-gDensity) + in1[i].x*sin(-gDensity);

/*
        std::cout << in1[i].x << '\t' << in1[i].y << '\t' << tmp.x << '\t' << tmp.y << '\t' << gDensity << '\n';
        std::cout << out[i].x - (tmp.x*in2[i].x - tmp.y*in2[i].y) << '\t'
                  << out[i].y - (tmp.x*in2[i].y + tmp.y*in2[i].x) << '\n';
*/

        if (abs(out[i].x - (tmp.x*in2[i].x - tmp.y*in2[i].y)) > thresh ||
            abs(out[i].y - (tmp.x*in2[i].y + tmp.y*in2[i].x)) > thresh){
                std::cout << in1[i].x << '\t' << in1[i].y << '\t' << tmp.x << '\t' << tmp.y << '\t' << gDensity << '\n';
                std::cout << out[i].x - (tmp.x*in2[i].x - tmp.y*in2[i].y) << '\t'
                          << out[i].y - (tmp.x*in2[i].y + tmp.y*in2[i].x) << '\n';

            result = false;
        }
    }

    if (result){
        std::cout << "cMultDens real-time test passed!\n";
    }
    else{
        std::cout << "cMultDens real-time test failed!\n";
        exit(1);
    }

}

